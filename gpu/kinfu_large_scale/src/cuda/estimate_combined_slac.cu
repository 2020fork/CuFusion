#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
*
*  Point Cloud Library (PCL) - www.pointclouds.org
*  Copyright (c) 2011, Willow Garage, Inc.
* 
*  All rights reserved.
*
*  Redistribution and use in source and binary forms, with or without
*  modification, are permitted provided that the following conditions
*  are met:
*
*   * Redistributions of source code must retain the above copyright
*     notice, this list of conditions and the following disclaimer.
*   * Redistributions in binary form must reproduce the above
*     copyright notice, this list of conditions and the following
*     disclaimer in the documentation and/or other materials provided
*     with the distribution.
*   * Neither the name of Willow Garage, Inc. nor the names of its
*     contributors may be used to endorse or promote products derived
*     from this software without specific prior written permission.
*
*  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
*  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
*  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
*  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
*  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
*  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
*  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
*  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
*  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
*  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
*  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
*  POSSIBILITY OF SUCH DAMAGE.
*
*/

//#include <pcl/gpu/utils/device/block.hpp>
//#include <pcl/gpu/utils/device/funcattrib.hpp>
#include "device.hpp"

namespace pcl
{
	namespace device
	{
		//typedef double float_type;
		typedef float float_type;

		struct Combined3
		{
			enum
			{
				CTA_SIZE_X = 32,
				CTA_SIZE_Y = 8,
				CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
			};

			struct plus
			{
				__forceinline__ __device__ float
					operator () (const float_type &lhs, const volatile float_type& rhs) const 
				{
					return (lhs + rhs);
				}
			};

			Mat33 Rcurr;
			float3 tcurr;

			PtrStep<float> vmap_curr;
			PtrStep<float> nmap_curr;

			Mat33 Rprev_inv;
			float3 tprev;

			Intr intr;

			PtrStep<float> vmap_g_prev;
			PtrStep<float> nmap_g_prev;

			float distThres;
			float angleThres;

			int cols;
			int rows;

			mutable PtrStep<float_type> gbuf;
			mutable float_type* gbuf_slac_triangle;
			mutable float_type* gbuf_slac_block;
			mutable float_type* gbuf_slac_b;

			__device__ __forceinline__ bool
				search (int x, int y, float3& n, float3& d, float3& s) const
			{
				float3 ncurr;
				ncurr.x = nmap_curr.ptr (y)[x];

				if (isnan (ncurr.x))
					return (false);

				float3 vcurr;
				vcurr.x = vmap_curr.ptr (y       )[x];
				vcurr.y = vmap_curr.ptr (y + rows)[x];
				vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

				float3 vcurr_g = Rcurr * vcurr + tcurr;

				float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

				int2 ukr;         //projection
				ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
				ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

				if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
					return (false);

				float3 nprev_g;
				nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

				if (isnan (nprev_g.x))
					return (false);

				float3 vprev_g;
				vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
				vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
				vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

				float dist = norm (vprev_g - vcurr_g);
				if (dist > distThres)
					return (false);

				ncurr.y = nmap_curr.ptr (y + rows)[x];
				ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

				float3 ncurr_g = Rcurr * ncurr;

				nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
				nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

				float sine = norm (cross (ncurr_g, nprev_g));

				if (sine >= angleThres)
					return (false);
				n = nprev_g;
				d = vprev_g;
				s = vcurr_g;
				return (true);
			}

			__device__ __forceinline__ void
				operator () () const
			{
				int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
				int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

				float3 n, d, s;
				bool found_coresp = false;

				if (x < cols && y < rows)
					found_coresp = search (x, y, n, d, s);

				int coo[3];         //projection
				coo[0] = __float2int_rd ( ( s.x + 1.5 ) / 0.375 );      //4
				coo[1] = __float2int_rd ( ( s.y + 1.5 ) / 0.375 );                      //4
				coo[2] = __float2int_rd ( ( s.z - 0.3 ) / 0.375 );

				float row[7];

				if (found_coresp && coo[0]>=0 && coo[0]<8 && coo[1]>=0 && coo[1]<8 && coo[2]>=0 &&coo[2]<8 )
				{
					*(float3*)&row[0] = cross (s, n);
					*(float3*)&row[3] = n;
					row[6] = dot (n, d - s);
					for ( int i = 0; i < 6; i++ )
						for ( int j = 0; j < 24; j++ )
							atomicAdd( gbuf_slac_block + i * 2187 + coo[0] * 81 + coo[1] * 9 + coo[2], 1.0 );
				}
				else
					row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

				int tid = Block::flattenedThreadId ();

				int shift = 0;
#pragma unroll
				for (int i = 0; i < 6; ++i)        //rows
				{
#pragma unroll
					for (int j = i; j < 7; ++j)          // cols + b
					{
						gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[i]*row[j];
					}
				}
			}
		};

		__global__ void
			combinedKernel3 (const Combined3 cs) 
		{
			cs ();
		}

		struct TranformReduction3
		{
			enum
			{
				CTA_SIZE = 512,
				STRIDE = CTA_SIZE,

				B = 6, COLS = 6, ROWS = 6, DIAG = 6,
				UPPER_DIAG_MAT = (COLS * ROWS - DIAG) / 2 + DIAG,
				TOTAL = UPPER_DIAG_MAT + B,

				GRID_X = TOTAL
			};

			PtrStep<float_type> gbuf;
			int length;
			mutable float_type* output;

			__device__ __forceinline__ void
				operator () () const
			{
				const float_type *beg = gbuf.ptr (blockIdx.x);
				const float_type *end = beg + length;

				int tid = threadIdx.x;

				float_type sum = 0.f;
				for (const float_type *t = beg + tid; t < end; t += STRIDE)
					sum += *t;

				__shared__ float_type smem[CTA_SIZE];

				smem[tid] = sum;
				__syncthreads ();

				Block::reduce<CTA_SIZE>(smem, Combined3::plus ());

				if (tid == 0)
					output[blockIdx.x] = smem[0];
			}
		};

		__global__ void
			TransformEstimatorKernel3 (const TranformReduction3 tr) 
		{
			tr ();
		}
	}
}

void pcl::device::estimateCombinedEx (const Mat33& Rcurr, const float3& tcurr, 
	const MapArr& vmap_curr, const MapArr& nmap_curr, 
	const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
	const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
	float distThres, float angleThres,
	DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf, 
	float_type* matrixA_host, float_type* vectorB_host,
	DeviceArray<float>& gbuf_slac_triangle, DeviceArray<float>& gbuf_slac_block, DeviceArray<float>& gbuf_slac_b,
	float* matrixSLAC_A_host, float* matrixSLAC_block_host, float* vectorSLAC_b_host)
{
	int cols = vmap_curr.cols ();
	int rows = vmap_curr.rows () / 3;
	dim3 block (Combined3::CTA_SIZE_X, Combined3::CTA_SIZE_Y);
	dim3 grid (1, 1, 1);
	grid.x = divUp (cols, block.x);
	grid.y = divUp (rows, block.y);

	Combined3 cs3;

	cs3.Rcurr = Rcurr;
	cs3.tcurr = tcurr;

	cs3.vmap_curr = vmap_curr;
	cs3.nmap_curr = nmap_curr;

	cs3.Rprev_inv = Rprev_inv;
	cs3.tprev = tprev;

	cs3.intr = intr;

	cs3.vmap_g_prev = vmap_g_prev;
	cs3.nmap_g_prev = nmap_g_prev;

	cs3.distThres = distThres;
	cs3.angleThres = angleThres;

	cs3.cols = cols;
	cs3.rows = rows;

	cs3.gbuf = gbuf;
	cs3.gbuf_slac_triangle = gbuf_slac_triangle;
	cs3.gbuf_slac_block = gbuf_slac_block;
	cs3.gbuf_slac_b = gbuf_slac_b;

	combinedKernel3<<<grid, block>>>(cs3);
	cudaSafeCall ( hipGetLastError () );

	TranformReduction3 tr3;
	tr3.gbuf = gbuf;
	tr3.length = cols * rows;
	tr3.output = mbuf;

	TransformEstimatorKernel3<<<TranformReduction3::TOTAL, TranformReduction3::CTA_SIZE>>>(tr3);
	cudaSafeCall (hipGetLastError ());
	cudaSafeCall (hipDeviceSynchronize ());

	float_type host_data[TranformReduction3::TOTAL];
	mbuf.download (host_data);

	int shift = 0;
	for (int i = 0; i < 6; ++i) {		//rows
		for (int j = i; j < 7; ++j) {   // cols + b
			float_type value = host_data[shift++];
			if (j == 6)       // vector b
				vectorB_host[i] = value;
			else
				matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
		}
	}
}
