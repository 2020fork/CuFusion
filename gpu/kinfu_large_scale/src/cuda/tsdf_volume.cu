#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

using namespace pcl::device;

/*__global__ */__device__
const float COS30 = 0.8660254f
    ,COS45 = 0.7071f
    ,COS60 = 0.5f
    ,COS75 = 0.258819f
    ,COS80 = 0.173649f
    ;

namespace pcl
{
  namespace device
  {
    template<typename T>
    __global__ void
    initializeVolume (PtrStep<T> volume)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;
      
      
      if (x < VOLUME_X && y < VOLUME_Y)
      {
          T *pos = volume.ptr(y) + x;
          int z_step = VOLUME_Y * volume.step / sizeof(*pos);

#pragma unroll
          for(int z = 0; z < VOLUME_Z; ++z, pos+=z_step)
             pack_tsdf (0.f, 0, *pos);
      }
    }
    
    //zc: ���ģ�� T ��ʵ��Ҫ���� bool
    template<typename T>
    __global__ void
    initFlagVolumeKernel(PtrStep<T> volume){
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;
      
      
      if (x < VOLUME_X && y < VOLUME_Y)
      {
          T *pos = volume.ptr(y) + x;
          int z_step = VOLUME_Y * volume.step / sizeof(*pos);

#pragma unroll
          for(int z = 0; z < VOLUME_Z; ++z, pos+=z_step)
             //pack_tsdf (0.f, 0, *pos);
             *pos = false; //���Ĵ˴�?
      }
    }//initFlagVolumeKernel

    //zc: ���ģ�� T ��Ҫ���� char3, char4
    template<typename T>
    __global__ void
    initVrayPrevVolumeKrnl (PtrStep<T> volume)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;
      
      
      if (x < VOLUME_X && y < VOLUME_Y)
      {
          T *pos = volume.ptr(y) + x;
          int z_step = VOLUME_Y * volume.step / sizeof(*pos);

#pragma unroll
          for(int z = 0; z < VOLUME_Z; ++z, pos+=z_step){
              (*pos).x = 0;
              (*pos).y = 0;
              (*pos).z = 0;
              (*pos).w = 0; //T Ŀǰ��Ȼ���� char4 (��Ϊ host �а� int �洢), ���Է����� w �� //2017-2-15 16:53:43
                   //��- ������ xyz ���� tsdf-v8 ����; ���������� w, �������� bool flagVolume; �˴�Լ��: 0-false-Ϲ��, 1-true-����; Ĭ����Ϊ 0,
          }
      }
    }//initVrayPrevVolumeKrnl


        template<typename T>
    __global__ void
    clearSliceKernel (PtrStep<T> volume, pcl::gpu::tsdf_buffer buffer, int3 minBounds, int3 maxBounds)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;
           
      //compute relative indices
      int idX, idY;
      
      if(x < minBounds.x)
        idX = x + buffer.voxels_size.x;
      else
        idX = x;
      
      if(y < minBounds.y)
        idY = y + buffer.voxels_size.y;
      else
        idY = y;	 
              
      
      if ( x < buffer.voxels_size.x && y < buffer.voxels_size.y)
      {
          if( (idX >= minBounds.x && idX <= maxBounds.x) || (idY >= minBounds.y && idY <= maxBounds.y) )
          {
              // BLACK ZONE => clear on all Z values
         
              ///Pointer to the first x,y,0			
              T *pos = volume.ptr(y) + x;
              
              ///Get the step on Z
              int z_step = buffer.voxels_size.y * volume.step / sizeof(*pos);
                                  
              ///Get the size of the whole TSDF memory
              int size = buffer.tsdf_memory_end - buffer.tsdf_memory_start + 1;
                                
              ///Move along z axis
    #pragma unroll
              for(int z = 0; z < buffer.voxels_size.z; ++z, pos+=z_step)
              {
                ///If we went outside of the memory, make sure we go back to the begining of it
                if(pos > buffer.tsdf_memory_end)
                  pos = pos - size;
                  
                pack_tsdf (0.f, 0, *pos);
              }
           }
           else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
           {
             
              ///RED ZONE  => clear only appropriate Z
             
              ///Pointer to the first x,y,0
              T *pos = volume.ptr(y) + x;
              
              ///Get the step on Z
              int z_step = buffer.voxels_size.y * volume.step / sizeof(*pos);
                           
              ///Get the size of the whole TSDF memory 
              int size = buffer.tsdf_memory_end - buffer.tsdf_memory_start + 1;
                            
              ///Move pointer to the Z origin
              pos+= minBounds.z * z_step;
              
              ///If the Z offset is negative, we move the pointer back
              if(maxBounds.z < 0)
                pos += maxBounds.z * z_step;
                
              ///We make sure that we are not already before the start of the memory
              if(pos < buffer.tsdf_memory_start)
                  pos = pos + size;

              int nbSteps = abs(maxBounds.z);
              
          #pragma unroll				
              for(int z = 0; z < nbSteps; ++z, pos+=z_step)
              {
                ///If we went outside of the memory, make sure we go back to the begining of it
                if(pos > buffer.tsdf_memory_end)
                  pos = pos - size;
                  
                pack_tsdf (0.f, 0, *pos);
              }
           } //else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
       } // if ( x < VOLUME_X && y < VOLUME_Y)
    } // clearSliceKernel
       
  }
}

void
pcl::device::initVolume (PtrStep<short2> volume)
{
  dim3 block (32, 16);
  dim3 grid (1, 1, 1);
  grid.x = divUp (VOLUME_X, block.x);      
  grid.y = divUp (VOLUME_Y, block.y);

  initializeVolume<<<grid, block>>>(volume);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

void
pcl::device::initFlagVolume(PtrStep<bool> volume){
    dim3 block (16, 16);
    dim3 grid (1, 1, 1);
    grid.x = divUp (VOLUME_X, block.x);      
    grid.y = divUp (VOLUME_Y, block.y);

    //initializeVolume<<<grid, block>>>(volume);
    initFlagVolumeKernel<<<grid, block>>>(volume);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
}//initFlagVolume

void
pcl::device::initVrayPrevVolume(PtrStep<char4> volume){
    dim3 block (16, 16);
    dim3 grid (1, 1, 1);
    grid.x = divUp (VOLUME_X, block.x);      
    grid.y = divUp (VOLUME_Y, block.y);

    //initializeVolume<<<grid, block>>>(volume);
    //initFlagVolumeKernel<<<grid, block>>>(volume); //magCnt ���� initFlagVolumeKernel, ��Ϊ����ģ�庯��, ��ʼ�� false �� 0 һ��
    initVrayPrevVolumeKrnl<<<grid, block>>>(volume);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
}//initVrayPrevVolume

namespace pcl
{
  namespace device
  {
    struct Tsdf
    {
      enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8,
        //MAX_WEIGHT = 1 << 7
        MAX_WEIGHT = 1 << 9
        //MAX_WEIGHT = 15
        //MAX_WEIGHT = 255
        //MAX_WEIGHT = 15

        ,MAX_WEIGHT_V13 = 1<<8
      };

      mutable PtrStep<short2> volume;
      float3 cell_size;

      Intr intr;

      Mat33 Rcurr_inv;
      float3 tcurr;

      PtrStepSz<ushort> depth_raw; //depth in mm

      float tranc_dist_mm;

      __device__ __forceinline__ float3
      getVoxelGCoo (int x, int y, int z) const
      {
        float3 coo = make_float3 (x, y, z);
        coo += 0.5f;         //shift to cell center;

        coo.x *= cell_size.x;
        coo.y *= cell_size.y;
        coo.z *= cell_size.z;

        return coo;
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
          return;

        short2 *pos = volume.ptr (y) + x;
        int elem_step = volume.step * VOLUME_Y / sizeof(*pos);

        for (int z = 0; z < VOLUME_Z; ++z, pos += elem_step)
        {
          float3 v_g = getVoxelGCoo (x, y, z);            //3 // p

          //tranform to curr cam coo space
          float3 v = Rcurr_inv * (v_g - tcurr);           //4

          int2 coo;           //project to current cam
          coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
          coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);

          if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)           //6
          {
            int Dp = depth_raw.ptr (coo.y)[coo.x];

            if (Dp != 0)
            {
              float xl = (coo.x - intr.cx) / intr.fx;
              float yl = (coo.y - intr.cy) / intr.fy;
              float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

              float sdf = 1000 * norm (tcurr - v_g) * lambda_inv - Dp; //mm

              sdf *= (-1);

              if (sdf >= -tranc_dist_mm)
              {
                float tsdf = fmin (1, sdf / tranc_dist_mm);

                int weight_prev;
                float tsdf_prev;

                //read and unpack
                unpack_tsdf (*pos, tsdf_prev, weight_prev);

                const int Wrk = 1;

                float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                int weight_new = min (weight_prev + Wrk, MAX_WEIGHT);

                pack_tsdf (tsdf_new, weight_new, *pos);
              }
            }
          }
        }
      }
    };

    __global__ void
    integrateTsdfKernel (const Tsdf tsdf) {
      tsdf ();
    }

    __global__ void
    tsdf2 (PtrStep<short2> volume, const float tranc_dist_mm, const Mat33 Rcurr_inv, float3 tcurr,
           const Intr intr, const PtrStepSz<ushort> depth_raw, const float3 cell_size)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      short2 *pos = volume.ptr (y) + x;
      int elem_step = volume.step * VOLUME_Y / sizeof(short2);

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_x = Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z;
      float v_y = Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z;
      float v_z = Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z;

//#pragma unroll
      for (int z = 0; z < VOLUME_Z; ++z)
      {
        float3 vr;
        vr.x = v_g_x;
        vr.y = v_g_y;
        vr.z = (v_g_z + z * cell_size.z);

        float3 v;
        v.x = v_x + Rcurr_inv.data[0].z * z * cell_size.z;
        v.y = v_y + Rcurr_inv.data[1].z * z * cell_size.z;
        v.z = v_z + Rcurr_inv.data[2].z * z * cell_size.z;

        int2 coo;         //project to current cam
        coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
        coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);


        if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)         //6
        {
          int Dp = depth_raw.ptr (coo.y)[coo.x]; //mm

          if (Dp != 0)
          {
            float xl = (coo.x - intr.cx) / intr.fx;
            float yl = (coo.y - intr.cy) / intr.fy;
            float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

            float sdf = Dp - norm (vr) * lambda_inv * 1000; //mm


            if (sdf >= -tranc_dist_mm)
            {
              float tsdf = fmin (1.f, sdf / tranc_dist_mm);

              int weight_prev;
              float tsdf_prev;

              //read and unpack
              unpack_tsdf (*pos, tsdf_prev, weight_prev);

              const int Wrk = 1;

              float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
              int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

              pack_tsdf (tsdf_new, weight_new, *pos);
            }
          }
        }
        pos += elem_step;
      }       /* for(int z = 0; z < VOLUME_Z; ++z) */
    }      /* __global__ */
  }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth_raw, const Intr& intr, const float3& volume_size,
                                  const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, 
                                  PtrStep<short2> volume)
{
  Tsdf tsdf;

  tsdf.volume = volume;  
  tsdf.cell_size.x = volume_size.x / VOLUME_X;
  tsdf.cell_size.y = volume_size.y / VOLUME_Y;
  tsdf.cell_size.z = volume_size.z / VOLUME_Z;
  
  tsdf.intr = intr;

  tsdf.Rcurr_inv = Rcurr_inv;
  tsdf.tcurr = tcurr;
  tsdf.depth_raw = depth_raw;

  tsdf.tranc_dist_mm = tranc_dist*1000; //mm

  dim3 block (Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

#if 01
   //tsdf2<<<grid, block>>>(volume, tranc_dist, Rcurr_inv, tcurr, intr, depth_raw, tsdf.cell_size);
   integrateTsdfKernel<<<grid, block>>>(tsdf);
#endif
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}


namespace pcl
{
  namespace device
  {
    __global__ void
    scaleDepth (const PtrStepSz<ushort> depth, PtrStep<float> scaled, const Intr intr)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= depth.cols || y >= depth.rows)
        return;

      int Dp = depth.ptr (y)[x];

      float xl = (x - intr.cx) / intr.fx;
      float yl = (y - intr.cy) / intr.fy;
      float lambda = sqrtf (xl * xl + yl * yl + 1);

	  float res = Dp * lambda/1000.f; //meters
	  if ( intr.trunc_dist > 0 && res > intr.trunc_dist )
		  scaled.ptr (y)[x] = 0;
	  else
		scaled.ptr (y)[x] = res;
    }

    __global__ void
    tsdf23 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
            //const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size, const pcl::gpu::tsdf_buffer buffer)
            const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size, const pcl::gpu::tsdf_buffer buffer, int3 vxlDbg) //zc: ����
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= buffer.voxels_size.x || y >= buffer.voxels_size.y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      short2* pos = volume.ptr (y) + x;
      
      // shift the pointer to relative indices
      shift_tsdf_pointer(&pos, buffer);
      
      int elem_step = volume.step * buffer.voxels_size.y / sizeof(short2);

//#pragma unroll
      for (int z = 0; z < buffer.voxels_size.z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos += elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        // As the pointer is incremented in the for loop, we have to make sure that the pointer is never outside the memory
        if(pos > buffer.tsdf_memory_end)
          pos -= (buffer.tsdf_memory_end - buffer.tsdf_memory_start + 1);
        
        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
		// old code
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("@tsdf23: Dp_scaled, sdf, tranc_dist: %f, %f, %f, %s\n", Dp_scaled, sdf, tranc_dist, 
                  sdf >= -tranc_dist ? "sdf >= -tranc_dist" : "");
          }

          if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          {
            float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

            //read and unpack
            float tsdf_prev;
            int weight_prev;
            unpack_tsdf (*pos, tsdf_prev, weight_prev);
            //v17, Ϊ��� v17 �� w ������ĩλ�����λ, ���������޸�: unpack ʱ /2, pack ʱ *2; @2018-1-22 02:01:27
            weight_prev = weight_prev >> 1;

            const int Wrk = 1;

            float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
            int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

            if(doDbgPrint){
                printf("tsdf_prev, tsdf, tsdf_new: %f, %f, %f\n", tsdf_prev, tsdf, tsdf_new);
            }

            weight_new = weight_new << 1; //ʡ����+0, v17 �ı��λĬ��ֵ=0
            pack_tsdf (tsdf_new, weight_new, *pos);
          }
        }
        else{ //NOT (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)
            if(doDbgPrint){
                printf("vxlDbg.xyz:= (%d, %d, %d), coo.xy:= (%d, %d)\n", vxlDbg.x, vxlDbg.y, vxlDbg.z, coo.x, coo.y);
            }
        }

		/*
		// this time, we need an interpolation to get the depth value
		float2 coof = { v_x * inv_z + intr.cx, v_y * inv_z + intr.cy };
        int2 coo =
        {
          __float2int_rd (v_x * inv_z + intr.cx),
          __float2int_rd (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols - 1 && coo.y < depthScaled.rows - 1 )         //6
        {
          //float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters
		  float a = coof.x - coo.x;
		  float b = coof.y - coo.y;
		  float d00 = depthScaled.ptr (coo.y)[coo.x];
		  float d01 = depthScaled.ptr (coo.y+1)[coo.x];
		  float d10 = depthScaled.ptr (coo.y)[coo.x+1];
		  float d11 = depthScaled.ptr (coo.y+1)[coo.x+1];

          float Dp_scaled = 0;

		  if ( d00 != 0 && d01 != 0 && d10 != 0 && d11 != 0 && a > 0 && a < 1 && b > 0 && b < 1 )
		    Dp_scaled = ( 1 - b ) * ( ( 1 - a ) * d00 + ( a ) * d10 ) + ( b ) * ( ( 1 - a ) * d01 + ( a ) * d11 );

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          {
            float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

            //read and unpack
            float tsdf_prev;
            int weight_prev;
            unpack_tsdf (*pos, tsdf_prev, weight_prev);

            const int Wrk = 1;

            float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
            int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

            pack_tsdf (tsdf_new, weight_new, *pos);
          }		  
		}
		*/
      }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__ tsdf23

    __global__ void
    tsdf23_s2s (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
            const float tranc_dist, const float eta, //s2s (delta, eta)
            const Mat33 Rcurr_inv, const float3 tcurr, 
            const Intr intr, const float3 cell_size, int3 vxlDbg) //zc: ����
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      short2* pos = volume.ptr (y) + x;
      int elem_step = volume.step * VOLUME_Y / sizeof(short2);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos += elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        // old code
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("@tsdf23_s2s: Dp_scaled, sdf, tranc_dist: %f, %f, %f, %s; sdf/tdist: %f, coo.xy: (%d, %d)\n", Dp_scaled, sdf, tranc_dist, 
                  sdf >= -tranc_dist ? "sdf >= -tranc_dist" : "", sdf/tranc_dist, coo.x, coo.y);
          }

          //if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          if (Dp_scaled != 0 && sdf >= -eta) //meters //�Ƚ� eta , ���� delta (tdist)
          {
            float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

            if(sdf < -tranc_dist)
                tsdf = -1.0f;

#if 10   //�����ۼ�
            //read and unpack
            float tsdf_prev;
            int weight_prev;
            unpack_tsdf (*pos, tsdf_prev, weight_prev);
            //v17, Ϊ��� v17 �� w ������ĩλ�����λ, ���������޸�: unpack ʱ /2, pack ʱ *2; @2018-1-22 02:01:27
            weight_prev = weight_prev >> 1;

            const int Wrk = 1;

            float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
            int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

            if(doDbgPrint){
                printf("tsdf_prev, tsdf_curr, tsdf_new: %f, %f, %f; wp, wnew: %d, %d\n", tsdf_prev, tsdf, tsdf_new, weight_prev, weight_new);
            }
#elif 1 //ֱ�� set volume Ϊ��ǰ dmap ӳ����
            float tsdf_new = tsdf;
            int weight_new = 1;
#endif
            weight_new = weight_new << 1; //ʡ����+0, v17 �ı��λĬ��ֵ=0
            pack_tsdf (tsdf_new, weight_new, *pos);
          }
          else{ //(Dp_scaled == 0 || sdf < -eta)
            //float tsdf_new = 0;
            //int weight_new = 0;
            //pack_tsdf (tsdf_new, weight_new, *pos);
            if(doDbgPrint)
                printf("NOT (Dp_scaled != 0 && sdf >= -eta)\n");
          }
        }
        else{ //NOT (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)
            if(doDbgPrint){
                printf("vxlDbg.xyz:= (%d, %d, %d), coo.xy:= (%d, %d)\n", vxlDbg.x, vxlDbg.y, vxlDbg.z, coo.x, coo.y);
            }
        }
      }       // for(int z = 0; z < VOLUME_Z; ++z)
    }//__global__ tsdf23_s2s

    enum{FUSE_KF_AVGE, //kf tsdf ԭ����
        FUSE_RESET, //i ��� i-1
        FUSE_IGNORE_CURR //���� i
        ,FUSE_FIX_PREDICTION //�ȸ�����, �������
    };

    __global__ void
    tsdf23_v11 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;

      //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if(doDbgPrint)
            printf("inv_z:= %f\n", inv_z);

        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
              printf("coo.xy:(%d, %d)\n", coo.x, coo.y);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];
#if 0
          if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
#else
          //��--v11.7: �� wmap (weight) ��̬�趨 tranc_dist ����, (����׼����:
          //float tranc_dist_real = tranc_dist * weiFactor;
          float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8

          if(doDbgPrint){
              printf("\ttranc_dist_real, weiFactor: %f, %f\n", tranc_dist_real, weiFactor);
          }

          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters
          //if (Dp_scaled != 0 && -tranc_dist_real <= sdf && sdf < tranc_dist) //meters, v11.8
#endif
          {
            float sdf_normed = sdf * tranc_dist_inv;
            float tsdf_curr = fmin (1.0f, sdf_normed);

            bool isInclined = (incidAngleMask.ptr(coo.y)[coo.x] != 0); //̫��б��, �����̫��
            float3 snorm_curr_g;
            snorm_curr_g.x = nmap_curr_g.ptr(coo.y)[coo.x];
            if(isnan(snorm_curr_g.x)){
                if(doDbgPrint)
                    printf("+++++++++++++++isnan(snorm_curr_g.x), weiFactor: %f\n", weiFactor);

                return;
            }

            snorm_curr_g.y = nmap_curr_g.ptr(coo.y + depthScaled.rows)[coo.x];
            snorm_curr_g.z = nmap_curr_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

            float3 vrayPrev;
            //�����ѹ�һ��, ��Ȼ char->float �����, ����������һ��
            vrayPrev.x = 1.f * (*vrayPrev_pos).x / CHAR_MAX; //char2float
            vrayPrev.y = 1.f * (*vrayPrev_pos).y / CHAR_MAX;
            vrayPrev.z = 1.f * (*vrayPrev_pos).z / CHAR_MAX;

            //v11.3: �� vrayPrev_pos[3] �� hadSeenConfidence, ȡ�� hadSeen ������: //2017-3-11 21:40:24
            signed char *seenConfid = &vrayPrev_pos->w;
            const int seenConfidTh = 15;

            float3 vray; //��β�����������Ҫ�ж�, �˴�ֻ���������� nmap ���ζԴ�
                            //v11.2 �ĳɶ�Ҫ��: ���� & ���淨��˫���ж� //2017-3-8 22:00:32
            vray.x = v_g_x;
            vray.y = v_g_y;
            vray.z = v_g_z;
            //float vray_norm = norm(vray);
            float3 vray_normed = normalized(vray); //��λ��������

            float cos_vray_norm = dot(snorm_curr_g, vray_normed);
            if(cos_vray_norm > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                //���費��֤�ⲿ����ȷԤ����
                snorm_curr_g.x *= -1;
                snorm_curr_g.y *= -1;
                snorm_curr_g.z *= -1;
            }

            float3 snormPrev;
            snormPrev.x = 1.f * (*snorm_pos).x / CHAR_MAX; //char2float
            snormPrev.y = 1.f * (*snorm_pos).y / CHAR_MAX;
            snormPrev.z = 1.f * (*snorm_pos).z / CHAR_MAX;

            //v11.9: ��ʱ�� snorm ����������س�ʼ��, ��ʵֵȴ������ȥ���� snorm @2017-4-11 17:03:51
            int snormPrevConfid = (*snorm_pos).w;
            const int snormPrevConfid_thresh = 5;

            //const bool hadSeen = *flag_pos; //���� hadSeen, ��׼ȷ
            const bool hadSeen = (*seenConfid > seenConfidTh); //v11.3: ����, ������ confid++, �ﵽ��ֵ֮��, �ű�� seen; ���ﲻ����ֵ, ��Ҫ--

            //bool isSnormPrevInit = (norm(snormPrev) > 1e-8);
            //bool isSnormPrevInit = ( (norm(snormPrev) > 1e-8) && (snormPrevConfid > snormPrevConfid_thresh) );
            bool isSnormPrevInit = (snormPrevConfid > snormPrevConfid_thresh); //ȥ�� X>1e-8 �ж�, ��Ϊ confid > th ʱ��Ȼ X �Ѿ���ʼ������

            if(doDbgPrint){
                printf("isInclined, %d\n", isInclined);
                printf("cos_vray_norm, %f; snorm_curr_g: [%f, %f, %f], vray_normed: [%f, %f, %f]\n", cos_vray_norm, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z, vray_normed.x, vray_normed.y, vray_normed.z);
                printf("(norm(snormPrev) == 0) == %s; (norm(snormPrev) < 1e-8) == %s\n",
                    norm(snormPrev) == 0 ? "T" : "F",
                    norm(snormPrev) < 1e-8 ? "T" : "F");
            }


            //read and unpack
            float tsdf_prev1;
            int weight_prev1;
            unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);

            float tsdf_prev2nd = -123;
            int weight_prev2nd = -233;
            unpack_tsdf (*pos2nd, tsdf_prev2nd, weight_prev2nd);

            //const int w2ndCntThresh = 10; //w2nd ��������ֵ������Ϯ
            const int w2ndCntThresh = 10 * 10; //v11.4 �� weiFactor ֮��

            if(doDbgPrint){
                printf("tsdf_prev: tsdf1st: %f, %d; tsdf2nd: %f, %d;\n", tsdf_prev1, weight_prev1, tsdf_prev2nd, weight_prev2nd);
            }

            int fuse_method = FUSE_KF_AVGE; //Ĭ��ԭ����
            bool doUpdateVrayAndSnorm = false;

            const float cosThreshVray = //0.8660254f; //cos(30��)
                //0.9396926f; //cos(20��) //�� largeIncidMask ȡ 80 ��ֵʱ, �˴�ӦΪ (90-x)*2
                0.9659258f; //cos(15��) //��ΪlargeIncidMask �� 75��Ϊ��ֵ, ���������� 90-75=15 Ϊ��ֵ
                //0.996194698; //cos(5��)
            const float cosThreshSnorm = 0.8660254f; //cos(30��), �� vray ���ֿ�, ���ø�������ֵ @2017-3-15 00:39:18

            float cos_norm = dot(snormPrev, snorm_curr_g);
            float cos_vray = dot(vrayPrev, vray_normed);
            bool isNewFace = (isSnormPrevInit && cos_norm < cosThreshSnorm && cos_vray < cosThreshVray); //snorm-init ֮����� newFace �ж� @2017-4-21 00:42:00
            //bool isNewFace = (isSnormPrevInit && cos_norm < cosThreshSnorm); //ȥ�� vray �ж�, ��! ԭ��: vray ��ֹ *�ӽ��ȶ���snorm ͻ�� (��Եetc.)* ����, ������ isNewFace=true

            if(doDbgPrint){
                printf("cos_norm: snormPrev, snorm_curr_g, %f, [%f, %f, %f], [%f, %f, %f]\n", cos_norm, 
                    snormPrev.x, snormPrev.y, snormPrev.z, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);
                printf("\tcos_vray, vrayPrev, vray_normed, %f, [%f, %f, %f], [%f, %f, %f]\n", cos_vray, 
                    vrayPrev.x, vrayPrev.y, vrayPrev.z, vray_normed.x, vray_normed.y, vray_normed.z);
                printf("%s, snormPrevConfid, snormPrevConfid_thresh: %d, %d\n", isNewFace ? "isNewFace-T" : "isNewFace-F", snormPrevConfid, snormPrevConfid_thresh);
                printf("\t%s\n", cos_norm > cosThreshSnorm ? "cos_norm > cosThreshSnorm" : "cos_norm <= cosThreshSnorm");
                printf("\t%s\n", cos_vray > cosThreshVray ? "cos_vray > cosThreshVray" : "cos_vray <= cosThreshVray");
            }


#if 01   //v11.3, v11.4, 
            if(isInclined){ //����Ե, doUpdateVray ���� false
                if(!hadSeen){ //�� seen-flag δ��ʼ����
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=F; ++FUSE_KF_AVGE\n");
                    fuse_method = FUSE_KF_AVGE;

                    //*seenConfid = max(0, *seenConfid - 1);
                    //��-- ��Ҫ -1 ��, ֻ������, ��ͬʱ seenConfidTh ��ֵ���� (5 -> 15), �ӻ��� flag=true   @2017-3-23 11:11:55
                }
                else{ //if(hadSeen) //��֮ǰ seen
#if 0   //���� sdf < 0 ����ж�Ϊʲô��, Ŀǰ�о��ᵼ����ƫ��, ����   @2017-3-9 15:06:22
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=T; %s; sdf: %f\n", sdf<0 ? "==FUSE_IGNORE_CURR" : "++FUSE_KF_AVGE", sdf);
                    if(sdf < 0)
                        fuse_method = FUSE_IGNORE_CURR;
                    else
                        fuse_method = FUSE_KF_AVGE;
#elif 1 //һ�� ignore
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=T; \n");
                    fuse_method = FUSE_IGNORE_CURR;
#endif
                }
            }
            else{ //if(!isInclined){ //���Ǳ�Ե, ���ڲ�
                //*seenConfid = min(Tsdf::MAX_WEIGHT, *seenConfid + 1); //v11.4 �� weiFactor ֮��, ���ﷴ���� BUG!!
                *seenConfid = min(SCHAR_MAX, *seenConfid + 1);

                if(!isSnormPrevInit){ //vray.prev ��δ��ʼ��, �� < epsilon �ж�
                    //if (*seenConfid > seenConfidTh) //����� hadSeen, ���Բ�Ҫ��ô�ж�
                        //doUpdateVrayAndSnorm = true;
                }


                if(!hadSeen){ //�� seen-flag δ��ʼ����
#if 0   //< v11.3
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=F; >>FUSE_RESET\n");
                    *flag_pos = true;
                    fuse_method = FUSE_RESET;
#elif 1 //v11.3
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=F; seenConfid, seenConfidTh: %d, %d, ++FUSE_KF_AVGE~~~~~\n", *seenConfid, seenConfidTh); //��Ҳû�� reset ��
                    fuse_method = FUSE_KF_AVGE;
#endif
                    //if (*seenConfid > seenConfidTh) //��Ȼ hadSeen �߼��Ĺ�, ��˴���Ȼһֱ false
                    //    doUpdateVrayAndSnorm = true;
                }
                else{ //if(hadSeen) //��֮ǰ seen, ��Ȼ������ ��isInclined-F; hadSeen=F���׶�, Ҳ��Ȼ isSnormPrevInit->true, ������ if-isSnormPrevInit
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=T;\n");

                    //if(cos_norm > cosThresh ){ //�нǽǶ� <30��, ����ͬ�ӽ�
                    if(!isNewFace){ //ͬ�ӽ�, ˫ cos �����ж�
                        //TODO...
                        fuse_method = FUSE_KF_AVGE; //��ʵĬ��

                        //if (*seenConfid > seenConfidTh) //����, ��Ϊ���� if-hadSeen ��֧��
                        if(cos_norm > cosThreshSnorm) //��֮ cos_norm < th ʱ, ���� newFace=false, ����Ӧ update
                            doUpdateVrayAndSnorm = true;

                        if(!isSnormPrevInit)
                            doUpdateVrayAndSnorm = true;
                    }
                    else{ // >30��, ������ͬ�ӽ�, ����ת��ͷ֮��
                        //if(!isSnormPrevInit) //newFace �Ľ�֮��, ���ﲻ���ٽ���
                        //    doUpdateVrayAndSnorm = true;

#if 10   //���಻����, �������岻��ȷ, ���� @2017-3-24 17:50:24
                        //����Ϊ����
                        if(tsdf_curr < 0 && tsdf_curr < tsdf_prev1){
                            if(doDbgPrint)
                                printf("\ttsdf < 0 && tsdf < tsdf_prev1; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);

                            fuse_method = FUSE_IGNORE_CURR;
                        }
                        else if(tsdf_prev1 < 0 && tsdf_prev1 < tsdf_curr){
                            if(doDbgPrint){
                                printf("\ttsdf_prev1 < 0 && tsdf_prev1 < tsdf; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                                printf("\t\t, weight_prev1, w2ndCntThresh: %d, %d\n", weight_prev1, w2ndCntThresh);
                            }
#if 0   //weight_prev1 �Ƿ�Ҫ�� w2ndCntThresh �Ա�?
                            if(weight_prev1 > w2ndCntThresh){
                                fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
                            }
                            else{
                                fuse_method = FUSE_KF_AVGE; //����Ĭ���Ƿ�������
                            }
#elif 1 //1st ���� w2ndCntThresh �Ա�, ��Ϊ�������Աȿ���: weight_new2nd > w2ndCntThresh
                            fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
#endif
                            //doUpdateSnorm = true; //�ŵ� FUSE_FIX_PREDICTION ���ж�
                        }
                        else if(tsdf_curr >=0 && tsdf_prev1 >= 0){
                            if(doDbgPrint){
                                printf("\ttsdf >=0 && tsdf_prev1 >= 0; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                            }
                            fuse_method = FUSE_KF_AVGE;
                            doUpdateVrayAndSnorm = true;
                        }
#elif 1 //ϸ�֡�����Ϊ7��, @2017-3-24 17:51:03
                        if(tsdf_prev1 >= 0){
                            if(tsdf_curr <0){
                                fuse_method = FUSE_IGNORE_CURR;
                                doUpdateVrayAndSnorm = true;
                                
                                if(doDbgPrint)
                                    printf("+~~-,, ==FUSE_IGNORE_CURR\n");
                            }
                            else{//tsdf_curr >=0
                                if(sdf < tranc_dist){
                                    fuse_method = FUSE_KF_AVGE;

                                    if(doDbgPrint)
                                        printf("+~~��+,, ++FUSE_KF_AVGE\n");
                                }
                                else{
                                    fuse_method = FUSE_IGNORE_CURR;

                                    if(doDbgPrint)
                                        printf("+~~��+,, ==FUSE_IGNORE_CURR\n");
                                }
                            }
                        }
                        else{ //tsdf_prev1 <0
                            float abs_tsdfcurr = abs(tsdf_curr);
                            if(abs_tsdfcurr < abs(tsdf_prev1)){
                                fuse_method = FUSE_FIX_PREDICTION;

                                if(doDbgPrint){
                                    if(tsdf_curr < 0)
                                        printf("-~~��-,, >>FUSE_FIX_PREDICTION\n");
                                    else
                                        printf("-~~��+,, >>FUSE_FIX_PREDICTION\n");
                                }
                            }
                            else{
                                fuse_method = FUSE_IGNORE_CURR;

                                if(doDbgPrint){
                                    if(tsdf_curr < 0)
                                        printf("-~~��-,, ==FUSE_IGNORE_CURR\n");
                                    else
                                        printf("-~~��+,, ==FUSE_IGNORE_CURR\n");
                                }
                            }
                        }
#endif
                    }//cos vs. cosTh
                }//if-hadSeen
            }//if-isInclined
#elif 0 //v11.5; //������, ������˼·������... @2017-3-16 00:05:51
            if(isInclined){
                if(doDbgPrint)
                    printf("isInclined-T; ++FUSE_KF_AVGE\n");

                fuse_method = FUSE_KF_AVGE;
                doUpdateVrayAndSnorm = false;
            }
            else{ //if(!isInclined){ //���Ǳ�Ե, ���ڲ�
                if(doDbgPrint)
                    printf("isInclined-F;\n");

                bool isSnormPrevInit = (norm(snormPrev) > 1e-8);
                if(!isSnormPrevInit){ //vray.prev ��δ��ʼ��, �� < epsilon �ж�
                    if(doDbgPrint)
                        printf("\tisSnormPrevInit-F\n");

                    fuse_method = FUSE_KF_AVGE;
                    doUpdateVrayAndSnorm = true;
                }
                else{ //vray+snorm ����ʼ������
                    if(!isNewFace){ //ͬ�ӽ�, ˫ cos �����ж�
                        if(doDbgPrint)
                            printf("\tisNewFace-F\n");

                        fuse_method = FUSE_KF_AVGE; //��ʵĬ��
                        doUpdateVrayAndSnorm = true;
                    }
                    else{ // isNewFace, ������ͬ�ӽ�, ����ת��ͷ֮��
                        if(doDbgPrint)
                            printf("\tisNewFace-T\n");

                        //����Ϊ����
                        if(tsdf_curr < 0 && tsdf_curr < tsdf_prev1){
                            if(doDbgPrint)
                                printf("\ttsdf < 0 && tsdf < tsdf_prev1; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);

                            fuse_method = FUSE_IGNORE_CURR;
                        }
                        else if(tsdf_prev1 < 0 && tsdf_prev1 < tsdf_curr){
                            if(doDbgPrint){
                                printf("\ttsdf_prev1 < 0 && tsdf_prev1 < tsdf; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                                printf("\t\t, weight_prev1, w2ndCntThresh: %d, %d\n", weight_prev1, w2ndCntThresh);
                            }
#if 0   //weight_prev1 �Ƿ�Ҫ�� w2ndCntThresh �Ա�?
                            if(weight_prev1 > w2ndCntThresh){
                                fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
                            }
                            else{
                                fuse_method = FUSE_KF_AVGE; //����Ĭ���Ƿ�������
                            }
#elif 1 //1st ���� w2ndCntThresh �Ա�, ��Ϊ�������Աȿ���: weight_new2nd > w2ndCntThresh
                            fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
#endif
                            //doUpdateSnorm = true; //�ŵ� FUSE_FIX_PREDICTION ���ж�
                        }
                        else if(tsdf_curr >=0 && tsdf_prev1 >= 0){
                            if(doDbgPrint){
                                printf("\ttsdf >=0 && tsdf_prev1 >= 0; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                            }
                            fuse_method = FUSE_KF_AVGE;
                            doUpdateVrayAndSnorm = true;
                        }
                    }//isNewFace
                }//vray+snorm ����ʼ������
            }
#elif 1 //v11.6: v11.5������, �ĳ� isInclined ֻ���ڿ��� vray+snorm �ĸ���; ȥ���� hadSeen-flag ����
            //�����Ǽ���, ���ǽ�������, �����Թ�
            bool isSnormPrevInit = (norm(snormPrev) > 1e-8);

            if(isInclined){
                doUpdateVrayAndSnorm = false;
            }
            else if(!isSnormPrevInit){
                doUpdateVrayAndSnorm = true;
            }

            if(!isSnormPrevInit){
                fuse_method = FUSE_KF_AVGE;
            }
            else{ //vray+snorm ����ʼ������
                if(!isNewFace){ //ͬ�ӽ�, ˫ cos �����ж�
                    if(doDbgPrint)
                        printf("\tisNewFace-F\n");

                    fuse_method = FUSE_KF_AVGE; //��ʵĬ��

                    if(!isInclined)
                        doUpdateVrayAndSnorm = true;
                }
                else{ // isNewFace, ������ͬ�ӽ�, ����ת��ͷ֮��
                    if(doDbgPrint)
                        printf("\tisNewFace-T\n");

                    //����Ϊ����
                    if(tsdf_curr < 0 && tsdf_curr < tsdf_prev1){
                        if(doDbgPrint)
                            printf("\ttsdf < 0 && tsdf < tsdf_prev1; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);

                        fuse_method = FUSE_IGNORE_CURR;
                    }
                    else if(tsdf_prev1 < 0 && tsdf_prev1 < tsdf_curr){
                        if(doDbgPrint){
                            printf("\ttsdf_prev1 < 0 && tsdf_prev1 < tsdf; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                            printf("\t\t, weight_prev1, w2ndCntThresh: %d, %d\n", weight_prev1, w2ndCntThresh);
                        }
                        fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
                        //doUpdateSnorm = true; //�ŵ� FUSE_FIX_PREDICTION ���ж�
                    }
                    else if(tsdf_curr >=0 && tsdf_prev1 >= 0){
                        if(doDbgPrint){
                            printf("\ttsdf >=0 && tsdf_prev1 >= 0; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                        }
                        fuse_method = FUSE_KF_AVGE;

                        if(!isInclined)
                            doUpdateVrayAndSnorm = true;
                    }
                }//isNewFace
            }//vray+snorm ����ʼ������
#endif
            const int Wrk = max(15 * weiFactor, 1.f);
            if(FUSE_KF_AVGE == fuse_method){
                float tsdf_new1 = (tsdf_prev1 * weight_prev1 + Wrk * tsdf_curr) / (weight_prev1 + Wrk);
                int weight_new1 = min (weight_prev1 + Wrk, Tsdf::MAX_WEIGHT);

                int weight_new2nd = max(weight_prev2nd - Wrk, 0); //--, ����ֹ <0

                pack_tsdf (tsdf_new1, weight_new1, *pos1);
                pack_tsdf(tsdf_prev2nd, weight_new2nd, *pos2nd); //���� 2nd �Ƿ�������ʼ����

                if(doDbgPrint)
                    printf("++FUSE_KF_AVGE, weight_new1, weight_new2nd, %d, %d\n", weight_new1, weight_new2nd);
            }
            else if(FUSE_FIX_PREDICTION == fuse_method){ //ȡ���ֱ� FUSE_RESET
#if 0   //factor/step ��ʽ����
//                   //const int pos_neg_factor = 8;
//                   int pos_neg_factor = min(weight_prev1 / 10, 1); //������ʱ���� w1 ��ʵ����, ���Բ��ֱܴ��趨�󲽳�
//                   int pnWrk = pos_neg_factor * Wrk;
//                   float tsdf_new2nd = (tsdf_prev2nd * weight_prev2nd + pnWrk * tsdf) / (weight_prev2nd + pnWrk);
//                   int weight_new2nd = min (weight_prev2nd + pnWrk, Tsdf::MAX_WEIGHT);
// 
//                   int weight_new1 = max(weight_prev1 - pnWrk, 0);
// 
//                   if(weight_new2nd > weight_new1){ //�� 2nd ��Ϯ, �򽻻� 1st/2nd, ��Զ���� 1st Ϊ��
#elif 1
                float tsdf_new2nd = (tsdf_prev2nd * weight_prev2nd + Wrk * tsdf_curr) / (weight_prev2nd + Wrk);
                int weight_new2nd = min (weight_prev2nd + Wrk, Tsdf::MAX_WEIGHT);

                //int weight_new1 = max(weight_prev1 - Wrk, 0);
                if(weight_new2nd > w2ndCntThresh){ //���� 1st/2nd, ��Զ���� 1st Ϊ�� //����ĳ�: 2nd ������Ϯ 1st, ֻҪ����ĳ������ֵ����
#endif
                    if(doDbgPrint){
                        printf("weight_new2nd > w2ndCntThresh,,, exchanging 1st-2nd\n");
                    }
                    pack_tsdf(tsdf_new2nd, weight_new2nd, *pos1); //new-2nd �ŵ� pos-1st ��
                    //pack_tsdf(tsdf_prev1, weight_new1, *pos2nd);

                    doUpdateVrayAndSnorm = true; //ֱ�� 2nd ��Ϯ, �����µ� snorm ���µ�ǰ vxl
                }
                else{ //����
                    //pack_tsdf(tsdf_prev1, weight_new1, *pos1);
                    pack_tsdf(tsdf_new2nd, weight_new2nd, *pos2nd);
                    doUpdateVrayAndSnorm = false;
                }

                if(doDbgPrint)
                    //printf("...>>FUSE_FIX_PREDICTION, weight_new1, weight_new2nd, %d, %d\n", weight_new1, weight_new2nd);
                    printf("...>>FUSE_FIX_PREDICTION, weight_new2nd, %d\n", weight_new2nd);

                //����: ����doDbgPrint, ȫ�����, ��������û���ߵ���һ���� vxl: @2017-3-11 21:22:59
                //��: ��!! ��Ϊ FUSE_FIX_PREDICTION Ŀǰ��� case: tsdf_prev1 < 0 && tsdf_prev1 < tsdf
                //printf("...>>FUSE_FIX_PREDICTION, weight_new2nd, %d,,, [xyz]=(%d, %d, %d)\n", weight_new2nd, x, y, z);
            }
            else if(FUSE_RESET == fuse_method){
                if(doDbgPrint)
                    printf(">>FUSE_RESET\n");

                pack_tsdf(tsdf_curr, 1, *pos1);
            }
            else if(FUSE_IGNORE_CURR == fuse_method){
                if(doDbgPrint)
                    printf("==FUSE_IGNORE_CURR\n");

                //DO-NOTHING!!! //��
                //IGNʱ, ҲҪ 2nd Ūһ�� @2017-3-16 03:53:08
                int weight_new2nd = max(weight_prev2nd - Wrk, 0); //--, ����ֹ <0
                pack_tsdf(tsdf_prev2nd, weight_new2nd, *pos2nd); //���� 2nd �Ƿ�������ʼ����
            }

            if(doDbgPrint)
                printf("doUpdateSnorm: %d\n", doUpdateVrayAndSnorm);

            if(doUpdateVrayAndSnorm){
                //max (-DIVISOR, min (DIVISOR, (int)nearbyintf (tsdf * DIVISOR))); //@pack_tsdf
                //��Ϊ vray_normed.xyz ��Ȼ�� <=1, ���Բ��� max/min... ��
                (*vrayPrev_pos).x = (int)nearbyintf(vray_normed.x * CHAR_MAX); //float2char
                (*vrayPrev_pos).y = (int)nearbyintf(vray_normed.y * CHAR_MAX);
                (*vrayPrev_pos).z = (int)nearbyintf(vray_normed.z * CHAR_MAX);

                //���� pcc �� nmap ����֮��, ��Ե����׼ (��Ϊ sobel?), Ҫ�е�; ������һЩ���� @2017-3-15 16:54:25
                //�� 4:=7/2+1
                const int edgeMarg = 4;
                if(coo.x < edgeMarg || coo.x >= depthScaled.cols - edgeMarg || coo.y < edgeMarg || coo.y >= depthScaled.rows - edgeMarg){
                    if(doDbgPrint)
                        printf("+++++++++++++++at edge, dont-update-snorm; coo.xy: (%d, %d)\n", coo.x, coo.y);
                }
                else{
                    //(*snorm_pos).w += 1; //�� snormPrevConfid
                    (*snorm_pos).w = min(SCHAR_MAX, snormPrevConfid + 1);

                    if(!isSnormPrevInit || isNewFace){
                        if(doDbgPrint)
                            printf("\t(!isSnormPrevInit || isNewFace): %d, %d; snormPrevConfid: %d\n", isSnormPrevInit, isNewFace, (*snorm_pos).w);

                        (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                        (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                        (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
                    }
                    else{ //isSnormPrevInit && !isNewFace //v11.6: ��snorm ��ʼ������, �ҵ�ǰû��ͻ��, ���� model �ķ���, ��Ϊ����ȶ�
                        if(doDbgPrint)
                            printf("\tisSnormPrevInit && !isNewFace\n");

                        float3 snorm_model_g;
                        snorm_model_g.x = nmap_model_g.ptr(coo.y)[coo.x];
                        snorm_model_g.y = nmap_model_g.ptr(coo.y + depthScaled.rows)[coo.x];
                        snorm_model_g.z = nmap_model_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

                        float cos_vray_norm_model = dot(snorm_model_g, vray_normed);
                        if(cos_vray_norm_model > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                            //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                            //���費��֤�ⲿ����ȷԤ����
                            snorm_model_g.x *= -1;
                            snorm_model_g.y *= -1;
                            snorm_model_g.z *= -1;
                        }
                        if(doDbgPrint)
                            printf("\t\tcos_vray_norm_model, %f; snorm_model_g: [%f, %f, %f], vray_normed: [%f, %f, %f]\n", cos_vray_norm_model, snorm_model_g.x, snorm_model_g.y, snorm_model_g.z, vray_normed.x, vray_normed.y, vray_normed.z);

                        float cos_norm_model_and_prev = dot(snorm_model_g, snormPrev);
                        //��--����˵, ��ʱ n_model, n_curr Ӧ�üнǺ�С (�Ѿ��������������� ��1 �˷�) //v11.7   @2017-3-17 15:52:25
                        //������Ϊ����, ���� n_model ƫ�����, ����ȫ������:
                        //if(cos_norm_model_and_prev > cosThreshSnorm){
                        //if(1){ //���� snormPrev ����

                        //zc: ���߼�: �� snorm-model/curr ����Ÿ��� @2017-4-25 21:24:23
                        float cos_norm_model_and_curr = dot(snorm_model_g, snorm_curr_g);
                        if(cos_norm_model_and_curr > cosThreshSnorm){
                            //���� __float2int_rd �� round-down �����˻�����, ��ֵ���ȶ�, ���� nearbyintf (������)?  @2017-3-15 15:33:33
                            (*snorm_pos).x = (int)nearbyintf(snorm_model_g.x * CHAR_MAX); //float2char
                            (*snorm_pos).y = (int)nearbyintf(snorm_model_g.y * CHAR_MAX);
                            (*snorm_pos).z = (int)nearbyintf(snorm_model_g.z * CHAR_MAX);
                        }
                        else{
                            //DO-NOTHING!!!
                        }
                    }
                }//cut-edgeMarg

                if(doDbgPrint){
                    printf("newVray: [%d, %d, %d]\n", (*vrayPrev_pos).x, (*vrayPrev_pos).y, (*vrayPrev_pos).z);
                    printf("\tnewSnorm: [%d, %d, %d]\n", (*snorm_pos).x, (*snorm_pos).y, (*snorm_pos).z);
                }
            }//if-(doUpdateVrayAndSnorm)
          }//if- (Dp_scaled != 0 && sdf >= -tranc_dist)
          else{
              if(doDbgPrint)
                  printf("NOT (Dp_scaled != 0 && sdf >= -tranc_dist)\n");
          }
        }//if- 0 < (x,y) < (cols,rows)
      }// for(int z = 0; z < VOLUME_Z; ++z)
    }//tsdf23_v11

    __global__ void
    tsdf23_v11_remake (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;

      //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
              printf("coo.xy:(%d, %d)\n", coo.x, coo.y);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];
#if 0
          if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
#else
          //��--v11.7: �� wmap (weight) ��̬�趨 tranc_dist ����, (����׼����:
          //float tranc_dist_real = tranc_dist * weiFactor;
          float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8

          if(doDbgPrint){
              printf("\ttranc_dist_real, weiFactor: %f, %f\n", tranc_dist_real, weiFactor);
          }

          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters
          //if (Dp_scaled != 0 && -tranc_dist_real <= sdf && sdf < tranc_dist) //meters, v11.8
#endif
          {
            float sdf_normed = sdf * tranc_dist_inv;
            float tsdf_curr = fmin (1.0f, sdf_normed);

            bool isInclined = (incidAngleMask.ptr(coo.y)[coo.x] != 0); //̫��б��, �����̫��
            float3 snorm_curr_g;
            snorm_curr_g.x = nmap_curr_g.ptr(coo.y)[coo.x];
            if(isnan(snorm_curr_g.x)){
                if(doDbgPrint)
                    printf("+++++++++++++++isnan(snorm_curr_g.x), weiFactor: %f\n", weiFactor);

                return;
            }

            snorm_curr_g.y = nmap_curr_g.ptr(coo.y + depthScaled.rows)[coo.x];
            snorm_curr_g.z = nmap_curr_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

            float3 vrayPrev;
            //�����ѹ�һ��, ��Ȼ char->float �����, ����������һ��
            vrayPrev.x = 1.f * (*vrayPrev_pos).x / CHAR_MAX; //char2float
            vrayPrev.y = 1.f * (*vrayPrev_pos).y / CHAR_MAX;
            vrayPrev.z = 1.f * (*vrayPrev_pos).z / CHAR_MAX;

            //v11.3: �� vrayPrev_pos[3] �� hadSeenConfidence, ȡ�� hadSeen ������: //2017-3-11 21:40:24
            signed char *seenConfid = &vrayPrev_pos->w;
            const int seenConfidTh = 15;

            float3 vray; //��β�����������Ҫ�ж�, �˴�ֻ���������� nmap ���ζԴ�
                            //v11.2 �ĳɶ�Ҫ��: ���� & ���淨��˫���ж� //2017-3-8 22:00:32
            vray.x = v_g_x;
            vray.y = v_g_y;
            vray.z = v_g_z;
            //float vray_norm = norm(vray);
            float3 vray_normed = normalized(vray); //��λ��������

            float cos_vray_norm = dot(snorm_curr_g, vray_normed);
            if(cos_vray_norm > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                //���費��֤�ⲿ����ȷԤ����
                snorm_curr_g.x *= -1;
                snorm_curr_g.y *= -1;
                snorm_curr_g.z *= -1;
            }

            float3 snormPrev;
            snormPrev.x = 1.f * (*snorm_pos).x / CHAR_MAX; //char2float
            snormPrev.y = 1.f * (*snorm_pos).y / CHAR_MAX;
            snormPrev.z = 1.f * (*snorm_pos).z / CHAR_MAX;

            //v11.9: ��ʱ�� snorm ����������س�ʼ��, ��ʵֵȴ������ȥ���� snorm @2017-4-11 17:03:51
            signed char *snormPrevConfid = &snorm_pos->w;
            const int snormPrevConfid_thresh = 5;

            //const bool hadSeen = *flag_pos; //���� hadSeen, ��׼ȷ
            const bool hadSeen = (*seenConfid > seenConfidTh); //v11.3: ����, ������ confid++, �ﵽ��ֵ֮��, �ű�� seen; ���ﲻ����ֵ, ��Ҫ--

            //bool isSnormPrevInit = (norm(snormPrev) > 1e-8);
            //bool isSnormPrevInit = ( (norm(snormPrev) > 1e-8) && (snormPrevConfid > snormPrevConfid_thresh) );
            bool isSnormPrevInit = (*snormPrevConfid > snormPrevConfid_thresh); //ȥ�� X>1e-8 �ж�, ��Ϊ confid > th ʱ��Ȼ X �Ѿ���ʼ������

            if(doDbgPrint){
                printf("isInclined, %d\n", isInclined);
                printf("cos_vray_norm, %f; snorm_curr_g: [%f, %f, %f], vray_normed: [%f, %f, %f]\n", cos_vray_norm, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z, vray_normed.x, vray_normed.y, vray_normed.z);
                printf("(norm(snormPrev) == 0) == %s; (norm(snormPrev) < 1e-8) == %s\n",
                    norm(snormPrev) == 0 ? "T" : "F",
                    norm(snormPrev) < 1e-8 ? "T" : "F");
            }


            //read and unpack
            float tsdf_prev1;
            int weight_prev1;
            unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);

            float tsdf_prev2nd = -123;
            int weight_prev2nd = -233;
            unpack_tsdf (*pos2nd, tsdf_prev2nd, weight_prev2nd);

            //const int w2ndCntThresh = 10; //w2nd ��������ֵ������Ϯ
            const int w2ndCntThresh = 10 * 10; //v11.4 �� weiFactor ֮��

            if(doDbgPrint){
                printf("tsdf_prev: tsdf1st: %f, %d; tsdf2nd: %f, %d;\n", tsdf_prev1, weight_prev1, tsdf_prev2nd, weight_prev2nd);
            }

            int fuse_method = FUSE_KF_AVGE; //Ĭ��ԭ����
            bool doUpdateVrayAndSnorm = false;

            const float cosThreshVray = //0.8660254f; //cos(30��)
                //0.9396926f; //cos(20��) //�� largeIncidMask ȡ 80 ��ֵʱ, �˴�ӦΪ (90-x)*2
                0.9659258f; //cos(15��) //��ΪlargeIncidMask �� 75��Ϊ��ֵ, ���������� 90-75=15 Ϊ��ֵ
                //0.996194698; //cos(5��)
            const float cosThreshSnorm = 0.8660254f; //cos(30��), �� vray ���ֿ�, ���ø�������ֵ @2017-3-15 00:39:18

            float cos_norm = dot(snormPrev, snorm_curr_g);
            float cos_vray = dot(vrayPrev, vray_normed);
            bool isNewFace = (isSnormPrevInit && cos_norm < cosThreshSnorm && cos_vray < cosThreshVray); //snorm-init ֮����� newFace �ж� @2017-4-21 00:42:00
            //bool isNewFace = (isSnormPrevInit && cos_norm < cosThreshSnorm); //ȥ�� vray �ж�, ��! ԭ��: vray ��ֹ *�ӽ��ȶ���snorm ͻ�� (��Եetc.)* ����, ������ isNewFace=true

            //zc: �����ж�, �� weight-factor ̫С(��, ��Ե����), ��ֱ�Ӿ�ֵ, �Ҳ� updateVray @2017-7-13 22:29:39
            if(weiFactor > 0.2){


            if(doDbgPrint){
                printf("cos_norm: snormPrev, snorm_curr_g, %f, [%f, %f, %f], [%f, %f, %f]\n", cos_norm, 
                    snormPrev.x, snormPrev.y, snormPrev.z, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);
                printf("\tcos_vray, vrayPrev, vray_normed, %f, [%f, %f, %f], [%f, %f, %f]\n", cos_vray, 
                    vrayPrev.x, vrayPrev.y, vrayPrev.z, vray_normed.x, vray_normed.y, vray_normed.z);
                printf("%s, snormPrevConfid, snormPrevConfid_thresh: %d, %d\n", isNewFace ? "isNewFace-T" : "isNewFace-F", *snormPrevConfid, snormPrevConfid_thresh);
                printf("\t%s\n", cos_norm > cosThreshSnorm ? "cos_norm > cosThreshSnorm" : "cos_norm <= cosThreshSnorm");
                printf("\t%s\n", cos_vray > cosThreshVray ? "cos_vray > cosThreshVray" : "cos_vray <= cosThreshVray");
            }


            if(isInclined){ //����Ե, doUpdateVray ���� false
                if(!hadSeen){ //�� seen-flag δ��ʼ����
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=F; ++FUSE_KF_AVGE\n");
                    fuse_method = FUSE_KF_AVGE;

                    //*seenConfid = max(0, *seenConfid - 1);
                    //��-- ��Ҫ -1 ��, ֻ������, ��ͬʱ seenConfidTh ��ֵ���� (5 -> 15), �ӻ��� flag=true   @2017-3-23 11:11:55
                }
                else{ //if(hadSeen) //��֮ǰ seen
#if 0   //���� sdf < 0 ����ж�Ϊʲô��, Ŀǰ�о��ᵼ����ƫ��, ����   @2017-3-9 15:06:22
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=T; %s; sdf: %f\n", sdf<0 ? "==FUSE_IGNORE_CURR" : "++FUSE_KF_AVGE", sdf);
                    if(sdf < 0)
                        fuse_method = FUSE_IGNORE_CURR;
                    else
                        fuse_method = FUSE_KF_AVGE;
#elif 1 //һ�� ignore
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=T; \n");
                    fuse_method = FUSE_IGNORE_CURR;
#endif
                }
            }
            else{ //if(!isInclined){ //���Ǳ�Ե, ���ڲ�
                //*seenConfid = min(Tsdf::MAX_WEIGHT, *seenConfid + 1); //v11.4 �� weiFactor ֮��, ���ﷴ���� BUG!!
                *seenConfid = min(SCHAR_MAX, *seenConfid + 1);

                if(!isSnormPrevInit){ //vray.prev ��δ��ʼ��, �� < epsilon �ж�
                    //if (*seenConfid > seenConfidTh) //����� hadSeen, ���Բ�Ҫ��ô�ж�
                        //doUpdateVrayAndSnorm = true;
                }


                if(!hadSeen){ //�� seen-flag δ��ʼ����
#if 0   //< v11.3
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=F; >>FUSE_RESET\n");
                    *flag_pos = true;
                    fuse_method = FUSE_RESET;
#elif 1 //v11.3
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=F; seenConfid, seenConfidTh: %d, %d, ++FUSE_KF_AVGE~~~~~\n", *seenConfid, seenConfidTh); //��Ҳû�� reset ��
                    fuse_method = FUSE_KF_AVGE;
#endif
                    //if (*seenConfid > seenConfidTh) //��Ȼ hadSeen �߼��Ĺ�, ��˴���Ȼһֱ false
                    //    doUpdateVrayAndSnorm = true;
                }
                else{ //if(hadSeen) //��֮ǰ seen, ��Ȼ������ ��isInclined-F; hadSeen=F���׶�, Ҳ��Ȼ isSnormPrevInit->true, ������ if-isSnormPrevInit
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=T;\n");

                    //if(cos_norm > cosThresh ){ //�нǽǶ� <30��, ����ͬ�ӽ�
                    if(!isNewFace){ //ͬ�ӽ�, ˫ cos �����ж�
                        //TODO...
                        fuse_method = FUSE_KF_AVGE; //��ʵĬ��

                        //if (*seenConfid > seenConfidTh) //����, ��Ϊ���� if-hadSeen ��֧��
#if 0
                        if(cos_norm > cosThreshSnorm) //��֮ cos_norm < th ʱ, ���� newFace=false, ����Ӧ update
                            doUpdateVrayAndSnorm = true;

                        if(!isSnormPrevInit)
                            doUpdateVrayAndSnorm = true;
#elif 1 //�ĳɱ�Ȼ update @2017-7-13 15:45:12
                        doUpdateVrayAndSnorm = true;
#endif
                    }
                    else{ // >30��, ������ͬ�ӽ�, ����ת��ͷ֮��
                        //if(!isSnormPrevInit) //newFace �Ľ�֮��, ���ﲻ���ٽ���
                        //    doUpdateVrayAndSnorm = true;

#if 10   //���಻����, �������岻��ȷ, ���� @2017-3-24 17:50:24
                        //����Ϊ����
                        if(tsdf_curr < 0 && tsdf_curr < tsdf_prev1){
                            if(doDbgPrint)
                                printf("\ttsdf < 0 && tsdf < tsdf_prev1; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);

                            fuse_method = FUSE_IGNORE_CURR;
                        }
                        else if(tsdf_prev1 < 0 && tsdf_prev1 < tsdf_curr){
                            if(doDbgPrint){
                                printf("\ttsdf_prev1 < 0 && tsdf_prev1 < tsdf; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                                printf("\t\t, weight_prev1, w2ndCntThresh: %d, %d\n", weight_prev1, w2ndCntThresh);
                            }
#if 0   //weight_prev1 �Ƿ�Ҫ�� w2ndCntThresh �Ա�?
                            if(weight_prev1 > w2ndCntThresh){
                                fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
                            }
                            else{
                                fuse_method = FUSE_KF_AVGE; //����Ĭ���Ƿ�������
                            }
#elif 1 //1st ���� w2ndCntThresh �Ա�, ��Ϊ�������Աȿ���: weight_new2nd > w2ndCntThresh
                            fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
#endif
                            //doUpdateSnorm = true; //�ŵ� FUSE_FIX_PREDICTION ���ж�
                        }
                        else if(tsdf_curr >=0 && tsdf_prev1 >= 0){
                            if(doDbgPrint){
                                printf("\ttsdf >=0 && tsdf_prev1 >= 0; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                            }
                            fuse_method = FUSE_KF_AVGE;

                            //if(cos_norm > 0) //��Լ��: ����ͻ�䲻�ܳ�90��; ��Ϊ�˷�ֹ��Ƭ����, ��ԭʼ���ͼ����, ���¾ɵı��淨�����;  @2017-11-17 15:39:06
                            //��--�Ƶ� v12 ��, ������ @2017-12-3 22:09:36
                            doUpdateVrayAndSnorm = true;
                        }
#endif
                    }//cos vs. cosTh
                }//if-hadSeen
            }//if-isInclined
            }//if-(weiFactor > 0.2)

            const int Wrk = max(15 * weiFactor, 1.f);
            if(FUSE_KF_AVGE == fuse_method){
                float tsdf_new1 = (tsdf_prev1 * weight_prev1 + Wrk * tsdf_curr) / (weight_prev1 + Wrk);
                int weight_new1 = min (weight_prev1 + Wrk, Tsdf::MAX_WEIGHT);

                int weight_new2nd = max(weight_prev2nd - Wrk, 0); //--, ����ֹ <0

                pack_tsdf (tsdf_new1, weight_new1, *pos1);
                pack_tsdf(tsdf_prev2nd, weight_new2nd, *pos2nd); //���� 2nd �Ƿ�������ʼ����

                if(doDbgPrint)
                    printf("++FUSE_KF_AVGE, tsdf_new1, weight_new1; tsdf_prev2nd, weight_new2nd, (%f, %d), (%f, %d)\n", tsdf_new1, weight_new1, tsdf_prev2nd, weight_new2nd);
            }
            else if(FUSE_FIX_PREDICTION == fuse_method){ //ȡ���ֱ� FUSE_RESET
#if 0   //factor/step ��ʽ����
//                   //const int pos_neg_factor = 8;
//                   int pos_neg_factor = min(weight_prev1 / 10, 1); //������ʱ���� w1 ��ʵ����, ���Բ��ֱܴ��趨�󲽳�
//                   int pnWrk = pos_neg_factor * Wrk;
//                   float tsdf_new2nd = (tsdf_prev2nd * weight_prev2nd + pnWrk * tsdf) / (weight_prev2nd + pnWrk);
//                   int weight_new2nd = min (weight_prev2nd + pnWrk, Tsdf::MAX_WEIGHT);
// 
//                   int weight_new1 = max(weight_prev1 - pnWrk, 0);
// 
//                   if(weight_new2nd > weight_new1){ //�� 2nd ��Ϯ, �򽻻� 1st/2nd, ��Զ���� 1st Ϊ��
#elif 1
                float tsdf_new2nd = (tsdf_prev2nd * weight_prev2nd + Wrk * tsdf_curr) / (weight_prev2nd + Wrk);
                int weight_new2nd = min (weight_prev2nd + Wrk, Tsdf::MAX_WEIGHT);

                //int weight_new1 = max(weight_prev1 - Wrk, 0);
                if(weight_new2nd > w2ndCntThresh){ //���� 1st/2nd, ��Զ���� 1st Ϊ�� //����ĳ�: 2nd ������Ϯ 1st, ֻҪ����ĳ������ֵ����
#endif
                    if(doDbgPrint){
                        printf("weight_new2nd > w2ndCntThresh,,, exchanging 1st-2nd\n");
                    }
                    pack_tsdf(tsdf_new2nd, weight_new2nd, *pos1); //new-2nd �ŵ� pos-1st ��
                    //pack_tsdf(tsdf_prev1, weight_new1, *pos2nd);

                    doUpdateVrayAndSnorm = true; //ֱ�� 2nd ��Ϯ, �����µ� snorm ���µ�ǰ vxl
                }
                else{ //����
                    //pack_tsdf(tsdf_prev1, weight_new1, *pos1);
                    pack_tsdf(tsdf_new2nd, weight_new2nd, *pos2nd);
                    doUpdateVrayAndSnorm = false;
                }

                if(doDbgPrint)
                    //printf("...>>FUSE_FIX_PREDICTION, weight_new1, weight_new2nd, %d, %d\n", weight_new1, weight_new2nd);
                    printf("...>>FUSE_FIX_PREDICTION, tsdf_new2nd, weight_new2nd, %f, %d\n", tsdf_new2nd, weight_new2nd);

                //����: ����doDbgPrint, ȫ�����, ��������û���ߵ���һ���� vxl: @2017-3-11 21:22:59
                //��: ��!! ��Ϊ FUSE_FIX_PREDICTION Ŀǰ��� case: tsdf_prev1 < 0 && tsdf_prev1 < tsdf
                //printf("...>>FUSE_FIX_PREDICTION, weight_new2nd, %d,,, [xyz]=(%d, %d, %d)\n", weight_new2nd, x, y, z);
            }
            else if(FUSE_RESET == fuse_method){
                if(doDbgPrint)
                    printf(">>FUSE_RESET\n");

                pack_tsdf(tsdf_curr, 1, *pos1);
            }
            else if(FUSE_IGNORE_CURR == fuse_method){
                if(doDbgPrint)
                    printf("==FUSE_IGNORE_CURR: weight_prev2nd, Wrk: %d, %d\n", weight_prev2nd, Wrk);

                //DO-NOTHING!!! //��
                //IGNʱ, ҲҪ 2nd Ūһ�� @2017-3-16 03:53:08
                int weight_new2nd = max(weight_prev2nd - Wrk, 0); //--, ����ֹ <0
                pack_tsdf(tsdf_prev2nd, weight_new2nd, *pos2nd); //���� 2nd �Ƿ�������ʼ����
            }

            if(doDbgPrint)
                printf("doUpdateSnorm: %d\n", doUpdateVrayAndSnorm);

            if(doUpdateVrayAndSnorm){
                //max (-DIVISOR, min (DIVISOR, (int)nearbyintf (tsdf * DIVISOR))); //@pack_tsdf
                //��Ϊ vray_normed.xyz ��Ȼ�� <=1, ���Բ��� max/min... ��
                (*vrayPrev_pos).x = (int)nearbyintf(vray_normed.x * CHAR_MAX); //float2char
                (*vrayPrev_pos).y = (int)nearbyintf(vray_normed.y * CHAR_MAX);
                (*vrayPrev_pos).z = (int)nearbyintf(vray_normed.z * CHAR_MAX);

                //���� pcc �� nmap ����֮��, ��Ե����׼ (��Ϊ sobel?), Ҫ�е�; ������һЩ���� @2017-3-15 16:54:25
                //�� 4:=7/2+1
                const int edgeMarg = 4;
                if(coo.x < edgeMarg || coo.x >= depthScaled.cols - edgeMarg || coo.y < edgeMarg || coo.y >= depthScaled.rows - edgeMarg){
                    if(doDbgPrint)
                        printf("+++++++++++++++at edge, dont-update-snorm; coo.xy: (%d, %d)\n", coo.x, coo.y);
                }
                else{
                    //(*snorm_pos).w += 1; //�� snormPrevConfid
                    *snormPrevConfid = min(SCHAR_MAX, *snormPrevConfid + 1);

                    if(!isSnormPrevInit || isNewFace){
                        if(doDbgPrint)
                            printf("\t(!isSnormPrevInit || isNewFace): %d, %d; snormPrevConfid: %d\n", isSnormPrevInit, isNewFace, (*snorm_pos).w);

                        (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                        (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                        (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
                    }
                    else{ //isSnormPrevInit && !isNewFace //v11.6: ��snorm ��ʼ������, �ҵ�ǰû��ͻ��, ���� model �ķ���, ��Ϊ����ȶ�
                        if(doDbgPrint)
                            printf("\tisSnormPrevInit && !isNewFace\n");

                        float3 snorm_model_g;
                        snorm_model_g.x = nmap_model_g.ptr(coo.y)[coo.x];
                        snorm_model_g.y = nmap_model_g.ptr(coo.y + depthScaled.rows)[coo.x];
                        snorm_model_g.z = nmap_model_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

                        float cos_vray_norm_model = dot(snorm_model_g, vray_normed);
                        if(cos_vray_norm_model > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                            //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                            //���費��֤�ⲿ����ȷԤ����
                            snorm_model_g.x *= -1;
                            snorm_model_g.y *= -1;
                            snorm_model_g.z *= -1;
                        }
                        if(doDbgPrint)
                            printf("\t\tcos_vray_norm_model, %f; snorm_model_g: [%f, %f, %f], vray_normed: [%f, %f, %f]\n", cos_vray_norm_model, snorm_model_g.x, snorm_model_g.y, snorm_model_g.z, vray_normed.x, vray_normed.y, vray_normed.z);

                        float cos_norm_model_and_prev = dot(snorm_model_g, snormPrev);
                        //��--����˵, ��ʱ n_model, n_curr Ӧ�üнǺ�С (�Ѿ��������������� ��1 �˷�) //v11.7   @2017-3-17 15:52:25
                        //������Ϊ����, ���� n_model ƫ�����, ����ȫ������:
                        //if(cos_norm_model_and_prev > cosThreshSnorm){
                        //if(1){ //���� snormPrev ����

                        //zc: ���߼�: �� snorm-model/curr ����Ÿ��� @2017-4-25 21:24:23
                        float cos_norm_model_and_curr = dot(snorm_model_g, snorm_curr_g);
                        if(cos_norm_model_and_curr > cosThreshSnorm){
                            //���� __float2int_rd �� round-down �����˻�����, ��ֵ���ȶ�, ���� nearbyintf (������)?  @2017-3-15 15:33:33
                            (*snorm_pos).x = (int)nearbyintf(snorm_model_g.x * CHAR_MAX); //float2char
                            (*snorm_pos).y = (int)nearbyintf(snorm_model_g.y * CHAR_MAX);
                            (*snorm_pos).z = (int)nearbyintf(snorm_model_g.z * CHAR_MAX);
                        }
                        else{
                            //DO-NOTHING!!!
                        }
                    }//if-(isSnormPrevInit && !isNewFace)
                }//cut-edgeMarg

                if(doDbgPrint){
                    printf("newVray: [%d, %d, %d]\n", (*vrayPrev_pos).x, (*vrayPrev_pos).y, (*vrayPrev_pos).z);
                    printf("\tnewSnorm: [%d, %d, %d]\n", (*snorm_pos).x, (*snorm_pos).y, (*snorm_pos).z);
                }
            }//if-(doUpdateVrayAndSnorm)
          }//if- (Dp_scaled != 0 && sdf >= -tranc_dist)
          else{
              if(doDbgPrint)
                  printf("NOT (Dp_scaled != 0 && sdf >= -tranc_dist)\n");
          }
        }//if- 0 < (x,y) < (cols,rows)
      }// for(int z = 0; z < VOLUME_Z; ++z)
    }//tsdf23_v11_remake


    __global__ void
    tsdf23_v12 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const PtrStepSz<short> diff_dmap, //v12.1
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;

      //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
              printf("coo.xy:(%d, %d)\n", coo.x, coo.y);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];
          short diff_depth = diff_dmap.ptr(coo.y)[coo.x];
#if 0
          if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
#else
          //��--v11.7: �� wmap (weight) ��̬�趨 tranc_dist ����, (����׼����:
          //float tranc_dist_real = tranc_dist * weiFactor;
          float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8

          if(doDbgPrint){
              printf("\ttranc_dist_real, weiFactor: (%f, %f); diff_depth:= %d\n", tranc_dist_real, weiFactor, diff_depth);
          }

          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters
          //if (Dp_scaled != 0 && -tranc_dist_real <= sdf && sdf < tranc_dist) //meters, v11.8
#endif
          {
            float sdf_normed = sdf * tranc_dist_inv;
            float tsdf_curr = fmin (1.0f, sdf_normed);

            bool isInclined = (incidAngleMask.ptr(coo.y)[coo.x] != 0); //̫��б��, �����̫��
            float3 snorm_curr_g;
            snorm_curr_g.x = nmap_curr_g.ptr(coo.y)[coo.x];
            if(isnan(snorm_curr_g.x)){
                if(doDbgPrint)
                    printf("+++++++++++++++isnan(snorm_curr_g.x), weiFactor: %f\n", weiFactor);

                return;
            }

            snorm_curr_g.y = nmap_curr_g.ptr(coo.y + depthScaled.rows)[coo.x];
            snorm_curr_g.z = nmap_curr_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

            float3 vrayPrev;
            //�����ѹ�һ��, ��Ȼ char->float �����, ����������һ��
            vrayPrev.x = 1.f * (*vrayPrev_pos).x / CHAR_MAX; //char2float
            vrayPrev.y = 1.f * (*vrayPrev_pos).y / CHAR_MAX;
            vrayPrev.z = 1.f * (*vrayPrev_pos).z / CHAR_MAX;

            //v11.3: �� vrayPrev_pos[3] �� hadSeenConfidence, ȡ�� hadSeen ������: //2017-3-11 21:40:24
            signed char *seenConfid = &vrayPrev_pos->w;
            const int seenConfidTh = 15;

            float3 vray; //��β�����������Ҫ�ж�, �˴�ֻ���������� nmap ���ζԴ�
                            //v11.2 �ĳɶ�Ҫ��: ���� & ���淨��˫���ж� //2017-3-8 22:00:32
            vray.x = v_g_x;
            vray.y = v_g_y;
            vray.z = v_g_z;
            //float vray_norm = norm(vray);
            float3 vray_normed = normalized(vray); //��λ��������

            float cos_vray_norm = dot(snorm_curr_g, vray_normed);
            if(cos_vray_norm > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                //���費��֤�ⲿ����ȷԤ����
                snorm_curr_g.x *= -1;
                snorm_curr_g.y *= -1;
                snorm_curr_g.z *= -1;
            }

            float3 snormPrev;
            snormPrev.x = 1.f * (*snorm_pos).x / CHAR_MAX; //char2float
            snormPrev.y = 1.f * (*snorm_pos).y / CHAR_MAX;
            snormPrev.z = 1.f * (*snorm_pos).z / CHAR_MAX;

            //v11.9: ��ʱ�� snorm ����������س�ʼ��, ��ʵֵȴ������ȥ���� snorm @2017-4-11 17:03:51
            signed char *snormPrevConfid = &snorm_pos->w;
            const int snormPrevConfid_thresh = 5;

            //const bool hadSeen = *flag_pos; //���� hadSeen, ��׼ȷ
            const bool hadSeen = (*seenConfid > seenConfidTh); //v11.3: ����, ������ confid++, �ﵽ��ֵ֮��, �ű�� seen; ���ﲻ����ֵ, ��Ҫ--

            //bool isSnormPrevInit = (norm(snormPrev) > 1e-8);
            //bool isSnormPrevInit = ( (norm(snormPrev) > 1e-8) && (snormPrevConfid > snormPrevConfid_thresh) );
            bool isSnormPrevInit = (*snormPrevConfid > snormPrevConfid_thresh); //ȥ�� X>1e-8 �ж�, ��Ϊ confid > th ʱ��Ȼ X �Ѿ���ʼ������

            if(doDbgPrint){
                printf("isInclined, %d\n", isInclined);
                printf("cos_vray_norm, %f; snorm_curr_g: [%f, %f, %f], vray_normed: [%f, %f, %f]\n", cos_vray_norm, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z, vray_normed.x, vray_normed.y, vray_normed.z);
                printf("(norm(snormPrev) == 0) == %s; (norm(snormPrev) < 1e-8) == %s\n",
                    norm(snormPrev) == 0 ? "T" : "F",
                    norm(snormPrev) < 1e-8 ? "T" : "F");
            }


            //read and unpack
            float tsdf_prev1;
            int weight_prev1;
            unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);

            float tsdf_prev2nd = -123;
            int weight_prev2nd = -233;
            unpack_tsdf (*pos2nd, tsdf_prev2nd, weight_prev2nd);

            //const int w2ndCntThresh = 10; //w2nd ��������ֵ������Ϯ
            const int w2ndCntThresh = 10 * 10; //v11.4 �� weiFactor ֮��

            if(doDbgPrint){
                printf("tsdf_prev: tsdf1st: %f, %d; tsdf2nd: %f, %d;\n", tsdf_prev1, weight_prev1, tsdf_prev2nd, weight_prev2nd);
            }

            int fuse_method = FUSE_KF_AVGE; //Ĭ��ԭ����
            bool doUpdateVrayAndSnorm = false;

            const float cosThreshVray = //0.8660254f; //cos(30��)
                //0.9396926f; //cos(20��) //�� largeIncidMask ȡ 80 ��ֵʱ, �˴�ӦΪ (90-x)*2
                0.9659258f; //cos(15��) //��ΪlargeIncidMask �� 75��Ϊ��ֵ, ���������� 90-75=15 Ϊ��ֵ
                //0.996194698; //cos(5��)
            const float cosThreshSnorm = 0.8660254f; //cos(30��), �� vray ���ֿ�, ���ø�������ֵ @2017-3-15 00:39:18

            float cos_norm = dot(snormPrev, snorm_curr_g);
            float cos_vray = dot(vrayPrev, vray_normed);
            bool isNewFace = (isSnormPrevInit && cos_norm < cosThreshSnorm && cos_vray < cosThreshVray); //snorm-init ֮����� newFace �ж� @2017-4-21 00:42:00
            //bool isNewFace = (isSnormPrevInit && cos_norm < cosThreshSnorm); //ȥ�� vray �ж�, ��! ԭ��: vray ��ֹ *�ӽ��ȶ���snorm ͻ�� (��Եetc.)* ����, ������ isNewFace=true

            //zc: �����ж�, �� weight-factor ̫С(��, ��Ե����), ��ֱ�Ӿ�ֵ, �Ҳ� updateVray @2017-7-13 22:29:39
            if(weiFactor > 0.2){


            if(doDbgPrint){
                printf("cos_norm: snormPrev, snorm_curr_g, %f, [%f, %f, %f], [%f, %f, %f]\n", cos_norm, 
                    snormPrev.x, snormPrev.y, snormPrev.z, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);
                printf("\tcos_vray, vrayPrev, vray_normed, %f, [%f, %f, %f], [%f, %f, %f]\n", cos_vray, 
                    vrayPrev.x, vrayPrev.y, vrayPrev.z, vray_normed.x, vray_normed.y, vray_normed.z);
                printf("%s, snormPrevConfid, snormPrevConfid_thresh: %d, %d\n", isNewFace ? "isNewFace-T" : "isNewFace-F", *snormPrevConfid, snormPrevConfid_thresh);
                printf("\t%s\n", cos_norm > cosThreshSnorm ? "cos_norm > cosThreshSnorm" : "cos_norm <= cosThreshSnorm");
                printf("\t%s\n", cos_vray > cosThreshVray ? "cos_vray > cosThreshVray" : "cos_vray <= cosThreshVray");
            }


            if(isInclined){ //����Ե, doUpdateVray ���� false
                if(!hadSeen){ //�� seen-flag δ��ʼ����
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=F; ++FUSE_KF_AVGE\n");
                    fuse_method = FUSE_KF_AVGE;

                    //*seenConfid = max(0, *seenConfid - 1);
                    //��-- ��Ҫ -1 ��, ֻ������, ��ͬʱ seenConfidTh ��ֵ���� (5 -> 15), �ӻ��� flag=true   @2017-3-23 11:11:55
                }
                else{ //if(hadSeen) //��֮ǰ seen
#if 0   //���� sdf < 0 ����ж�Ϊʲô��, Ŀǰ�о��ᵼ����ƫ��, ����   @2017-3-9 15:06:22
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=T; %s; sdf: %f\n", sdf<0 ? "==FUSE_IGNORE_CURR" : "++FUSE_KF_AVGE", sdf);
                    if(sdf < 0)
                        fuse_method = FUSE_IGNORE_CURR;
                    else
                        fuse_method = FUSE_KF_AVGE;
#elif 1 //һ�� ignore
                    if(doDbgPrint)
                        printf("isInclined-T; hadSeen=T; \n");
                    fuse_method = FUSE_IGNORE_CURR;
#endif
                }
            }
            else{ //if(!isInclined){ //���Ǳ�Ե, ���ڲ�
                //*seenConfid = min(Tsdf::MAX_WEIGHT, *seenConfid + 1); //v11.4 �� weiFactor ֮��, ���ﷴ���� BUG!!
                *seenConfid = min(SCHAR_MAX, *seenConfid + 1);

                if(!isSnormPrevInit){ //vray.prev ��δ��ʼ��, �� < epsilon �ж�
                    //if (*seenConfid > seenConfidTh) //����� hadSeen, ���Բ�Ҫ��ô�ж�
                        //doUpdateVrayAndSnorm = true;
                }


                if(!hadSeen){ //�� seen-flag δ��ʼ����
#if 0   //< v11.3
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=F; >>FUSE_RESET\n");
                    *flag_pos = true;
                    fuse_method = FUSE_RESET;
#elif 1 //v11.3
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=F; seenConfid, seenConfidTh: %d, %d, ++FUSE_KF_AVGE~~~~~\n", *seenConfid, seenConfidTh); //��Ҳû�� reset ��
                    fuse_method = FUSE_KF_AVGE;
#endif
                    //if (*seenConfid > seenConfidTh) //��Ȼ hadSeen �߼��Ĺ�, ��˴���Ȼһֱ false
                    //    doUpdateVrayAndSnorm = true;
                }
                else{ //if(hadSeen) //��֮ǰ seen, ��Ȼ������ ��isInclined-F; hadSeen=F���׶�, Ҳ��Ȼ isSnormPrevInit->true, ������ if-isSnormPrevInit
                    if(doDbgPrint)
                        printf("isInclined-F; hadSeen=T;\n");

                    //if(cos_norm > cosThresh ){ //�нǽǶ� <30��, ����ͬ�ӽ�
                    if(!isNewFace){ //ͬ�ӽ�, ˫ cos �����ж�
                        //TODO...
                        fuse_method = FUSE_KF_AVGE; //��ʵĬ��

                        //if (*seenConfid > seenConfidTh) //����, ��Ϊ���� if-hadSeen ��֧��
#if 0
                        if(cos_norm > cosThreshSnorm) //��֮ cos_norm < th ʱ, ���� newFace=false, ����Ӧ update
                            doUpdateVrayAndSnorm = true;

                        if(!isSnormPrevInit)
                            doUpdateVrayAndSnorm = true;
#elif 1 //�ĳɱ�Ȼ update @2017-7-13 15:45:12
                        doUpdateVrayAndSnorm = true;
#endif
                    }
                    else{ // >30��, ������ͬ�ӽ�, ����ת��ͷ֮��
                        //if(!isSnormPrevInit) //newFace �Ľ�֮��, ���ﲻ���ٽ���
                        //    doUpdateVrayAndSnorm = true;

#if 10   //���಻����, �������岻��ȷ, ���� @2017-3-24 17:50:24
                        //����Ϊ����
                        if(tsdf_curr < 0 && tsdf_curr < tsdf_prev1){
                            if(doDbgPrint)
                                printf("\ttsdf < 0 && tsdf < tsdf_prev1; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);

                            fuse_method = FUSE_IGNORE_CURR;
                        }
                        else if(tsdf_prev1 < 0 && tsdf_prev1 < tsdf_curr){
                            if(doDbgPrint){
                                printf("\ttsdf_prev1 < 0 && tsdf_prev1 < tsdf; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                                printf("\t\t, weight_prev1, w2ndCntThresh: %d, %d\n", weight_prev1, w2ndCntThresh);
                            }
#if 0   //weight_prev1 �Ƿ�Ҫ�� w2ndCntThresh �Ա�?
                            if(weight_prev1 > w2ndCntThresh){
                                fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
                            }
                            else{
                                fuse_method = FUSE_KF_AVGE; //����Ĭ���Ƿ�������
                            }
#elif 0 //1st ���� w2ndCntThresh �Ա�, ��Ϊ�������Աȿ���: weight_new2nd > w2ndCntThresh
                            fuse_method = FUSE_FIX_PREDICTION; //�ñ��� volume, ����-��+
#elif 0 //v12.1 �ĳ�: ���帺ʱ, �ж� diff_depth @2017-12-3 22:29:04
                            //����, ûɶ�� @2017-12-4 02:59:18
                            if(tsdf_curr <= 0){ //ͬ��
                                fuse_method = FUSE_FIX_PREDICTION;
                            }
                            else{ //if(tsdf_curr > 0) //���帺
                                if(diff_depth > 30) //diff�㹻��, ������FIX; ������AVG; //������ diff<0
                                    fuse_method = FUSE_FIX_PREDICTION;
                                else
                                    fuse_method = FUSE_KF_AVGE; //��ʵĬ��
                            }
#elif 1 //v12.2 ��Ƭ��, �������帺ʱ, �ڲ������, ���ⶼ����, �����Ҳ��������
                            //�˲���˼·: �����(��)֮��ֻ��һ��voxΪ��ʱ, ����vox��ֵΪ��, ��Ҫ���帺; ����� vox�ϴ�ʱ�Ƚ�����; �Բ��� @2017-12-10 22:29:45

                            if(tsdf_curr < 0) //ͬ��, ���� FIX
                                fuse_method = FUSE_FIX_PREDICTION; 
                            else{
                                //���´��� ���帺 ����:
                                int grid_dx, grid_dy, grid_dz;
                                grid_dx = grid_dy = grid_dz = 0;

                                //����ֵ, �ж� vray ������Χ 27(ʵ��26) �������һ��; 
                                //�򲻻ᶼ< sqrt(1/3), �ʲ��õ��� dxyz=000
                                const float vray_which_grid_thresh = 0.577350269; //sqrt(1/3)

                                if(vray_normed.x > vray_which_grid_thresh)
                                    grid_dx = 1;
                                else if(vray_normed.x < -vray_which_grid_thresh)
                                    grid_dx = -1;
                                //else grid_dx = 0; //Ĭ��

                                if(vray_normed.y > vray_which_grid_thresh)
                                    grid_dy = 1;
                                else if(vray_normed.y < -vray_which_grid_thresh)
                                    grid_dy = -1;

                                if(vray_normed.z > vray_which_grid_thresh)
                                    grid_dz = 1;
                                else if(vray_normed.z < -vray_which_grid_thresh)
                                    grid_dz = -1;

                                int nbr_x, nbr_y, nbr_z;
                                nbr_x = min(VOLUME_X-1, max(0, x+grid_dx));
                                nbr_y = min(VOLUME_Y-1, max(0, y+grid_dy));
                                nbr_z = min(VOLUME_Z-1, max(0, z+grid_dz));

                                //volume1 ��, �����߷���, ��ǰ vox ���ڽ�(nbr) vox:
                                short2 *nbr_pos1 = volume1.ptr(nbr_y) + nbr_x;
                                nbr_pos1 += nbr_z * elem_step;

                                float nbr_tsdf_prev1;
                                int nbr_weight_prev1;
                                unpack_tsdf(*nbr_pos1, nbr_tsdf_prev1, nbr_weight_prev1);

                                char4 *nbr_vrayPrev_pos = vrayPrevVolume.ptr(nbr_y) + nbr_x;
                                //int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);
                                nbr_vrayPrev_pos += nbr_z * vrayPrev_elem_step;

                                float3 nbr_vrayPrev;

                                //�����ѹ�һ��, ��Ȼ char->float �����, ����������һ��
                                nbr_vrayPrev.x = 1.f * (*nbr_vrayPrev_pos).x / CHAR_MAX; //char2float
                                nbr_vrayPrev.y = 1.f * (*nbr_vrayPrev_pos).y / CHAR_MAX;
                                nbr_vrayPrev.z = 1.f * (*nbr_vrayPrev_pos).z / CHAR_MAX;

                                float cos_vrayCurr_nbrPrev = dot(nbr_vrayPrev, vray_normed);

                                if(nbr_tsdf_prev1 < 0)
                                    fuse_method = FUSE_FIX_PREDICTION;
                                else{ //if(nbr_tsdf_prev1 >= 0) 
                                    if(cos_vrayCurr_nbrPrev >= 0)
                                        fuse_method = FUSE_FIX_PREDICTION;
                                    else //if(cos_vrayCurr_nbrPrev < 0) //��ʱ��Ҫ FIX, ����������� tsdf ͬ��
                                        fuse_method = FUSE_KF_AVGE; 
                                }
                            }//if-(tsdf_curr >= 0)
#elif 1 //v12.3 //˼·:������, �����˲���vox2, �� vox2 �Ƿ��ȶ�
                            //���㹲ʶ: 
                            //1, ���帺ʱ, ���� diff>0
                            //2, ���ȶ�����, �ó�; �ȶ�����, ���ó�


#endif
                            //doUpdateSnorm = true; //�ŵ� FUSE_FIX_PREDICTION ���ж�
                        }
                        else if(tsdf_curr >=0 && tsdf_prev1 >= 0){
                            if(doDbgPrint){
                                printf("\ttsdf >=0 && tsdf_prev1 >= 0; [:=prev1, curr: %f, %f\n", tsdf_prev1, tsdf_curr);
                            }
                            fuse_method = FUSE_KF_AVGE;

                            if(cos_norm > 0) //��Լ��: ����ͻ�䲻�ܳ�90��; ��Ϊ�˷�ֹ��Ƭ����, ��ԭʼ���ͼ����, ���¾ɵı��淨�����;  @2017-11-17 15:39:06
                                doUpdateVrayAndSnorm = true;
                        }
#endif
                    }//cos vs. cosTh
                }//if-hadSeen
            }//if-isInclined
            }//if-(weiFactor > 0.2)

            const int Wrk = max(15 * weiFactor, 1.f);
            if(FUSE_KF_AVGE == fuse_method){
                float tsdf_new1 = (tsdf_prev1 * weight_prev1 + Wrk * tsdf_curr) / (weight_prev1 + Wrk);
                int weight_new1 = min (weight_prev1 + Wrk, Tsdf::MAX_WEIGHT);

                int weight_new2nd = max(weight_prev2nd - Wrk, 0); //--, ����ֹ <0

                pack_tsdf (tsdf_new1, weight_new1, *pos1);
                pack_tsdf(tsdf_prev2nd, weight_new2nd, *pos2nd); //���� 2nd �Ƿ�������ʼ����

                if(doDbgPrint)
                    printf("++FUSE_KF_AVGE, tsdf_new1, weight_new1; tsdf_prev2nd, weight_new2nd, (%f, %d), (%f, %d)\n", tsdf_new1, weight_new1, tsdf_prev2nd, weight_new2nd);
            }
            else if(FUSE_FIX_PREDICTION == fuse_method){ //ȡ���ֱ� FUSE_RESET
#if 0   //factor/step ��ʽ����
//                   //const int pos_neg_factor = 8;
//                   int pos_neg_factor = min(weight_prev1 / 10, 1); //������ʱ���� w1 ��ʵ����, ���Բ��ֱܴ��趨�󲽳�
//                   int pnWrk = pos_neg_factor * Wrk;
//                   float tsdf_new2nd = (tsdf_prev2nd * weight_prev2nd + pnWrk * tsdf) / (weight_prev2nd + pnWrk);
//                   int weight_new2nd = min (weight_prev2nd + pnWrk, Tsdf::MAX_WEIGHT);
// 
//                   int weight_new1 = max(weight_prev1 - pnWrk, 0);
// 
//                   if(weight_new2nd > weight_new1){ //�� 2nd ��Ϯ, �򽻻� 1st/2nd, ��Զ���� 1st Ϊ��
#elif 1
                float tsdf_new2nd = (tsdf_prev2nd * weight_prev2nd + Wrk * tsdf_curr) / (weight_prev2nd + Wrk);
                int weight_new2nd = min (weight_prev2nd + Wrk, Tsdf::MAX_WEIGHT);

                //int weight_new1 = max(weight_prev1 - Wrk, 0);
                //if(weight_new2nd > w2ndCntThresh){ //���� 1st/2nd, ��Զ���� 1st Ϊ�� //����ĳ�: 2nd ������Ϯ 1st, ֻҪ����ĳ������ֵ����
                if(weight_new2nd > weight_prev1 / 2){ //��ô���� w2 ���ȶ���? �����ó�����ֵ, ���� w1/2 (���Ǿ����Գ���), ����: �� w1 ���ȶ�, �� w2 ��Ϯ��(��)�� @2017-12-10 22:42:57
#endif
                    if(doDbgPrint){
                        printf("weight_new2nd > w2ndCntThresh,,, exchanging 1st-2nd\n");
                    }
                    pack_tsdf(tsdf_new2nd, weight_new2nd, *pos1); //new-2nd �ŵ� pos-1st ��
                    //pack_tsdf(tsdf_prev1, weight_new1, *pos2nd);

                    doUpdateVrayAndSnorm = true; //ֱ�� 2nd ��Ϯ, �����µ� snorm ���µ�ǰ vxl
                }
                else{ //����
                    //pack_tsdf(tsdf_prev1, weight_new1, *pos1);
                    pack_tsdf(tsdf_new2nd, weight_new2nd, *pos2nd);
                    doUpdateVrayAndSnorm = false;
                }

                if(doDbgPrint)
                    //printf("...>>FUSE_FIX_PREDICTION, weight_new1, weight_new2nd, %d, %d\n", weight_new1, weight_new2nd);
                    printf("...>>FUSE_FIX_PREDICTION, tsdf_new2nd, weight_new2nd, (%f, %d); tprev1, wprev1: (%f, %d)\n", tsdf_new2nd, weight_new2nd, tsdf_prev1, weight_prev1);

                //����: ����doDbgPrint, ȫ�����, ��������û���ߵ���һ���� vxl: @2017-3-11 21:22:59
                //��: ��!! ��Ϊ FUSE_FIX_PREDICTION Ŀǰ��� case: tsdf_prev1 < 0 && tsdf_prev1 < tsdf
                //printf("...>>FUSE_FIX_PREDICTION, weight_new2nd, %d,,, [xyz]=(%d, %d, %d)\n", weight_new2nd, x, y, z);
            }
            else if(FUSE_RESET == fuse_method){
                if(doDbgPrint)
                    printf(">>FUSE_RESET\n");

                pack_tsdf(tsdf_curr, 1, *pos1);
            }
            else if(FUSE_IGNORE_CURR == fuse_method){
                if(doDbgPrint)
                    printf("==FUSE_IGNORE_CURR: weight_prev2nd, Wrk: %d, %d\n", weight_prev2nd, Wrk);

                //DO-NOTHING!!! //��
                //IGNʱ, ҲҪ 2nd Ūһ�� @2017-3-16 03:53:08
                int weight_new2nd = max(weight_prev2nd - Wrk, 0); //--, ����ֹ <0
                pack_tsdf(tsdf_prev2nd, weight_new2nd, *pos2nd); //���� 2nd �Ƿ�������ʼ����
            }

            if(doDbgPrint)
                printf("doUpdateSnorm: %d\n", doUpdateVrayAndSnorm);

            if(doUpdateVrayAndSnorm){
                //max (-DIVISOR, min (DIVISOR, (int)nearbyintf (tsdf * DIVISOR))); //@pack_tsdf
                //��Ϊ vray_normed.xyz ��Ȼ�� <=1, ���Բ��� max/min... ��
                (*vrayPrev_pos).x = (int)nearbyintf(vray_normed.x * CHAR_MAX); //float2char
                (*vrayPrev_pos).y = (int)nearbyintf(vray_normed.y * CHAR_MAX);
                (*vrayPrev_pos).z = (int)nearbyintf(vray_normed.z * CHAR_MAX);

                //���� pcc �� nmap ����֮��, ��Ե����׼ (��Ϊ sobel?), Ҫ�е�; ������һЩ���� @2017-3-15 16:54:25
                //�� 4:=7/2+1
                const int edgeMarg = 4;
                if(coo.x < edgeMarg || coo.x >= depthScaled.cols - edgeMarg || coo.y < edgeMarg || coo.y >= depthScaled.rows - edgeMarg){
                    if(doDbgPrint)
                        printf("+++++++++++++++at edge, dont-update-snorm; coo.xy: (%d, %d)\n", coo.x, coo.y);
                }
                else{
                    //(*snorm_pos).w += 1; //�� snormPrevConfid
                    *snormPrevConfid = min(SCHAR_MAX, *snormPrevConfid + 1);

                    if(!isSnormPrevInit || isNewFace){
                        if(doDbgPrint)
                            printf("\t(!isSnormPrevInit || isNewFace): %d, %d; snormPrevConfid: %d\n", isSnormPrevInit, isNewFace, (*snorm_pos).w);

                        (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                        (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                        (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
                    }
                    else{ //isSnormPrevInit && !isNewFace //v11.6: ��snorm ��ʼ������, �ҵ�ǰû��ͻ��, ���� model �ķ���, ��Ϊ����ȶ�
                        if(doDbgPrint)
                            printf("\tisSnormPrevInit && !isNewFace\n");

                        float3 snorm_model_g;
                        snorm_model_g.x = nmap_model_g.ptr(coo.y)[coo.x];
                        snorm_model_g.y = nmap_model_g.ptr(coo.y + depthScaled.rows)[coo.x];
                        snorm_model_g.z = nmap_model_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

                        float cos_vray_norm_model = dot(snorm_model_g, vray_normed);
                        if(cos_vray_norm_model > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                            //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                            //���費��֤�ⲿ����ȷԤ����
                            snorm_model_g.x *= -1;
                            snorm_model_g.y *= -1;
                            snorm_model_g.z *= -1;
                        }
                        if(doDbgPrint)
                            printf("\t\tcos_vray_norm_model, %f; snorm_model_g: [%f, %f, %f], vray_normed: [%f, %f, %f]\n", cos_vray_norm_model, snorm_model_g.x, snorm_model_g.y, snorm_model_g.z, vray_normed.x, vray_normed.y, vray_normed.z);

                        float cos_norm_model_and_prev = dot(snorm_model_g, snormPrev);
                        //��--����˵, ��ʱ n_model, n_curr Ӧ�üнǺ�С (�Ѿ��������������� ��1 �˷�) //v11.7   @2017-3-17 15:52:25
                        //������Ϊ����, ���� n_model ƫ�����, ����ȫ������:
                        //if(cos_norm_model_and_prev > cosThreshSnorm){
                        //if(1){ //���� snormPrev ����

                        //zc: ���߼�: �� snorm-model/curr ����Ÿ��� @2017-4-25 21:24:23
                        float cos_norm_model_and_curr = dot(snorm_model_g, snorm_curr_g);
                        if(cos_norm_model_and_curr > cosThreshSnorm){
                            //���� __float2int_rd �� round-down �����˻�����, ��ֵ���ȶ�, ���� nearbyintf (������)?  @2017-3-15 15:33:33
                            (*snorm_pos).x = (int)nearbyintf(snorm_model_g.x * CHAR_MAX); //float2char
                            (*snorm_pos).y = (int)nearbyintf(snorm_model_g.y * CHAR_MAX);
                            (*snorm_pos).z = (int)nearbyintf(snorm_model_g.z * CHAR_MAX);
                        }
                        else{
                            //DO-NOTHING!!!
                        }
                    }//if-(isSnormPrevInit && !isNewFace)
                }//cut-edgeMarg

                if(doDbgPrint){
                    printf("newVray: [%d, %d, %d]\n", (*vrayPrev_pos).x, (*vrayPrev_pos).y, (*vrayPrev_pos).z);
                    printf("\tnewSnorm: [%d, %d, %d]\n", (*snorm_pos).x, (*snorm_pos).y, (*snorm_pos).z);
                }
            }//if-(doUpdateVrayAndSnorm)
          }//if- (Dp_scaled != 0 && sdf >= -tranc_dist)
          else{
              if(doDbgPrint)
                  printf("NOT (Dp_scaled != 0 && sdf >= -tranc_dist)\n");
          }
        }//if- 0 < (x,y) < (cols,rows)
      }// for(int z = 0; z < VOLUME_Z; ++z)
    }//tsdf23_v12

    enum{   //v13.2
        SAME_SIDE_VIEW
        ,OPPOSITE_VIEW
        ,GRAZING_VIEW   //�ݶ�: ���ں� @2017-12-22 14:44:08
        ,GRAZING_VIEW_POS
        ,GRAZING_VIEW_NEG
    };

    enum{
        WEIGHT_RESET_FLAG = -1
        ,WEIGHT_SCALE = 10 //���� w float ����ʱ, w<1 �ᱻ int �ض�, �� unpack ��/pack ǰ ��� scale, �����м�����ʱ int�ضϵ��³���

        ,TDIST_MIN_MM = 5 //5mm
        ,TDIST_MAX_MM = 25 //25mm
    };
#define SLIGHT_POSITIVE 1e-2

    //������ʱ��ͬ v12, host ���������ݽ��� v12 ��    @2018-1-5 16:30:48
    __global__ void
    tsdf23_v13 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const PtrStepSz<short> diff_dmap, //v12.1
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;
      float pendingFixThresh = cell_size.x * tranc_dist_inv * 3; //v13.4+ �õ�: �ݶ� 3*vox ���

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;

      //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
              printf("coo.xy:(%d, %d)\n", coo.x, coo.y);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];

          //zc: ���v11, �ݷ��� tranc_dist_real ����, ���Կ� @2017-12-13 10:54:29
          //if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          
          //��--���û� tranc_dist_real; Ч������, �ȵ�������Ȩ�غ�, ��Եֻ����������? @2017-12-29 10:58:14
          float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8
          if(doDbgPrint) printf("\ttranc_dist_real, weiFactor: %f, %f\n", tranc_dist_real, weiFactor);

          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters
          {
            //����Ľ�:
            //1, tsdf=sdf_normed, ֱ���� sdf ֵ, ���� fmin (1.0f, sdf_normed);
            //2, snorm ���»���: curr & prev_model ˭ abs-tsdf С, ��˭�� norm?  //�������£�
            //3, ��ͬ�ӽ��ж�����: ��������vray, ֻ�� snorm; �ݶ�����ѹ���� char4; c&prev snorm-angle >30��
            //  ��--��Ȼ���� vray-snorm_p �н���Ϊ��ͬ�ӽ��ж�ָ��
            //4, �����ж���ͬ�ӽ�
            //5, FIX ���Բ�Ҫ�� volume2nd Ӱ��, ֱ���ô�Ȩ��
            //6, ������ wmap, incidMask, ƽʱ��Ȩ��, ׷����ͨ�����Ϲ⻬

            float sdf_normed = sdf * tranc_dist_inv;
            float tsdf_curr = fmin (1.0f, sdf_normed);
            //float tsdf_curr = sdf_normed; //����ԭ����: tsdf ���ǽض�, ���ò��ضϵļ���, ���� sdf_normed @2017-12-25 01:53:06

            float3 snorm_curr_g;
            snorm_curr_g.x = nmap_curr_g.ptr(coo.y)[coo.x];
            if(isnan(snorm_curr_g.x)){
                if(doDbgPrint)
                    printf("+++++++++++++++isnan(snorm_curr_g.x), weiFactor: %f\n", weiFactor);

                return;
            }

            snorm_curr_g.y = nmap_curr_g.ptr(coo.y + depthScaled.rows)[coo.x];
            snorm_curr_g.z = nmap_curr_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

            float3 vray;
            vray.x = v_g_x;
            vray.y = v_g_y;
            vray.z = v_g_z;
            //float vray_norm = norm(vray);
            float3 vray_normed = normalized(vray); //��λ��������

            float cos_vray_norm_curr = dot(snorm_curr_g, vray_normed);
            if(cos_vray_norm_curr > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                //���費��֤�ⲿ����ȷԤ����
                snorm_curr_g.x *= -1;
                snorm_curr_g.y *= -1;
                snorm_curr_g.z *= -1;
            }

            float3 snorm_prev_g;
            snorm_prev_g.x = 1.f * (*snorm_pos).x / CHAR_MAX; //char2float
            snorm_prev_g.y = 1.f * (*snorm_pos).y / CHAR_MAX;
            snorm_prev_g.z = 1.f * (*snorm_pos).z / CHAR_MAX;

            //v11.9: ��ʱ�� snorm ����������س�ʼ��, ��ʵֵȴ������ȥ���� snorm @2017-4-11 17:03:51
            signed char *snormPrevConfid = &snorm_pos->w;
            const int snormPrevConfid_thresh = 5;

            //bool isSnormPrevInit = (norm(snormPrev) > 1e-8);
            //bool isSnormPrevInit = ( (norm(snormPrev) > 1e-8) && (snormPrevConfid > snormPrevConfid_thresh) );
            bool isSnormPrevInit = (*snormPrevConfid > snormPrevConfid_thresh); //ȥ�� X>1e-8 �ж�, ��Ϊ confid > th ʱ��Ȼ X �Ѿ���ʼ������

            //read and unpack
            float tsdf_prev1;
            int weight_prev1;
            unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);

            int fuse_method = FUSE_KF_AVGE; //Ĭ��ԭ����
            bool doUpdateVrayAndSnorm = false;

            const float COS30 = 0.8660254f
                       ,COS45 = 0.7071f
                       ,COS60 = 0.5f
                       ,COS75 = 0.258819f
                       ;
            const float cosThreshSnorm = COS30; //cos(30��), �� vray ���ֿ�, ���ø�������ֵ @2017-3-15 00:39:18

            float cos_snorm_p_c = dot(snorm_prev_g, snorm_curr_g);
            //bool isNewFace = (isSnormPrevInit && cos_snorm_p_c < cosThreshSnorm /*&& cos_vray < cosThreshVray*/); //snorm-init ֮����� newFace �ж� @2017-4-21 00:42:00
            //��--ȥ�� vray �ж�
            //����--����, ����Ե��ʼʱ�����ȶ�, �������޷����� @2017-12-20 09:38:00
            float cos_vray_norm_prev = dot(snorm_prev_g, vray_normed);
            //bool isNewFace = (isSnormPrevInit && cos_vray_norm_prev > 0); //��֮ǰsnormУ������, ������Ϊ: ͬ���ӽ���, cos(vray, n_p)<0
            int view_case = SAME_SIDE_VIEW; //����ȡ�� isNewFace @2017-12-22 10:58:03
            if(isSnormPrevInit){ //����δ snorm-init, ������Ĭ�� same-view
                if(abs(cos_vray_norm_prev) < COS75){ //б���ж�
                    view_case = GRAZING_VIEW; //v13.3: ��DEPRECATED�� ��, �� p�ڱ�Ե���·���-���߼нǺܴ�, ���޷��� c����; ��ʼ��,֮���,����޸�?

                    //if(abs(cos_vray_norm_curr) < COS75) //v13.3.2: ���뵱ǰ֡Ҳ��б��, ���򱣳� same-side ���� ��DEPRECATED��
                    //    view_case = GRAZING_VIEW;

                    //v13.4: �˻ص� vray ֻ�� snorm-prev �Ƚ�, �� ���� pos-neg-graz ���ֲ��Էֿ�����, ���ںϲ��Ը�Ϊ: 
                    //1. �� p>0:: ����: �� snorm-confid== MAX, ������ c����, ������ (wc=0); �� else: wc=1 �ں�;    snorm ��������
                    //2. �� p<0, ���� �� |p| > cellSz/tdist * �� ��e.g.: 600mm/256=2.34mm, ��/25=0.09375 �ǹ�һ���ľ���vox�߶�; ���Ǿ���ϵ��, �ݶ� 3, ��Ҫ |p|>3����
                    //                  ������ vox: snorm=0, confid=0, tsdf=SLIGHT_POSITIVE(΢>0, Ϊ���й����, ��ȡ����; ���ֺ�ҪС, �Ա�����ױ�����֡����)
                    //             �� �� else, �� c����
                    
                    //������ GRAZING_VIEW, ���� ENUM pos-neg-graz, ���ڴ������, �� cos><0 & p><0 ���жϡ� @2017-12-24 23:53:48
                    //if(cos_vray_norm_prev < 0)
                    //    view_case = GRAZING_VIEW_POS;
                    //else
                    //    view_case = GRAZING_VIEW_NEG;
                }
                else if(cos_vray_norm_prev < -COS75){ //ͬ������
                    view_case = SAME_SIDE_VIEW;
                }
                else{ //if(cos_vray_norm_prev > COS75) //��������
                    view_case = OPPOSITE_VIEW;
                }

            }


            if(doDbgPrint){
                printf("vray_normed: [%f, %f, %f]; cos_vray_norm_prev, %f; cos_vray_norm_curr, %f (%s, ALWAYS cos<0)\n", 
                    vray_normed.x, vray_normed.y, vray_normed.z, cos_vray_norm_prev, cos_vray_norm_curr, cos_vray_norm_curr>0? "��":"��");
                //�����ӡ snorm У��֮ǰ�� cos-vray-snorm_c (У��֮���Ȼ cos <0 ��); snorm ȴ��У��֮��� @2017-12-20 10:43:19
                printf("cos_snorm_p_c: %f ---snorm_prev_g, snorm_curr_g: [%f, %f, %f], [%f, %f, %f]\n", 
                    cos_snorm_p_c, snorm_prev_g.x, snorm_prev_g.y, snorm_prev_g.z, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);

                printf("isSnormPrevInit: %s, --snormPrevConfid: %d\n", 
                    isSnormPrevInit ? "TRUE":"FALSE", *snormPrevConfid);

                //printf("%s isNewFace:::", isNewFace? "YES":"NOT");
                printf("%s", view_case==SAME_SIDE_VIEW ? "SAME-SIDE" : (view_case==GRAZING_VIEW ? "GRAZING" : "OPPO-SIDE") );
                printf("::: tsdf_prev1, tsdf_curr: %f, %f\n", tsdf_prev1, tsdf_curr);
            }

            //1, weighting ����
            //float weight_curr = 1; //AVG, FIX, IGN, ������, ��Ȩ�ؾ���һ�� @2017-12-14 10:53:54
            float weight_curr = 0; //���� view_case ������֮��, Ĭ��Ȩ������
            float tsdf_new1 = SLIGHT_POSITIVE; //����º�� tsdf & w
            int weight_new = WEIGHT_RESET_FLAG;
            bool grazing_reset = false;

            //if(!isNewFace){ //ͬ�ӽ�, 
            if(SAME_SIDE_VIEW == view_case){
                /*
                //��DEPRECATED��
                if(tsdf_curr < 0 && tsdf_prev1 >= 0){ //������
                    //����: 1, curr ��ǳ; 2, prev�ܴ�, ��۲⵽Զ����������, ��֮ǰ��ΪĳЩ��Եȫ����, ����ΪԶ������, ��Ҫ����
                    if(tsdf_prev1 > 1)
                        fuse_method = FUSE_FIX_PREDICTION;
                    //else //Ĭ�� AVG
                        //fuse_method = FUSE_KF_AVGE;
                }
                else if(tsdf_curr < 0 && tsdf_prev1 < 0){ //���帺
                    //Ĭ�� AVG
                }
                else if(tsdf_curr >= 0 && tsdf_prev1 >= 0){ //������
                    //Ĭ�� AVG
                }
                else{ //if(tsdf_curr >=0 && tsdf_prev1 < 0) //���帺

                }
                */

                /*
                //��DEPRECATED��
                if(tsdf_prev1 >= 0){ //prev��
                    //weight_curr = min(1, tsdf_prev1 / min(1, abs(tsdf_curr)) ); //��, ���� min
                    //weight_curr = max(1, tsdf_prev1 / max(1, abs(tsdf_curr)) ); //��ĸ����, �ȼۼ�
                    weight_curr = max(1.f, min(tsdf_prev1, tsdf_prev1 / abs(tsdf_curr)) ); //

                    //��-ReLU, ��ȡmax; ���� LReLU, ��: 
                    //http://blog.csdn.net/mao_xiao_feng/article/details/53242235?locationNum=9&fps=1
                }
                else{ //if-(tsdf_prev1 < 0)
                    //w_curr Ĭ��=1
                    //�� c(+)>>p(-), ��ʵ
                }
                */

                weight_curr = (abs(tsdf_prev1)<=1 && abs(tsdf_curr)<=1) ? 1 : abs(tsdf_prev1 / tsdf_curr); //�� tsdf û�� sdf, ��˴������1, �� pack ʱ��ǿ�ƹ�һ��, ���Դ˴���Ч
                weight_curr = weight_curr * weight_curr;
            }
#if 0   //v13.4: GRAZING_VIEW, not graz-pos-neg
            else if(GRAZING_VIEW == view_case){//�Ը���
                if(*snormPrevConfid > Tsdf::MAX_WEIGHT_V13 / 2.f){
                    weight_curr = 0;
                }
                else{//snormPrevConfid ��С
                    if(tsdf_prev1 > 0){
                        weight_curr = 1;
                    }
                    else{ //tsdf_prev1 <0
                        float pendingFixThresh = cell_size.x * tranc_dist_inv * 3; //�ݶ� 3*vox ���
                        if(doDbgPrint){
                            printf("GRAZING_VIEW-snormPrevConfid(<Th)-tsdf_prev1(<0)-pendingFixThresh: %f\n", pendingFixThresh);
                            printf("\ttsdf_prev1 > -pendingFixThresh: %s; sdf_normed: %f\n", tsdf_prev1 > -pendingFixThresh ? "TTT" : "FFF", sdf_normed);
                        }

                        if(tsdf_prev1 > -pendingFixThresh){
                            weight_curr = 0;
                        }
                        else{//tsdf_prev1 < -pendingFixThresh
                            if(sdf_normed > 1){
                                *snormPrevConfid = 0;
                                //snorm=0�� �ݲ���, ��Ϊ confid ����, ������Ȼ�������� snorm?

                                grazing_reset = true;
                                //tsdf_new1 = 0.1f; //�ź���, ������ᱻ���渲�ǻ���
                                //weight_new = 0;
                            }
                        }
                    }

                }
            }
#elif 0 //v13.5: GRAZ �ж����ȼ�: �� c��1; if-c>>1, ��� graz-pos-neg; �� ???
            //�� sdf-normed ��, ���ں��� tsdf
            else if(GRAZING_VIEW == view_case){//�Ը���
                if(doDbgPrint){
                    printf("GRAZING_VIEW--sdf_normed=%f (%s)--cos_V_N_p=%f (%s)\
                           --tsdf_prev1=%f (%s)-pendingFixThresh: %f\n", 
                        sdf_normed, sdf_normed > 1 ? ">1" : "<=1", 
                        cos_vray_norm_prev, cos_vray_norm_prev < 0 ? "<0" : ">=0",
                        tsdf_prev1, tsdf_prev1 > 0 ? ">0" : "<=0",
                        pendingFixThresh);
                    printf("\tabs(tsdf_prev1) < pendingFixThresh: %s;\n", \
                        abs(tsdf_prev1) < pendingFixThresh ? "TTT" : "FFF");
                }
                if(sdf_normed <= 1){ //��ʵҲ�� |..|<=1, ��Ϊ������ ..>= -1
                    weight_curr = 1;
                }
                else{//sdf_normed > 1
                    if(cos_vray_norm_prev < 0){ //��֮ǰ������ GRAZING_VIEW_POS
                        weight_curr = 0; //��: 1, ����ȫ����, ����Զ�������ӳ���������Чpx
                    }
                    else{ //cos_vray_norm_prev > 0, �� graz-neg
                        if(tsdf_prev1 > 0){
                            weight_curr = 0;
                        }
                        else{ //tsdf_prev1 <0
                            //if(doDbgPrint){
                            //    printf("GRAZING_VIEW-(sdf_normed > 1)-(cos_vray_norm_prev < 0)-(tsdf_prev1 <0)-pendingFixThresh: %f\n", pendingFixThresh);
                            //    printf("\ttsdf_prev1 > -pendingFixThresh: %s; sdf_normed: %f\n", tsdf_prev1 > -pendingFixThresh ? "TTT" : "FFF", sdf_normed);
                            //}

                            if(tsdf_prev1 > -pendingFixThresh)
                                weight_curr = 0;
                            else{//tsdf_prev1 < -pendingFixThresh
                                //if(sdf_normed > 1){ //�����ж���
                                *snormPrevConfid = 0;
                                //snorm=0�� �ݲ���, ��Ϊ confid ����, ������Ȼ�������� snorm?

                                grazing_reset = true;
                                //tsdf_new1 = 0.1f; //�ź���, ������ᱻ���渲�ǻ���
                                //weight_new = 0;
                            }
                        }
                    }
                }
            }
#elif 1 //v13.6: �� v13.5 �߼�:: �� |sdf|<1, AVG(w=0); ��ĳ����Լ�� RESET(confid=0); ��else IGN(w=0);
            else if(GRAZING_VIEW == view_case){//�Ը���
                weight_curr = 0; //�߼���������ȫ������

                if(doDbgPrint){
                    printf("GRAZING_VIEW--sdf_normed=%f (%s)--cos_V_N_p=%f (%s)"
                           "--tsdf_prev1=%f (%s)-pendingFixThresh: %f\n", 
                           sdf_normed, sdf_normed > 1 ? ">1" : "<=1", 
                           cos_vray_norm_prev, cos_vray_norm_prev < 0 ? "<0" : ">=0",
                           tsdf_prev1, tsdf_prev1 > 0 ? ">0" : "<=0",
                           pendingFixThresh);
                    printf("\tabs(tsdf_prev1) < pendingFixThresh: %s;\n", \
                        abs(tsdf_prev1) < pendingFixThresh ? "TTT" : "FFF");
                }
                if(sdf_normed <= 1){ //��ʵҲ�� |..|<=1, ��Ϊ������ ..>= -1; ���߼����� sdf==tsdf
#if 0   //v13.6 ���� |sdf|<1 �� wc=1 ����; ����: sdf_prev=-0.2, confid=127, sdf_curr=0.7, ��ô��? GRAZ ״̬��, �� curr ���ó�� prev
                    weight_curr = 1;
#elif 0 //v13.7��DEPRECATED�� ֮ǰΪɶ�趨 wc=1�� Ϊ���չ�graz������, ��Ե������Ϊ �������, �ж�Ϊ graz, ��Ҫƽ����
                    //���Դ��߼�Ҳ����, �޷���ȷ��������� @2017-12-29 09:11:04
                    if(cos_snorm_p_c > COS45)
                        weight_curr = 1;
                    else
                        weight_curr = 0;
#elif 0 //v13.8 ���� w �� confid �� p-c-dist (Dpc) ˫��������, ģ���˹����/��������: 
                    //�� confid Խ��, sigmaԽС, ������ curr Խ�ϸ�; �� p-c-dist Խ��, curr �� mu ԽԶ, Ȩ��ԽС
                    //�����߽���Ϊ: ��������:wc= min(0, max(1, 1-c*(Dpc-th_min)/(M*(TH-th)) ) ) 
                    //��--��, if Dpc<th: wc=1; ��, elif Dpc>TH; wc=0; �� else �м�״̬: wc= 1-c*(Dpc-th_min)/[M*(TH-th)]

                    const float tsdf_th_min = 0.2, //e.g.: 25mm*0.2=5mm
                        tsdf_TH_max = 0.6;    //e.g.: 25mm*0.6=15mm

                    float dpc = abs(tsdf_curr - tsdf_prev1);
                    weight_curr = 1 - 1.f * *snormPrevConfid / SCHAR_MAX * (dpc - tsdf_th_min) / (tsdf_TH_max - tsdf_th_min);
                    weight_curr = max(0.f, min(1.f, weight_curr));
#elif 1 //v13.9, �޸ķֶκ�����ʽ, ��Ҫ���� th, TH; Ҫ���� confid ��̬�仯�� sigma
                    float dpc = abs(tsdf_curr - tsdf_prev1);
                    float sigma = 1 - 1.f * *snormPrevConfid / SCHAR_MAX; //confid~(0,127) --> sigma~(1,0)
                    sigma = 0.2 * sigma + 0.1; //(0,1)--> (0.1, 0.3)

                    weight_curr = 1 - 1.f * *snormPrevConfid / SCHAR_MAX * (dpc - sigma) / (2 * sigma); //��ĸ�� 3��-��=2��
                    weight_curr = max(0.f, min(1.f, weight_curr));
#endif
                }
                else{//sdf_normed > 1 //���ڽ������, ������Զ�����ӵ�����
                    if(cos_vray_norm_prev > 0 && tsdf_prev1 < -pendingFixThresh) //��, 1, �����ӽ�; 2, �ܸ�, p<<0
                    //if(tsdf_prev1 < -pendingFixThresh) //v13.10, ���ж� p<<0, ȥ���������ӽǡ�Լ�� \
                            ��--������ cos_vray_norm_prev ����Ϊ: grazʱ, ��ʱ��Եȫ����, ����Զ�������"����", ���� pendingFixThresh ����, ��Ϊ���ֵ���ܲ��ȶ�, 
                    {
                        //��--��Ҫ��ʵ3D����ࡰ1/8������ֵȫ <0, ȷ�����ƻ������; ���� v12
                        int sx = snorm_prev_g.x > 0 ? 1 : -1, //sign, ������
                            sy = snorm_prev_g.y > 0 ? 1 : -1,
                            sz = snorm_prev_g.z > 0 ? 1 : -1;
                        bool doBreak = false;
                        int nbr_x = -1,
                            nbr_y = -1,
                            nbr_z = -1;
                        float nbr_tsdf;
                        int nbr_weight;
                        for(int ix=0; ix<=1 && !doBreak; ix++){
                            for(int iy=0; iy<=1 && !doBreak; iy++){
                                for(int iz=0; iz<=1 && !doBreak; iz++){
                                    if(0==ix && 0==iy && 0==iz)
                                        continue;

                                    nbr_x = min(VOLUME_X-1, max(0, x + ix*sx));
                                    nbr_y = min(VOLUME_Y-1, max(0, y + iy*sy));
                                    nbr_z = min(VOLUME_Z-1, max(0, z + iz*sz));

                                    short2 *nbr_pos = volume1.ptr(nbr_y) + nbr_x;
                                    nbr_pos += nbr_z * elem_step;

                                    //float nbr_tsdf;
                                    //int nbr_weight;
                                    unpack_tsdf(*nbr_pos, nbr_tsdf, nbr_weight);
                                    if(WEIGHT_RESET_FLAG != nbr_weight && nbr_tsdf > 0){
                                        doBreak = true;
                                        break; //����ʽ�ж���ʵҲ����ν����
                                    }
                                }
                            }
                        }//for-ix

                        if(doDbgPrint){
                            printf("\tdoBreak: %s\n", doBreak ? "doBreakTTT" : "doBreakFFF-grazing_reset");
                            printf("\tNBR-XYZ: %d, %d, %d; NBR-TSDF/w: %f, %d\n", nbr_x, nbr_y, nbr_z, nbr_tsdf, nbr_weight);
                        }

                        if(false == doBreak){
                            *snormPrevConfid = 0;
                            grazing_reset = true;
                        }
                        else
                            weight_curr = 0;
                    }//if-cos>0 & p<<0
                    else
                        weight_curr = 0;
                }//else-sdf_normed > 1
            }//elif-(GRAZING_VIEW == view_case)
#elif 1
            else if(GRAZING_VIEW_POS == view_case){
                if(snormPrevConfid < Tsdf::MAX_WEIGHT_V13 / 2.f)
                    weight_curr = 1;
                else
                    weight_curr = 0;
            }
            else if(GRAZING_VIEW_NEG == view_case){
            }
#endif
            //else{ //if-isNewFace //v13.2 ����
            else if(OPPOSITE_VIEW == view_case){ //֮ǰ if-isNewFace 
#if 0   //v13.old   ֮ǰ: ���� p, c tsdf ֵ, ���� w �ں�Ȩ��; ����w ���ȶ�, ����; ��Ӧ�����ȷ @2018-1-2 07:32:41

                //���ӽ�, ԭ����: 
                //1, ����������Ƶ�����; 
                //2, ���ͼ���˶�ģ��/��׼����, ���²�׼, Ӱ�쵽ĳЩvox; ��Ҫ��: ĳʱ�� Dmap(i) ���(�������) ���б��污Ƭ�ṹʱ, Ҫ���⴦��
                if(tsdf_prev1 >= 0){ //prev��
                    //weight_curr = max(0, tsdf_prev1 / tsdf_curr);
                    //��-�� curr<0ʱ, w=max(0, -X)=0; curr>0ʱ, c<<p ��Ȩ�ش�
                    //weight_curr = min(tsdf_prev1, max(0, tsdf_prev1 / tsdf_curr) ); //������: ���� prev<1 ����
                    //weight_curr = min(max(1.f, tsdf_prev1), max(0.f, tsdf_prev1 / tsdf_curr) );
                    weight_curr = min(max(1.f, tsdf_prev1), max(0.f, tsdf_prev1 / (tsdf_curr + (tsdf_curr>0 ? 1 : -1) * 0.01)) ); //�������
                }
                else{ //if-(tsdf_prev1 < 0) //��Ȼ p>-1, ����̫��
                    //w_curr Ĭ��=1
                    //��ǰ: �� tprev ��������; ���: �� diffDmap //��������

                    //if(tsdf_curr + tsdf_prev1 > 0) //��ǰ֡����������: 1, ����۲�, ����������׼, ���¾ֲ����ͷ��; 2, ����۲⵽Զ����������
                    //    weight_curr = 1;
                    //else
                    //    weight_curr = (tsdf_curr+1) / max(tsdf_prev1+1, 0.1);
                    weight_curr = tsdf_curr + tsdf_prev1 > 0 ? 
                        //1 : (tsdf_curr+1) / max(tsdf_prev1+1, 0.1); //1 ������, ����: �� tcurr �ܴ�, ��Ӧ��СȨ��, 1̫��
                        (-tsdf_prev1 / tsdf_curr) : (tsdf_curr+1) / max(tsdf_prev1+1, 0.1);
                }//if-tprev><0
#elif 1 //v13.10
                if(doDbgPrint){
                    printf("\tabs(tsdf_prev1) < abs(tsdf_curr): %s\n", abs(tsdf_prev1) < abs(tsdf_curr) ? "TTT-curr��Զ" : "FFF+curr����");
                }

                //if(tsdf_prev1 >= 0)
                if(abs(tsdf_prev1) < abs(tsdf_curr)) //prev ����������
                    weight_curr = 0;
                else //curr ����������
                    weight_curr = 10;


#endif
            //}//if-isNewFace OR NOT
            }//if-OPPOSITE_VIEW
            
            if(doDbgPrint){
                printf("\tweight_prev1, weight_curr:: %d, %f\n", weight_prev1, weight_curr);
            }

            //2, ���� tsdf, weight, snorm
            if(WEIGHT_RESET_FLAG != weight_prev1) //�����ĸ����
                tsdf_new1 = (tsdf_prev1 * weight_prev1 + tsdf_curr * weight_curr) 
                / (weight_prev1 + weight_curr);
            weight_new = weight_prev1; //Ĭ�ϲ�����

            //if(!isNewFace){ //��ͬ��, 
            if(SAME_SIDE_VIEW == view_case){
                //if(grazing_reset) //grazing_reset �ֲ�����, ���������ж�
                if(WEIGHT_RESET_FLAG == weight_prev1 && sdf_normed > 1){ //GRAZ ʱ, sdf>1 ʱ grazing_reset �Ľ��, 
                    if(doDbgPrint)
                        printf("\tWEIGHT_RESET_FLAG == weight_prev1 && sdf_normed > 1\n");
                }
                else{ //�� ���� same-side, δ�ܹ� grazing_reset Ӱ��; �� �� graz-reset, ���� sdf<1;
                    //Ȩ���ۻ�
                    weight_new = min(weight_prev1 + weight_curr, (float)Tsdf::MAX_WEIGHT_V13);

                    if(isSnormPrevInit){
                        //if(doDbgPrint) printf("snorm_curr_g-111: [%f, %f, %f]\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);

                        //�𲽡���΢���·�����
                        snorm_curr_g = (snorm_prev_g * weight_prev1 + snorm_curr_g * weight_curr) 
                            * (1./(weight_prev1 + weight_curr) ); //float3 û���س���

                        //if(doDbgPrint) printf("snorm_curr_g-222: [%f, %f, %f], norm(snorm_curr_g):= %f\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z, norm(snorm_curr_g));

                        //snorm_curr_g *= 1./norm(snorm_curr_g);
                        snorm_curr_g = normalized(snorm_curr_g);

                        //if(doDbgPrint) printf("snorm_curr_g-333: [%f, %f, %f]\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);
                    }

                    //�᲻����Ϊ char �洢, ǰ��ĸ���û����? ��֪�� @2017-12-18 00:55:39
                    (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                    (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                    (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);

                    //�������Ŷ�+1
                    //(*snormPrevConfid) +=1; //Ҫ������!
                    *snormPrevConfid = min(SCHAR_MAX, *snormPrevConfid + 1);

                    if(doDbgPrint){
                        printf("\t*snormPrevConfid+1\n");
                        //printf("��snorm_pos.x��: %d, %d, %f, %f, %d\n", (*snorm_pos).x, snorm_pos->x, snorm_curr_g.x * CHAR_MAX, nearbyintf(snorm_curr_g.x * CHAR_MAX), (int)nearbyintf(snorm_curr_g.x * CHAR_MAX));
                        //printf("��snorm_pos.y��: %d, %d, %f, %f, %d\n", (*snorm_pos).y, snorm_pos->y, snorm_curr_g.y * CHAR_MAX, nearbyintf(snorm_curr_g.y * CHAR_MAX), (int)nearbyintf(snorm_curr_g.y * CHAR_MAX));
                        //printf("��snorm_pos.z��: %d, %d, %f, %f, %d\n", (*snorm_pos).z, snorm_pos->z, snorm_curr_g.z * CHAR_MAX, nearbyintf(snorm_curr_g.z * CHAR_MAX), (int)nearbyintf(snorm_curr_g.z * CHAR_MAX));
                    }
                }
            }
            //else{ //������, 
            else if(OPPOSITE_VIEW == view_case){
                //Ȩ�صݼ���һ�� //���� @2017-12-17 23:56:00
                //weight_new = max(weight_prev1 - weight_curr, Tsdf::MAX_WEIGHT_V13 / 2.f);
                //��-����, ��Ȩ��û�� MAX/2 ��? //�� snorm-initialized-confidence-thresh, ��Ϊ�ﲻ���� thresh �����ߵ������֧
                weight_new = max(int(weight_prev1 - weight_curr), snormPrevConfid_thresh);

                //���� w_curr �ϴ�ʱ, ��Ҫ curr �� prev ʱ, �Źĵ�����
                if(weight_curr > 1){
                    (*snormPrevConfid) -=1;

                    if(doDbgPrint){
                        printf("*snormPrevConfid---1\n");
                    }
                }
                if(*snormPrevConfid <= snormPrevConfid_thresh){
                    *snormPrevConfid = snormPrevConfid_thresh + 1;

                    //ֱ���� curr ����:
                    (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                    (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                    (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
                }
            }
            else if(GRAZING_VIEW == view_case){
                //DO-NOTHING
                if(grazing_reset){
                    tsdf_new1 = SLIGHT_POSITIVE;
                    weight_new = WEIGHT_RESET_FLAG; //-1, �Ǹ����, ��ʾ grazing_reset ��
                }
                else /*if(WEIGHT_RESET_FLAG != weight_new)*/{
                    //����ע�� WEIGHT_RESET_FLAG
                    if(WEIGHT_RESET_FLAG == weight_prev1)
                        weight_prev1 = 0;

                    //���� same-side, Ȩ���ۻ�, norm Ҳ����У��, ǰ�� GRAZING_VIEW ������Ѿ����� weight_curr
                    weight_new = min(weight_prev1 + weight_curr, (float)Tsdf::MAX_WEIGHT_V13);

                    //�𲽡���΢���·�����
                    snorm_curr_g = (snorm_prev_g * weight_prev1 + snorm_curr_g * weight_curr) 
                        * (1./(weight_prev1 + weight_curr) ); //float3 û���س���
                    snorm_curr_g = normalized(snorm_curr_g);

                    (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                    (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                    (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);

                    //graz �£��������Ŷȡ���+1��
                    //*snormPrevConfid = min(SCHAR_MAX, *snormPrevConfid + 1);

                }
            }//if-(GRAZING_VIEW == view_case)

            //if(WEIGHT_RESET_FLAG != weight_prev1)
                pack_tsdf(tsdf_new1, weight_new, *pos1);

            if(doDbgPrint){
                printf("\ttsdf_new1, weight_new:: %f, %d\n", tsdf_new1, weight_new);
                printf("\tnew-snorm(*snorm_pos): [%d, %d, %d]\n", snorm_pos->x, snorm_pos->y, snorm_pos->z);
                printf("\tnew-snorm(*snorm_pos): [%f, %f, %f]\n", 1.f * (*snorm_pos).x / CHAR_MAX, 1.f * (*snorm_pos).y / CHAR_MAX, 1.f * (*snorm_pos).z / CHAR_MAX);
            }

          }//if-(Dp_scaled != 0 && sdf >= -tranc_dist) 
          else{
              if(doDbgPrint)
                  printf("NOT (Dp_scaled != 0 && sdf >= -tranc_dist)\n");
          }
        }//if- 0 < (x,y) < (cols,rows)
      }// for(int z = 0; z < VOLUME_Z; ++z)
    }//tsdf23_v13

    //v13 ��������: ������ snormPrevConfid ��Ϊ�� weight_curr & weight_new ��û�зֲ棿 ���ʼ��һ�£��Ƿ����һ��������    @2018-1-5 16:41:01
    //v14 ʧ��: ��ѵ:= �� ��Ҫֱ�� reset!! û�к��ҩ; �� ��������, ��ȷʵ�����׵���ƫ�� bias (2017��ƪ��ʿ����Ҳ�ᵽ); ���ܲ����˹/����ƽ��
    __global__ void
    tsdf23_v14 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const PtrStepSz<short> diff_dmap, //v12.1
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;
      float pendingFixThresh = cell_size.x * tranc_dist_inv * 3; //v13.4+ �õ�: �ݶ� 3*vox ���

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;

      //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
              printf("coo.xy:(%d, %d)\n", coo.x, coo.y);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];

          float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8
          if(doDbgPrint) printf("\ttranc_dist_real, weiFactor: %f, %f\n", tranc_dist_real, weiFactor);

          //if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters
          {
            float sdf_normed = sdf * tranc_dist_inv;
            float tsdf_curr = fmin (1.0f, sdf_normed);

            float3 snorm_curr_g;
            snorm_curr_g.x = nmap_curr_g.ptr(coo.y)[coo.x];
            if(isnan(snorm_curr_g.x)){
                if(doDbgPrint)
                    printf("+++++++++++++++isnan(snorm_curr_g.x), weiFactor: %f\n", weiFactor);

                return;
            }

            snorm_curr_g.y = nmap_curr_g.ptr(coo.y + depthScaled.rows)[coo.x];
            snorm_curr_g.z = nmap_curr_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

            float3 vray;
            vray.x = v_g_x;
            vray.y = v_g_y;
            vray.z = v_g_z;
            //float vray_norm = norm(vray);
            float3 vray_normed = normalized(vray); //��λ��������

            float cos_vray_norm_curr = dot(snorm_curr_g, vray_normed);
            if(cos_vray_norm_curr > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                //���費��֤�ⲿ����ȷԤ����
                snorm_curr_g.x *= -1;
                snorm_curr_g.y *= -1;
                snorm_curr_g.z *= -1;
            }

            float3 snorm_prev_g;
            snorm_prev_g.x = 1.f * (*snorm_pos).x / CHAR_MAX; //char2float
            snorm_prev_g.y = 1.f * (*snorm_pos).y / CHAR_MAX;
            snorm_prev_g.z = 1.f * (*snorm_pos).z / CHAR_MAX;

            //read and unpack
            float tsdf_prev1;
            int weight_prev1;
            unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);

            //signed char *snormPrevConfid = &snorm_pos->w;
            //��-v14 ����ȥ�� snormPrevConfid ������, �� w �������
            //const int snormPrevConfid_thresh = 5;

            //bool isSnormPrevInit = (*snormPrevConfid > snormPrevConfid_thresh); //ȥ�� X>1e-8 �ж�, ��Ϊ confid > th ʱ��Ȼ X �Ѿ���ʼ������
            bool isSnormPrevInit = weight_prev1 > 0; //v14 ������ w ��� snormPrevConfid ������

            const float COS30 = 0.8660254f
                       ,COS45 = 0.7071f
                       ,COS60 = 0.5f
                       ,COS75 = 0.258819f
                       ;
            const float cosThreshSnorm = COS30; //cos(30��), �� vray ���ֿ�, ���ø�������ֵ @2017-3-15 00:39:18

            float cos_snorm_p_c = dot(snorm_prev_g, snorm_curr_g);
            float cos_vray_norm_prev = dot(snorm_prev_g, vray_normed);

            int view_case = SAME_SIDE_VIEW; //����ȡ�� isNewFace @2017-12-22 10:58:03
            if(isSnormPrevInit){ //v14: ���� w
                if(abs(cos_vray_norm_prev) < COS75){ //б���ж�
                    view_case = GRAZING_VIEW; //v13.3: �� p�ڱ�Ե���·���-���߼нǺܴ�, ��ʼ��,֮���,����޸�?
                }
                else if(cos_vray_norm_prev < -COS75){ //ͬ������
                    view_case = SAME_SIDE_VIEW;
                }
                else{ //if(cos_vray_norm_prev > COS75) //��������
                    view_case = OPPOSITE_VIEW;
                }
            }

            if(doDbgPrint){
                printf("vray_normed: [%f, %f, %f]; cos_vray_norm_prev, %f; cos_vray_norm_curr, %f (%s, ALWAYS cos<0)\n", 
                    vray_normed.x, vray_normed.y, vray_normed.z, cos_vray_norm_prev, cos_vray_norm_curr, cos_vray_norm_curr>0? "��":"��");
                //�����ӡ snorm У��֮ǰ�� cos-vray-snorm_c (У��֮���Ȼ cos <0 ��); snorm ȴ��У��֮��� @2017-12-20 10:43:19
                printf("cos_snorm_p_c: %f ---snorm_prev_g, snorm_curr_g: [%f, %f, %f], [%f, %f, %f]\n", 
                    cos_snorm_p_c, snorm_prev_g.x, snorm_prev_g.y, snorm_prev_g.z, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);

                printf("isSnormPrevInit: %s, \n", isSnormPrevInit ? "TTT" : "FFF");
                //printf("isSnormPrevInit: %s, --snormPrevConfid: %d\n", 
                //    isSnormPrevInit ? "TRUE":"FALSE", *snormPrevConfid);

                //printf("%s isNewFace:::", isNewFace? "YES":"NOT");
                printf("%s", view_case==SAME_SIDE_VIEW ? "SAME-SIDE" : (view_case==GRAZING_VIEW ? "GRAZING" : "OPPO-SIDE") );
                printf("::: tsdf_prev1, tsdf_curr: %f, %f\n", tsdf_prev1, tsdf_curr);
            }

            //1, weighting ����
            //float weight_curr = 1; //AVG, FIX, IGN, ������, ��Ȩ�ؾ���һ�� @2017-12-14 10:53:54
            float weight_curr = 0; //���� view_case ������֮��, Ĭ��Ȩ������
            float tsdf_new1 = SLIGHT_POSITIVE; //����º�� tsdf & w
            int weight_new = WEIGHT_RESET_FLAG;
            bool grazing_reset = false;

            if(SAME_SIDE_VIEW == view_case){
                weight_curr = 1;
            }
            else if(GRAZING_VIEW == view_case){//�Ը���
                weight_curr = 0; //�߼���������ȫ������

                if(doDbgPrint){
                    printf("GRAZING_VIEW--sdf_normed=%f (%s)--cos_V_N_p=%f (%s)"
                        "--tsdf_prev1=%f (%s)-pendingFixThresh: %f\n", 
                        sdf_normed, sdf_normed > 1 ? ">1" : "<=1", 
                        cos_vray_norm_prev, cos_vray_norm_prev < 0 ? "<0" : ">=0",
                        tsdf_prev1, tsdf_prev1 > 0 ? ">0" : "<=0",
                        pendingFixThresh);
                    printf("\tabs(tsdf_prev1) < pendingFixThresh: %s;\n", \
                        abs(tsdf_prev1) < pendingFixThresh ? "TTT" : "FFF");
                }
                if(sdf_normed <= 1){ //��ʵҲ�� |..|<=1, ��Ϊ������ ..>= -1; ���߼����� sdf==tsdf
                    //v13.9, ��������, �޸ķֶκ�����ʽ, ��Ҫ���� th, TH; Ҫ���� confid ��̬�仯�� sigma
                    float dpc = abs(tsdf_curr - tsdf_prev1);
                    float sigma = 1 - 1.f * weight_prev1 / Tsdf::MAX_WEIGHT_V13; //confid~(0,127) --> sigma~(1,0)
                    sigma = 0.2 * sigma + 0.1; //(0,1)--> (0.1, 0.3)

                    weight_curr = 1 - 1.f * weight_prev1 / Tsdf::MAX_WEIGHT_V13 * (dpc - sigma) / (2 * sigma); //��ĸ�� 3��-��=2��
                    weight_curr = max(0.f, min(1.f, weight_curr));
                }
                else{//sdf_normed > 1 //���ڽ������, ������Զ�����ӵ�����
                    if(cos_vray_norm_prev > 0 && tsdf_prev1 < -pendingFixThresh) //��, 1, �����ӽ�; 2, �ܸ�, p<<0
                        //if(tsdf_prev1 < -pendingFixThresh) //v13.10, ���ж� p<<0, ȥ���������ӽǡ�Լ�� \
                        ��--������ cos_vray_norm_prev ����Ϊ: grazʱ, ��ʱ��Եȫ����, ����Զ�������"����", ���� pendingFixThresh ����, ��Ϊ���ֵ���ܲ��ȶ�, 
                    {
                        //��--��Ҫ��ʵ3D����ࡰ1/8������ֵȫ <0, ȷ�����ƻ������; ���� v12
                        int sx = snorm_prev_g.x > 0 ? 1 : -1, //sign, ������
                            sy = snorm_prev_g.y > 0 ? 1 : -1,
                            sz = snorm_prev_g.z > 0 ? 1 : -1;
                        bool doBreak = false;
                        int nbr_x = -1,
                            nbr_y = -1,
                            nbr_z = -1;
                        float nbr_tsdf;
                        int nbr_weight;
                        for(int ix=0; ix<=1 && !doBreak; ix++){
                            for(int iy=0; iy<=1 && !doBreak; iy++){
                                for(int iz=0; iz<=1 && !doBreak; iz++){
                                    if(0==ix && 0==iy && 0==iz)
                                        continue;

                                    nbr_x = min(VOLUME_X-1, max(0, x + ix*sx));
                                    nbr_y = min(VOLUME_Y-1, max(0, y + iy*sy));
                                    nbr_z = min(VOLUME_Z-1, max(0, z + iz*sz));

                                    short2 *nbr_pos = volume1.ptr(nbr_y) + nbr_x;
                                    nbr_pos += nbr_z * elem_step;

                                    //float nbr_tsdf;
                                    //int nbr_weight;
                                    unpack_tsdf(*nbr_pos, nbr_tsdf, nbr_weight);
                                    if(WEIGHT_RESET_FLAG != nbr_weight && nbr_tsdf > 0){
                                        doBreak = true;
                                        break; //����ʽ�ж���ʵҲ����ν����
                                    }
                                }
                            }
                        }//for-ix

                        if(doDbgPrint){
                            printf("\tdoBreak: %s\n", doBreak ? "doBreakTTT" : "doBreakFFF-grazing_reset");
                            printf("\tNBR-XYZ: %d, %d, %d; NBR-TSDF/w: %f, %d\n", nbr_x, nbr_y, nbr_z, nbr_tsdf, nbr_weight);
                        }

                        if(false == doBreak){
                            //*snormPrevConfid = 0;
                            weight_new = WEIGHT_RESET_FLAG; //֮������� vox ��Ȼ��Ч
                            grazing_reset = true; //����ǰѭ������Ч
                        }
                        else
                            weight_curr = 0;
                    }//if-cos>0 & p<<0
                    else
                        weight_curr = 0; //��дһ��, �ö�, ��ʵĬ��
                }//else-sdf_normed > 1
            }//elif-(GRAZING_VIEW == view_case)
            else if(OPPOSITE_VIEW == view_case){ //֮ǰ if-isNewFace 
                //v13.10
                if(doDbgPrint){
                    printf("\tabs(tsdf_prev1) < abs(tsdf_curr): %s\n", abs(tsdf_prev1) < abs(tsdf_curr) ? "TTT-curr��Զ" : "FFF+curr����");
                }

                weight_curr = 0; //Ĭ������

                //if(tsdf_prev1 >= 0){ //�� p+, ���� c+/- �����ܳ�
                //    weight_curr = 0;
                //}
                //if(abs(tsdf_prev1) < abs(tsdf_curr)) //prev ����������
                //    weight_curr = 0;
                //else //curr ����������
                //    weight_curr = 10;

                if(tsdf_prev1 < 0 && abs(tsdf_prev1) > abs(tsdf_curr)){
                    //��=���� p-, �� |p|>|c|, �š����ܡ�c �� p; ����Ҫ�ж��� norm_p ����, ���� nbr ȫ<0, ȷ�������

                    //������ ���� GRAZING_VIEW �߼����� @2018-1-7 21:25:12
                    int sx = snorm_prev_g.x > 0 ? 1 : -1, //sign, ������
                        sy = snorm_prev_g.y > 0 ? 1 : -1,
                        sz = snorm_prev_g.z > 0 ? 1 : -1;
                    bool doBreak = false;
                    int nbr_x = -1,
                        nbr_y = -1,
                        nbr_z = -1;
                    float nbr_tsdf;
                    int nbr_weight;
                    for(int ix=0; ix<=1 && !doBreak; ix++){
                        for(int iy=0; iy<=1 && !doBreak; iy++){
                            for(int iz=0; iz<=1 && !doBreak; iz++){
                                if(0==ix && 0==iy && 0==iz)
                                    continue;

                                nbr_x = min(VOLUME_X-1, max(0, x + ix*sx));
                                nbr_y = min(VOLUME_Y-1, max(0, y + iy*sy));
                                nbr_z = min(VOLUME_Z-1, max(0, z + iz*sz));

                                short2 *nbr_pos = volume1.ptr(nbr_y) + nbr_x;
                                nbr_pos += nbr_z * elem_step;

                                //float nbr_tsdf;
                                //int nbr_weight;
                                unpack_tsdf(*nbr_pos, nbr_tsdf, nbr_weight);
                                if(WEIGHT_RESET_FLAG != nbr_weight && nbr_tsdf > 0){
                                    doBreak = true;
                                    break; //����ʽ�ж���ʵҲ����ν����
                                }
                            }
                        }
                    }//for-ix

                    if(doDbgPrint){
                        printf("\tdoBreak: %s\n", doBreak ? "doBreakTTT" : "doBreakFFF-grazing_reset");
                        printf("\tNBR-XYZ: %d, %d, %d; NBR-TSDF/w: %f, %d\n", nbr_x, nbr_y, nbr_z, nbr_tsdf, nbr_weight);
                    }

                    if(false == doBreak){
                        //weight_curr = 10;

                        grazing_reset = true;
                    }

                }
            }//if-OPPOSITE_VIEW

            if(doDbgPrint){
                printf("\tweight_prev1, weight_curr:: %d, %f\n", weight_prev1, weight_curr);
            }

            //2, ���� tsdf, weight, snorm
            if(WEIGHT_RESET_FLAG != weight_prev1) //�����ĸ����
                tsdf_new1 = (tsdf_prev1 * weight_prev1 + tsdf_curr * weight_curr) 
                / (weight_prev1 + weight_curr);
            weight_new = weight_prev1; //Ĭ�ϲ�����

            if(SAME_SIDE_VIEW == view_case){
                //if(grazing_reset) //grazing_reset �ֲ�����, ���������ж�
                if(WEIGHT_RESET_FLAG == weight_prev1 && sdf_normed > 1){ //GRAZ ʱ, sdf>1 ʱ grazing_reset �Ľ��, 
                    if(doDbgPrint)
                        printf("\tWEIGHT_RESET_FLAG == weight_prev1 && sdf_normed > 1\n");
                }
                else{ //�� ���� same-side, δ�ܹ� grazing_reset Ӱ��; �� �� graz-reset, ���� sdf<1;
                    //Ȩ���ۻ�
                    if(WEIGHT_RESET_FLAG == weight_prev1)
                        weight_prev1 = 0;
                    weight_new = min(weight_prev1 + weight_curr, (float)Tsdf::MAX_WEIGHT_V13);

                    if(isSnormPrevInit){
                        //if(doDbgPrint) printf("snorm_curr_g-111: [%f, %f, %f]\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);

                        //�𲽡���΢���·�����
                        snorm_curr_g = (snorm_prev_g * weight_prev1 + snorm_curr_g * weight_curr) 
                            * (1./(weight_prev1 + weight_curr) ); //float3 û���س���

                        //if(doDbgPrint) printf("snorm_curr_g-222: [%f, %f, %f], norm(snorm_curr_g):= %f\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z, norm(snorm_curr_g));

                        //snorm_curr_g *= 1./norm(snorm_curr_g);
                        snorm_curr_g = normalized(snorm_curr_g);

                        //if(doDbgPrint) printf("snorm_curr_g-333: [%f, %f, %f]\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);
                    }

                    //�᲻����Ϊ char �洢, ǰ��ĸ���û����? ��֪�� @2017-12-18 00:55:39
                    (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                    (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                    (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
                }
            }
            else if(GRAZING_VIEW == view_case){
                if(grazing_reset){
                    tsdf_new1 = SLIGHT_POSITIVE;
                    weight_new = WEIGHT_RESET_FLAG; //-1, �Ǹ����, ��ʾ grazing_reset ��
                }
                else /*if(WEIGHT_RESET_FLAG != weight_new)*/{
                    //����ע�� WEIGHT_RESET_FLAG
                    if(WEIGHT_RESET_FLAG == weight_prev1)
                        weight_prev1 = 0;

                    //���� same-side, Ȩ���ۻ�, norm Ҳ����У��, ǰ�� GRAZING_VIEW ������Ѿ����� weight_curr
                    weight_new = min(weight_prev1 + weight_curr, (float)Tsdf::MAX_WEIGHT_V13);

                    //�𲽡���΢���·�����
                    snorm_curr_g = (snorm_prev_g * weight_prev1 + snorm_curr_g * weight_curr) 
                        * (1./(weight_prev1 + weight_curr) ); //float3 û���س���
                    snorm_curr_g = normalized(snorm_curr_g);

                    (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                    (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                    (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);

                    //graz �£��������Ŷȡ���+1��
                    //*snormPrevConfid = min(SCHAR_MAX, *snormPrevConfid + 1);

                }
            }//if-(GRAZING_VIEW == view_case)
            else if(OPPOSITE_VIEW == view_case){
#if 0 //v14: �����ϼ�С w-new, ֱ���� vox ��� SAME �߼�

                weight_new = max(int(weight_prev1 - weight_curr), 0);

                //���� w_curr �ϴ�ʱ, ��Ҫ curr �� prev ʱ, �Źĵ�����
                //if(weight_curr > 1){
                //    (*snormPrevConfid) -=1;

                //    if(doDbgPrint){
                //        printf("*snormPrevConfid---1\n");
                //    }
                //}

                //if(*snormPrevConfid <= snormPrevConfid_thresh){
                //    *snormPrevConfid = snormPrevConfid_thresh + 1;

                //    //ֱ���� curr ����:
                //    (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                //    (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                //    (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
                //}
#elif 1 //v14.1: oppo �����ý���Ȩ��, ֱ�� reset
                if(grazing_reset){
                    tsdf_new1 = SLIGHT_POSITIVE;
                    weight_new = WEIGHT_RESET_FLAG; //-1, �Ǹ����, ��ʾ grazing_reset ��

                    (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                    (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                    (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
                }
#endif
            }//if-(OPPOSITE_VIEW == view_case)

            if(WEIGHT_RESET_FLAG != weight_prev1)
                pack_tsdf(tsdf_new1, weight_new, *pos1);

            if(doDbgPrint){
                printf("\ttsdf_new1, weight_new:: %f, %d\n", tsdf_new1, weight_new);
                printf("\tnew-snorm(*snorm_pos): [%d, %d, %d]\n", snorm_pos->x, snorm_pos->y, snorm_pos->z);
                printf("\tnew-snorm(*snorm_pos): [%f, %f, %f]\n", 1.f * (*snorm_pos).x / CHAR_MAX, 1.f * (*snorm_pos).y / CHAR_MAX, 1.f * (*snorm_pos).z / CHAR_MAX);
            }
          }//if-(Dp_scaled != 0 && sdf >= -tranc_dist) 
          else{
              if(doDbgPrint)
                  printf("NOT (Dp_scaled != 0 && sdf >= -tranc_dist)\n");
          }
        }//if- 0 < (x,y) < (cols,rows)
      }// for(int z = 0; z < VOLUME_Z; ++z)
    }//tsdf23_v14

    //���� v14 ��ѵ, ��˼·: ���жϸ�ֵ��; ���ҽ���: �� p<0 ��ֵ����; �� w����, ��˵��֮ǰ�۲�"����"; �� cos-vray-n_p >cos75��, ������oppo�۲�, ��grazing; �� ������ n_p ����, ����ȷ�����ڹ����
    __global__ void
    tsdf23_v15 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const PtrStepSz<short> diff_dmap, //v12.1
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;
      float pendingFixThresh = cell_size.x * tranc_dist_inv * 3; //v13.4+ �õ�: �ݶ� 3*vox ���

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;

       //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
              printf("coo.xy:(%d, %d)\n", coo.x, coo.y);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];

          float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8
          if(doDbgPrint) printf("\ttranc_dist_real, weiFactor: %f, %f\n", tranc_dist_real, weiFactor);

          //if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters
          {
            float sdf_normed = sdf * tranc_dist_inv;
            float tsdf_curr = fmin (1.0f, sdf_normed);

            float3 snorm_curr_g;
            snorm_curr_g.x = nmap_curr_g.ptr(coo.y)[coo.x];
            if(isnan(snorm_curr_g.x)){
                if(doDbgPrint)
                    printf("+++++++++++++++isnan(snorm_curr_g.x), weiFactor: %f\n", weiFactor);

                return;
            }

            snorm_curr_g.y = nmap_curr_g.ptr(coo.y + depthScaled.rows)[coo.x];
            snorm_curr_g.z = nmap_curr_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

            float3 vray;
            vray.x = v_g_x;
            vray.y = v_g_y;
            vray.z = v_g_z;
            //float vray_norm = norm(vray);
            float3 vray_normed = normalized(vray); //��λ��������

            float cos_vray_norm_curr = dot(snorm_curr_g, vray_normed);
            if(cos_vray_norm_curr > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                //���費��֤�ⲿ����ȷԤ����
                snorm_curr_g.x *= -1;
                snorm_curr_g.y *= -1;
                snorm_curr_g.z *= -1;
            }

            float3 snorm_prev_g;
            snorm_prev_g.x = 1.f * (*snorm_pos).x / CHAR_MAX; //char2float
            snorm_prev_g.y = 1.f * (*snorm_pos).y / CHAR_MAX;
            snorm_prev_g.z = 1.f * (*snorm_pos).z / CHAR_MAX;

            //read and unpack
            float tsdf_prev1;
            float weight_prev1;
            int weight_prev1_scaled;
            unpack_tsdf (*pos1, tsdf_prev1, weight_prev1_scaled);
            weight_prev1 = 1.f * weight_prev1_scaled / WEIGHT_SCALE; //���ڽ���������� float w<1 ת int �ضϵĴ���

            //signed char *snormPrevConfid = &snorm_pos->w;
            //��-v14 ����ȥ�� snormPrevConfid ������, �� w �������
            //const int snormPrevConfid_thresh = 5;

            //bool isSnormPrevInit = (*snormPrevConfid > snormPrevConfid_thresh); //ȥ�� X>1e-8 �ж�, ��Ϊ confid > th ʱ��Ȼ X �Ѿ���ʼ������
            //bool isSnormPrevInit = weight_prev1 > 0; //v14 ������ w ��� snormPrevConfid ������
            bool isSnormPrevInit = weight_prev1 > 1; //v15 ��Ϊ global_time_ == 0 ʱ, �Ѿ� w=1

            const float COS30 = 0.8660254f
                       ,COS45 = 0.7071f
                       ,COS60 = 0.5f
                       ,COS75 = 0.258819f
                       ;
            const float cosThreshSnorm = COS30; //cos(30��), �� vray ���ֿ�, ���ø�������ֵ @2017-3-15 00:39:18

            float cos_snorm_p_c = dot(snorm_prev_g, snorm_curr_g);
            float cos_vray_norm_prev = dot(snorm_prev_g, vray_normed);

            int view_case = SAME_SIDE_VIEW; //����ȡ�� isNewFace @2017-12-22 10:58:03
            if(isSnormPrevInit){ //v14: ���� w
#if 0   //OLD, 
                if(abs(cos_vray_norm_prev) < COS75){ //б���ж�
                    view_case = GRAZING_VIEW; //v13.3: �� p�ڱ�Ե���·���-���߼нǺܴ�, ��ʼ��,֮���,����޸�?
                }
                else if(cos_vray_norm_prev < -COS75){ //ͬ������
                    view_case = SAME_SIDE_VIEW;
                }
                else{ //if(cos_vray_norm_prev > COS75) //��������
                    view_case = OPPOSITE_VIEW;
                }
#elif 1 //v15.2: Ϊ��Ӧ oppo ���νض�, �ſ� graz ����, �� oppo �������ϸ�
                if(cos_vray_norm_prev < -COS75){ //ͬ������
                    view_case = SAME_SIDE_VIEW;
                }
                else if(abs(cos_vray_norm_prev) < COS75 || abs(cos_vray_norm_curr) < COS75){
                    view_case = GRAZING_VIEW; //v13.3: �� p�ڱ�Ե���·���-���߼нǺܴ�, ��ʼ��,֮���,����޸�?
                }
                else{ //if(cos_vray_norm_prev > COS75) //��������
                    view_case = OPPOSITE_VIEW;
                }

#endif
            }

            if(doDbgPrint){
                printf("vray_normed: [%f, %f, %f]; cos_vray_norm_prev, %f; cos_vray_norm_curr, %f (%s, ALWAYS cos<0)\n", 
                    vray_normed.x, vray_normed.y, vray_normed.z, cos_vray_norm_prev, cos_vray_norm_curr, cos_vray_norm_curr>0? "��":"��");
                //�����ӡ snorm У��֮ǰ�� cos-vray-snorm_c (У��֮���Ȼ cos <0 ��); snorm ȴ��У��֮��� @2017-12-20 10:43:19
                printf("cos_snorm_p_c: %f ---snorm_prev_g, snorm_curr_g: [%f, %f, %f], [%f, %f, %f]\n", 
                    cos_snorm_p_c, snorm_prev_g.x, snorm_prev_g.y, snorm_prev_g.z, snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z);

                printf("isSnormPrevInit: %s, \n", isSnormPrevInit ? "TTT" : "FFF");
                //printf("isSnormPrevInit: %s, --snormPrevConfid: %d\n", 
                //    isSnormPrevInit ? "TRUE":"FALSE", *snormPrevConfid);

                //printf("%s isNewFace:::", isNewFace? "YES":"NOT");
                printf("%s", view_case==SAME_SIDE_VIEW ? "SAME-SIDE" : (view_case==GRAZING_VIEW ? "GRAZING" : "OPPO-SIDE") );
                printf("::: tsdf_prev1, tsdf_curr: %f, %f\n", tsdf_prev1, tsdf_curr);
            }

            //1, weighting ����
            //float weight_curr = 1; //AVG, FIX, IGN, ������, ��Ȩ�ؾ���һ�� @2017-12-14 10:53:54
            float weight_curr = 0; //���� view_case ������֮��, Ĭ��Ȩ������
            float tsdf_new1 = SLIGHT_POSITIVE; //����º�� tsdf & w
            float weight_new = WEIGHT_RESET_FLAG; //v15 ���� reset-flag ��? �ǲ�ס��
            int weight_new_scaled;
            bool grazing_reset = false;

            if(SAME_SIDE_VIEW == view_case){
                weight_curr = 1;
            }
            else if(GRAZING_VIEW == view_case){
                //weight_curr = 1;    //v15.0: graz ʱ��Ȼ w=1, graz ������???  @2018-1-9 14:53:21
                //��-����, б�ӱ���ʱ, e.g., -0.1 �� 1 ������ʴ, 

                //v15.1: ���� v13.9, ��������, �޸ķֶκ�����ʽ, ��Ҫ���� th, TH; Ҫ���� confid ��̬�仯�� sigma
                float dpc = abs(tsdf_curr - tsdf_prev1);
                float sigma = 1 - 1.f * weight_prev1 / Tsdf::MAX_WEIGHT_V13; //confid~(0,127) --> sigma~(1,0)
                sigma = 0.2 * sigma + 0.1; //(0,1)--> (0.1, 0.3)

                weight_curr = 1 - 1.f * weight_prev1 / Tsdf::MAX_WEIGHT_V13 * (dpc - sigma) / (2 * sigma); //��ĸ�� 3��-��=2��
                weight_curr = max(0.f, min(1.f, weight_curr));

            }
            else if(OPPOSITE_VIEW == view_case){ //֮ǰ if-isNewFace 
                //weight_curr = 0; //OLD, �ĳ�: ������ w, ��Ϊ�ܻ���� bias  
                if(tsdf_prev1 > 0){ //������ wc, ����Ҫô 0, Ҫô -wp (���������� w_new = 0, ���νض�)
                    weight_curr = 0; //��ֵ������
                }
                else if(tsdf_prev1 < 0)
                    //&& weight_prev1 > 50) //����ֵ
                {
                    //���� v14 ��ѵ, ��˼·: ���жϸ�ֵ��; ���ҽ���: �� p<0 ��ֵ����; �� w����, ��˵��֮ǰ�۲�"����"; �� cos-vray-n_p >cos75��, �����桾oppo���۲�, ��grazing; �� ������ n_p ����, ����ȷ�����ڹ����

                    int sx = snorm_prev_g.x > 0 ? 1 : -1, //sign, ������
                        sy = snorm_prev_g.y > 0 ? 1 : -1,
                        sz = snorm_prev_g.z > 0 ? 1 : -1;
                    bool doBreak = false;
                    int nbr_x = -1,
                        nbr_y = -1,
                        nbr_z = -1;
                    float nbr_tsdf;
                    int nbr_weight;
                    for(int ix=0; ix<=1 && !doBreak; ix++){
                        for(int iy=0; iy<=1 && !doBreak; iy++){
                            for(int iz=0; iz<=1 && !doBreak; iz++){
                                if(0==ix && 0==iy && 0==iz)
                                    continue;

                                nbr_x = min(VOLUME_X-1, max(0, x + ix*sx));
                                nbr_y = min(VOLUME_Y-1, max(0, y + iy*sy));
                                nbr_z = min(VOLUME_Z-1, max(0, z + iz*sz));

                                short2 *nbr_pos = volume1.ptr(nbr_y) + nbr_x;
                                nbr_pos += nbr_z * elem_step;

                                //float nbr_tsdf;
                                //int nbr_weight;
                                unpack_tsdf(*nbr_pos, nbr_tsdf, nbr_weight);
                                //if(WEIGHT_RESET_FLAG != nbr_weight && nbr_tsdf > 0){
                                if(0 != nbr_weight && nbr_tsdf > 0){ //v15.0: w_new ������ WEIGHT_RESET_FLAG, ����ֱ������
                                    doBreak = true;
                                    break; //����ʽ�ж���ʵҲ����ν����
                                }
                            }
                        }
                    }//for-ix

                    if(doDbgPrint){
                        printf("\tdoBreak: %s\n", doBreak ? "doBreakTTT=����" : "doBreakFFF-����reset");
                        printf("\tNBR-XYZ: %d, %d, %d; NBR-TSDF/w: %f, %d\n", nbr_x, nbr_y, nbr_z, nbr_tsdf, nbr_weight);
                    }

                    if(false == doBreak){
                        weight_curr = -weight_prev1;
                    }
                }//if=p<0 & w> th
            }//if-OPPOSITE_VIEW

            if(doDbgPrint){
                printf("\tweight_prev1, weight_curr:: %f, %f\n", weight_prev1, weight_curr);
            }

            //2, ���� tsdf, weight, snorm
            weight_new = min(weight_prev1 + weight_curr, (float)Tsdf::MAX_WEIGHT_V13);
            if(0 == weight_new){
                tsdf_new1 = 0;
            }
            else{ //��ĸ��Ϊ��
                tsdf_new1 = (tsdf_prev1 * weight_prev1 + tsdf_curr * weight_curr) / weight_new;
            }
            weight_new_scaled = (int)nearbyintf(weight_new * WEIGHT_SCALE);
            pack_tsdf(tsdf_new1, weight_new_scaled, *pos1);

            //2.2 ���� snorm
            if(SAME_SIDE_VIEW == view_case){
                //�𲽡���΢���·�����
                if(0 != weight_new){
                    snorm_curr_g = (snorm_prev_g * weight_prev1 + snorm_curr_g * weight_curr) 
                        * (1./weight_new ); //float3 û���س���
                    snorm_curr_g = normalized(snorm_curr_g);
                }
                (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
            }
            else if(GRAZING_VIEW == view_case){
                //DO-NOTHING
            }
            else if(OPPOSITE_VIEW == view_case){
                (*snorm_pos).x = 0;
                (*snorm_pos).y = 0;
                (*snorm_pos).z = 0;
            }

            if(doDbgPrint){
                printf("\ttsdf_new1, weight_new:: %f, %f\n", tsdf_new1, weight_new);
                printf("\tnew-snorm(*snorm_pos): [%d, %d, %d]\n", snorm_pos->x, snorm_pos->y, snorm_pos->z);
                printf("\tnew-snorm(*snorm_pos): [%f, %f, %f]\n", 1.f * (*snorm_pos).x / CHAR_MAX, 1.f * (*snorm_pos).y / CHAR_MAX, 1.f * (*snorm_pos).z / CHAR_MAX);
            }

          }//if-(Dp_scaled != 0 && sdf >= -tranc_dist) 
          else{
              if(doDbgPrint)
                  printf("NOT (Dp_scaled != 0 && sdf >= -tranc_dist)\n");
          }
        }//if- 0 < (x,y) < (cols,rows)
      }// for(int z = 0; z < VOLUME_Z; ++z)

    }//tsdf23_v15

    //v16: ���԰�, ���Խ��� tranc_dist_real ����, ���� tdist �ϴ�, ��ԵʲôЧ�� @2018-1-18 10:31:39
    __global__ void
    tsdf23_v16 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const PtrStepSz<short> diff_dmap, //v12.1
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;
      float pendingFixThresh = cell_size.x * tranc_dist_inv * 3; //v13.4+ �õ�: �ݶ� 3*vox ���

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;

       //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
              printf("coo.xy:(%d, %d)\n", coo.x, coo.y);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];

          float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8
          //float tranc_dist_real = max(cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8

          if(doDbgPrint) printf("\ttranc_dist_real, weiFactor: %f, %f\n", tranc_dist_real, weiFactor);

          //if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters
          {
            float sdf_normed = sdf * tranc_dist_inv;
            float tsdf_curr = fmin (1.0f, sdf_normed);

            //read and unpack
            float tsdf_prev;
            int weight_prev;
            unpack_tsdf (*pos1, tsdf_prev, weight_prev);

            const int Wrk = 1;

            float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf_curr) / (weight_prev + Wrk);
            int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

            if(doDbgPrint){
                printf("tsdf_prev, tsdf, tsdf_new: %f, %f, %f\n", tsdf_prev, tsdf_curr, tsdf_new);
            }

            pack_tsdf (tsdf_new, weight_new, *pos1);
          }
        }
        else{ //NOT (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)
            if(doDbgPrint){
                printf("vxlDbg.xyz:= (%d, %d, %d), coo.xy:= (%d, %d)\n", vxlDbg.x, vxlDbg.y, vxlDbg.z, coo.x, coo.y);
            }
        }
      }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__ tsdf23_v16

    //v13~v15 ʧ��, ��ѵ: ����ֱ�� reset, ������ƫ��, ��������³��, �����׵��� bias
    //v17 ���Բ���: ˫ tsdf, ���� tdist, ��̬ѡ��, �ĸ��������ĸ�; ��ȱ�㡿�� �� ����Ч����Ȼ��, �����ֹ�; �� raycast, march-cubes ������Ҫ��֮��Ķ� @2018-1-18 15:26:21
    __global__ void
    tsdf23_v17 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const PtrStepSz<short> diff_dmap, //v12.1
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;
      float pendingFixThresh = cell_size.x * tranc_dist_inv * 3; //v13.4+ �õ�: �ݶ� 3*vox ���; //ֵ������� tranc_dist ��һ������

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;
      const float tdist2nd_m = TDIST_MIN_MM / 1e3; //v17

      //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if(doDbgPrint)
            printf("inv_z:= %f\n", inv_z);

        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if(doDbgPrint)
            printf("coo.xy:(%d, %d)\n", coo.x, coo.y);

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];

          float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8
          if(doDbgPrint) printf("\ttranc_dist_real, weiFactor: %f, %f\n", tranc_dist_real, weiFactor);

          //if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters
          {
            //v17.3 ��� cos-vray-snorm_c ���ڴ˿���ǰ��, ����������� sdf ��ʼ��λ�� @2018-1-30 17:15:23
            float3 snorm_curr_g;
            snorm_curr_g.x = nmap_curr_g.ptr(coo.y)[coo.x];
            if(isnan(snorm_curr_g.x)){
                if(doDbgPrint)
                    printf("+++++++++++++++isnan(snorm_curr_g.x), weiFactor: %f\n", weiFactor);

                //return; //��, v18.x ʱ�ŷ��� @2018-3-8 15:29:28
                continue;
            }

            snorm_curr_g.y = nmap_curr_g.ptr(coo.y + depthScaled.rows)[coo.x];
            snorm_curr_g.z = nmap_curr_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

            float3 vray;
            vray.x = v_g_x;
            vray.y = v_g_y;
            vray.z = v_g_z;
            //float vray_norm = norm(vray);
            float3 vray_normed = normalized(vray); //��λ��������

            float cos_vray_norm_curr = dot(snorm_curr_g, vray_normed);
            if(cos_vray_norm_curr > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
                //printf("ERROR+++++++++++++++cos_vray_norm > 0");

                //���費��֤�ⲿ����ȷԤ����
                snorm_curr_g.x *= -1;
                snorm_curr_g.y *= -1;
                snorm_curr_g.z *= -1;
            }

            //v17.3: sdf ���� cos-vray-snorm_c ͶӰ, �ݲ��� snorm_p //����֤: Ч������, �ڱ���(��ֵ��)����, ȷʵ��Ҫ�˷�, ȷ����ȷ, ���������νض�(neg_near_zero) ����
            float sdf_cos = abs(cos_vray_norm_curr) * sdf;
            if(doDbgPrint){
                printf("snorm_curr_g, vray_normed: [%f, %f, %f], [%f, %f, %f]\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z, vray_normed.x, vray_normed.y, vray_normed.z);
                printf("sdf-orig: %f,, cos_vray_norm_curr: %f,, sdf_cos: %f\n", sdf, cos_vray_norm_curr, sdf_cos);
            }

            sdf = sdf_cos;
            float sdf_normed = sdf * tranc_dist_inv;
            float tsdf_curr = fmin (1.0f, sdf_normed);
            float sdf_normed_mm = sdf_normed * 1e3;


            float3 snorm_prev_g;
            snorm_prev_g.x = 1.f * (*snorm_pos).x / CHAR_MAX; //char2float
            snorm_prev_g.y = 1.f * (*snorm_pos).y / CHAR_MAX;
            snorm_prev_g.z = 1.f * (*snorm_pos).z / CHAR_MAX;

            //read and unpack
            float tsdf_prev1;
            int weight_prev1;
            unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);
            bool use_tdist2nd = weight_prev1 % 2; //v17.1: �� w������ĩλ=1, ���ñ��� tdist (Ŀǰ�����Ǽ�С�ض�)
            weight_prev1 = weight_prev1 >> 1; //ȥ��ĩλ, ������
            if(doDbgPrint)
                printf("use_tdist2nd-prev: %d,, tsdf_prev1: %f,, weight_prev1: %d\n", use_tdist2nd, tsdf_prev1, weight_prev1);


            float tsdf_prev1_real_m = tsdf_prev1 * (use_tdist2nd ? tdist2nd_m : tranc_dist); //

            int Wrk = 1; //Ĭ��1

            if(use_tdist2nd){
                //�˿��ڽ��޸� tsdf_curr
                tsdf_curr = fmin (1.0f, sdf / tdist2nd_m);
                if(sdf < -tdist2nd_m)
                    Wrk = 0;
            }

#if 0   //v17.0, �� volume-2nd, ����δ���; ���Ƿ�����һ�� vol �͹��� (��Ϊֻ��Ҫһ���ء�����tdist���λ��), ���Դ��߼��������ݷ���������
            float tsdf_prev2nd = -123;
            int weight_prev2nd = -233;
            unpack_tsdf (*pos2nd, tsdf_prev2nd, weight_prev2nd);

            //volume-2nd ֱ�� pack, ��������
            if(sdf >= -tdist2nd_m){
                const int Wrk = 1;
                float tsdf_curr2nd = fmin (1.0f, sdf / tdist2nd_m); //volume-2nd �趨���� tdist=5mm 
                float tsdf_new2nd = (tsdf_prev2nd * weight_prev2nd + tsdf_curr2nd * Wrk) / (weight_prev2nd + Wrk);
                int weight_new2nd = min (weight_prev2nd + Wrk, Tsdf::MAX_WEIGHT);
                pack_tsdf(tsdf_new2nd, weight_new2nd, *pos2nd);
            }

            //v17.0: �� snorm_pos->w ��¼ tdist, ÿ�� vox ����, ���� mm ����
            signed char *trunc_dist_mm = &snorm_pos->w;
            if(0 == *trunc_dist_mm) //�����λ ��û��ʼ��, ���ú���������ʼ��; ����, ���Ѵ�ı��ֵ
                *trunc_dist_mm = int(tranc_dist * 1e3 + 0.5);
            float trunc_dist_m = trunc_dist_mm / 1e3;
#endif

            //v17.2: ��֮ǰ"���濴��ֵvox, ������ǰ�������, ���vox ����" 
            //��Ϊ: ���濴��ֵ vox, �� w �ﵽĳ��ֵ, ���� "���������", ����, �ٵ����濴ʱ, ���б��, �򲻶�
            //���� snorm_pos->w �����λ, �ݲ����� w(short) @2018-1-29 00:46:48
            //signed char *neg_near_zero = &snorm_pos->w;
            bool neg_near_zero = snorm_pos->w; //��ʼ 0->false
            const int weight_neg_th = 30; 
            if(tsdf_prev1 < 0 && weight_prev1 > weight_neg_th && !neg_near_zero)//��: ��ֵ, ��Ȩ�شﵽ��ֵ, �ұ��λ��δ��ʼ��
            {
                //��ԵҪ��Ҫ�ж�, �Ա����Ե��ƽ��? ��ȷ��, �ݲ�, 
                //weiFactor

                if(tsdf_prev1_real_m > 1.1 * cell_size.x){ //�为, ������������ (��ֵ��) //���� max(x,y,z); �ж���ֵ�� csz.x, ����� //�� projTSDF ��������, ���Ը��� sdf_cos
                    neg_near_zero = true;
                    snorm_pos->w = 1; //neg_near_zero=true
                }
            }

            const float COS30 = 0.8660254f
                ,COS45 = 0.7071f
                ,COS60 = 0.5f
                ,COS75 = 0.258819f
                ;

            float cos_snorm_p_c = dot(snorm_prev_g, snorm_curr_g);

            //v17.X: snorm-p-c �н� >60��, ��Ϊ����, �����, ���ܵĲ���: 
            //�� Զ�˸���, ��Ҫ��; 
            //�� ������: a, ������, ��; b, ���帺, ??? ���������ܻᵼ�� bias, Ҫ��������!��

            if(doDbgPrint){
                printf("snorm_prev_g.xyz: (%f, %f, %f)\n", snorm_prev_g.x, snorm_prev_g.y, snorm_prev_g.z);
                printf("snorm_curr_g.xyz: (%f, %f, %f); cos_snorm_p_c: %f\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z, cos_snorm_p_c);
            }

            bool isSnormPrevInit = (norm(snorm_prev_g) > 1e-8);
            if(!isSnormPrevInit && sdf < tranc_dist){ //ֻ���ڽ����� (���� tdist ����Ǹ��ж�), �ų�ʼ�� snorm
                (*snorm_pos).x = (int)nearbyintf(snorm_curr_g.x * CHAR_MAX); //float2char
                (*snorm_pos).y = (int)nearbyintf(snorm_curr_g.y * CHAR_MAX);
                (*snorm_pos).z = (int)nearbyintf(snorm_curr_g.z * CHAR_MAX);
            }
            else if(isSnormPrevInit && cos_snorm_p_c < COS60){ //�� norm-p ��ʼ����, �� cos-n-p-c ��������
                //v17.0: ����������, ������������ tdist, �Ȱѻ������̡�������Ūͨ �����ԡ�
                //sdf //m, ��ǰ֡
                //tranc_dist, tranc_dist_inv //m, ��ǰ��������
                //tsdf_prev1 //0~1, 

                if(!use_tdist2nd){ //��2nd ���λû������, ˵����һ�ν���, Ҫ: 1, ����λ; 2, w_p=0
                    use_tdist2nd = true;

                    //float tsdf_prev1_real_m = tsdf_prev1 * tranc_dist; //֮ǰӦ�ö��õĺ������� tranc_dist //����Ҫ��, ���Էŵ�����
                    tsdf_prev1 = tsdf_prev1_real_m / tdist2nd_m; //�ݲ��� fmin(1, ..), ������ w=0
                    //if(tsdf_prev1_real_m < -tdist2nd_m){ //�� ��� tdist2nd, ̫��, ������, ��Ϊ���� -tdist2nd ��������Ҳ��δ��ʼ��״̬
                    if(tsdf_prev1_real_m < -tdist2nd_m && !neg_near_zero){ 
                        weight_prev1 = 0;
                        tsdf_prev1 = 0; //��ʵ����, ��ʽд��, �����Ķ�

                        snorm_pos->x = snorm_pos->y = snorm_pos->z = 0;
                    }
                }
                //���� use_tdist2nd T/F, t_curr �϶�Ҫ�� td-2nd ����:
                tsdf_curr = fmin (1.0f, sdf / tdist2nd_m);
                if(sdf < -tdist2nd_m){
                    Wrk = 0;
                    tsdf_curr = 0; //��ʵ����, ��ʽд��, �����Ķ�
                }
                else{
                    ////v17.5
                    //if(sdf > tdist2nd_m //��: �۲쵽Զ�˱���
                    //    && 0 != weight_prev1) //�Ҵ� vox ����֮���ֱ�Զ�˸��¹� //use_tdist2nd �Ѿ� true, �������ж�ָ��; �� weight_prev1 �ж�
                    //    Wrk = 0; //�Ͳ��ٸ���, 
                    
                    //v17.6.1: �򵥴ֱ�: �� w >th, ��Ϊ t_p �㹻�ȶ�, ��������߼� cos(n-p-c)<COS60, ����ֱ��������ǰ: w_c = 0
                    if(tsdf_curr < tsdf_prev1 && weight_prev1 > weight_neg_th) //��: c<p
                        Wrk = 0;
                }

                //v17.2: ��
                if(neg_near_zero)
                    Wrk = 0;

                //v17.7: 17.5 �Ƶ����, ������ isSnormPrevInit / cos_snorm_p_c ɶ��, ֻҪ w_c !=0, Զ��һ�ɲ����� w @2018-2-4 11:30:27
                if(sdf > (use_tdist2nd ? tdist2nd_m : tranc_dist) //��: �۲쵽Զ�˱���
                    && 0 != weight_prev1) //�Ҵ� vox ����֮���ֱ�Զ�˸��¹� //use_tdist2nd �Ѿ� true, �������ж�ָ��; �� weight_prev1 �ж�
                    Wrk = 0; //�Ͳ��ٸ���, 

                //v17.x: Զ�˸���(���帺), һ�ɲ�Ҫ��
                //v17.x: Զ�˸���(���帺), �������ж�, ����"����"�ӽ�ʱ, ������ tdist
            }//cos-norm-p-c < COS60

            //v17.4
            if(!neg_near_zero){ //��֮ǰ��̫��������ʱ, ���� t_c ����Ȩ��
                //��--��: wrk ����; �� {t_c} < {t_p}
                if(abs(Wrk) > 1e-5 && abs(tsdf_curr) < abs(tsdf_prev1) )
                {
                    float tpc_ratio = abs(tsdf_prev1) / (abs(tsdf_curr) + 1e-2); //�˿��ڽ����Ȼ >1; ��ĸtrickΪ�˱������
                    //v17.4.1: ֱ���� ratio ��Ȩ��:
                    Wrk = (int)fmin(10.f, tpc_ratio);

                    //v17.4.2: �� ratio^2, Ŀ��: �� t_c ����̫Сʱ, ��Ȼ���� t_c Ӱ����
                    Wrk = (int)fmin(10.f, tpc_ratio * tpc_ratio);
                }
            }

            float tsdf_new1 = (tsdf_prev1 * weight_prev1 + tsdf_curr * Wrk) / (weight_prev1 + Wrk);
            int weight_new1 = min (weight_prev1 + Wrk, Tsdf::MAX_WEIGHT);
            
            if(doDbgPrint){
                printf("����tsdf_prev1: %f,, weight_prev1: %d; tsdf_prev1_real_m: %f, neg_near_zero: %s\n", tsdf_prev1, weight_prev1, tsdf_prev1_real_m, neg_near_zero ? "TTT":"FFF");
                printf("����tsdf_curr: %f,, Wrk: %d; \n", tsdf_curr, Wrk);
                printf("tsdf_new1: %f,, weight_new1: %d;;; use_tdist2nd: %d\n", tsdf_new1, weight_new1, use_tdist2nd);
            }
            //pack ǰ, ��� w_new Ҫ���ϱ��λ:
            weight_new1 = (weight_new1 << 1) + use_tdist2nd;

            pack_tsdf (tsdf_new1, weight_new1, *pos1);

          }//if-(Dp_scaled != 0 && sdf >= -tranc_dist) 
          else{
              if(doDbgPrint)
                  printf("NOT (Dp_scaled != 0 && sdf >= -tranc_dist)\n");
          }
        }//if- 0 < (x,y) < (cols,rows)
      }// for(int z = 0; z < VOLUME_Z; ++z)
    }//tsdf23_v17

    //for v18, Ϊ�˲��� krnl �Ƿ� thread, block ��ʵ����, ���: OK
    __global__ void
    test_kernel (int3 vxlDbg){
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
        if(vxlDbg.x == x && vxlDbg.y == y)
            printf("dbg@test_kernel>>>xy: %d, %d\n", x, y);

    }//test_kernel

    __global__ void
    tsdf23_v18 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume1, 
        PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, const PtrStepSz<unsigned char> incidAngleMask,
        const PtrStep<float> nmap_curr_g, const PtrStep<float> nmap_model_g,
        /*��--ʵ��˳��: volume2nd, flagVolume, surfNormVolume, incidAngleMask, nmap_g,*/
        const PtrStep<float> weight_map, //v11.4
        const PtrStepSz<ushort> depthModel,
        const PtrStepSz<short> diff_dmap, //v12.1
        const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
        , int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;
      //printf("tsdf23_v18, xy: %d, %d\n", x, y);
      //if(vxlDbg.x == x && vxlDbg.y == y)
      //    printf("dbg@tsdf23_v18>>>xy: %d, %d\n", x, y);

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;
      float pendingFixThresh = cell_size.x * tranc_dist_inv * 3; //v13.4+ �õ�: �ݶ� 3*vox ���; //ֵ������� tranc_dist ��һ������

      short2* pos1 = volume1.ptr (y) + x;
      int elem_step = volume1.step * VOLUME_Y / sizeof(short2);

      //�ҵĿ�������:
      short2 *pos2nd = volume2nd.ptr(y) + x;
      const float tdist2nd_m = TDIST_MIN_MM / 1e3; //v17

      //hadSeen-flag:
      bool *flag_pos = flagVolume.ptr(y) + x;
      int flag_elem_step = flagVolume.step * VOLUME_Y / sizeof(bool);

      //vray.prev
      char4 *vrayPrev_pos = vrayPrevVolume.ptr(y) + x;
      int vrayPrev_elem_step = vrayPrevVolume.step * VOLUME_Y / sizeof(char4);

      //surface-norm.prev
      char4 *snorm_pos = surfNormVolume.ptr(y) + x;
      int snorm_elem_step = surfNormVolume.step * VOLUME_Y / sizeof(char4);

      //if(vxlDbg.x == x && vxlDbg.y == y)
      //    printf("dbg@tsdf23_v18-before-for-loop>>>xy: %d, %d\n", x, y);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos1 += elem_step,

           pos2nd += elem_step,
           flag_pos += flag_elem_step,

           vrayPrev_pos += vrayPrev_elem_step,
           snorm_pos += snorm_elem_step)
      {
        //v18.2 ���ѽ��, ��ѭ���ڲ����� return��
        //if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
        //    && vxlDbg.x == x && vxlDbg.y == y)// && vxlDbg.z == z)
        //{   //��ʱ����: ����Щ vox �޷���λ��, �ƺ�������������߼���; @2018-3-1 22:47:15
        //    printf("dbg@for-loop>>>xyz: %d, %d, %d\n", x, y, z);
        //}
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if(doDbgPrint)
            printf("inv_z:= %f\n", inv_z);

        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if(doDbgPrint)
            printf("coo.xy:(%d, %d)\n", coo.x, coo.y);

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if(doDbgPrint){
              printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
          }

          float weiFactor = weight_map.ptr(coo.y)[coo.x];
          //float tranc_dist_real = max(2*cell_size.x, tranc_dist * weiFactor); //�ضϲ���̫��, v11.8
          float tranc_dist_real = max(0.3, weiFactor) * tranc_dist; //v18.4: ��Ե���� w_factor=0, 

          float3 snorm_curr_g;
          snorm_curr_g.x = nmap_curr_g.ptr(coo.y)[coo.x];

           if(isnan(snorm_curr_g.x)){
               if(doDbgPrint)
                   printf("+++++++++++++++isnan(snorm_curr_g.x), weiFactor: %f\n", weiFactor);
 
               //return;    //��ѭ��, ÿ�ζ�Ҫ�߱� z��, ���� ����
               continue;    //v18.2
           }

          snorm_curr_g.y = nmap_curr_g.ptr(coo.y + depthScaled.rows)[coo.x];
          snorm_curr_g.z = nmap_curr_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

          float3 vray;
          vray.x = v_g_x;
          vray.y = v_g_y;
          vray.z = v_g_z;
          //float vray_norm = norm(vray);
          float3 vray_normed = normalized(vray); //��λ��������

          float cos_vray_norm_curr = dot(snorm_curr_g, vray_normed);
          if(cos_vray_norm_curr > 0){ //����assert, Ҫ�����: �н�>90��, ��������볯�������ͷ
              //printf("ERROR+++++++++++++++cos_vray_norm > 0");

              //���費��֤�ⲿ����ȷԤ����
              snorm_curr_g.x *= -1;
              snorm_curr_g.y *= -1;
              snorm_curr_g.z *= -1;
          }

          //float sdf_cos = abs(cos_vray_norm_curr) * sdf;
          float sdf_cos = max(COS75, abs(cos_vray_norm_curr)) * sdf; //v18.3: �������Ӳ���С�� COS75

          if(doDbgPrint){
              printf("snorm_curr_g, vray_normed: [%f, %f, %f], [%f, %f, %f]\n", snorm_curr_g.x, snorm_curr_g.y, snorm_curr_g.z, vray_normed.x, vray_normed.y, vray_normed.z);
              printf("sdf-orig: %f,, cos_vray_norm_curr: %f,, sdf_cos: %f\n", sdf, cos_vray_norm_curr, sdf_cos);
              printf("\ttranc_dist_real, weiFactor: %f, %f\n", tranc_dist_real, weiFactor);
          }

          sdf = sdf_cos;

          //��--v18.17: unpack Ų������
          //read and unpack
          float tsdf_prev1;
          int weight_prev1;
          unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);
          bool prev_always_edge = weight_prev1 % 2; //��DEL v17.1�� //v18.15: �����Ϊ: �Ƿ�һֱ���ڱ�Ե (��ֵ:=0:=false) @2018-3-28 15:56:33
          weight_prev1 = weight_prev1 >> 1; //ȥ��ĩλ, ֻ��Ϊ���� v17 ����һ��, ������� ������Ϊ tsdf23 �� w*2 ��
          if(doDbgPrint)
              printf("prev_always_edge-prev: %d,, tsdf_prev1: %f,, weight_prev1: %d\n", prev_always_edge, tsdf_prev1, weight_prev1);

          //if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          if (Dp_scaled != 0 && sdf >= -tranc_dist_real) //meters //v18.4
          //if (Dp_scaled != 0 && tranc_dist_real >= sdf && sdf >= -tranc_dist_real) //meters //v18.6: ������ֵԶ�˽ض�; ����������ڲ�����, �ⲿ(�����Ե)����; �ķ��ں���, �� v18.7
          {
            float tsdf_curr = fmin (1.0f, sdf * tranc_dist_inv);

            //��--�������, Ų�������� v18.17
            ////read and unpack
            //float tsdf_prev1;
            //int weight_prev1;
            //unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);
            //bool prev_always_edge = weight_prev1 % 2; //��DEL v17.1�� //v18.15: �����Ϊ: �Ƿ�һֱ���ڱ�Ե (��ֵ:=0:=false) @2018-3-28 15:56:33
            //weight_prev1 = weight_prev1 >> 1; //ȥ��ĩλ, ֻ��Ϊ���� v17 ����һ��, ������� ������Ϊ tsdf23 �� w*2 ��
            //if(doDbgPrint)
            //    printf("prev_always_edge-prev: %d,, tsdf_prev1: %f,, weight_prev1: %d\n", prev_always_edge, tsdf_prev1, weight_prev1);

            //const int Wrk = 1;
            int Wrk = 1; //v18.5: ����ȫ����: diff_dmap + ������� (�� nmap_model_g, ���� nmap-curr �ж�) @2018-3-11 11:58:55
            short diff_c_p = diff_dmap.ptr(coo.y)[coo.x]; //mm, curr-prev, +��ֵΪ��ǰ����
            ushort depth_prev = depthModel.ptr(coo.y)[coo.x];

            const int diff_c_p_thresh = 20; //20mm
            if(doDbgPrint)
                printf("depth_prev: %u; diff_c_p: %d\n", depth_prev, diff_c_p);

            if(depth_prev > 0 //����Ҫ model �� px ��Ч���ѳ�ʼ����
                && diff_c_p > diff_c_p_thresh){
                float3 snorm_prev_g;
                snorm_prev_g.x = nmap_model_g.ptr(coo.y)[coo.x];
                if(isnan(snorm_prev_g.x)){
                    if(doDbgPrint)
                        printf("\t+++++isnan(snorm_prev_g.x)\n");

                    Wrk = 0;
                }
                else{
                    snorm_prev_g.y = nmap_model_g.ptr(coo.y + depthScaled.rows)[coo.x];
                    snorm_prev_g.z = nmap_model_g.ptr(coo.y + 2 * depthScaled.rows)[coo.x];

                    float cos_vray_norm_prev = dot(snorm_prev_g, vray_normed);
                    if(doDbgPrint)
                        printf("\tsnorm_prev_g.xyz: (%f, %f, %f), cos_vray_norm_prev: %f\n", 
                            snorm_prev_g.x, snorm_prev_g.y, snorm_prev_g.z, cos_vray_norm_prev);

                    if(abs(cos_vray_norm_prev) < COS75)
                        Wrk = 0;
                }
            }//if-(diff_c_p > diff_c_p_thresh)

            //v18.7: ��Ϊ: ��һ��(w=0)�۲⵽Զ��, ��ֹ��ʼ��; 
            //�����1, ��/������� v18.6, �ڲ����� v18.5, 2, �����ⲿ���в�����Ƭ����; 3, ����ͼ(raycast���)���ѿ�!    ���ݴ桿
//             if(0 == weight_prev1 && sdf > tranc_dist_real){
//                 Wrk = 0;
//             }

            const float W_FACTOR_EDGE_THRESH = 0.99f;
            bool is_curr_edge = weiFactor < W_FACTOR_EDGE_THRESH;

            if(Wrk != 0){
                //if(0 == weight_prev1 && is_curr_edge){ //�� w-prev��δ��ʼ������ curr �ڱ�Ե
                if(weight_prev1 <= 1 && is_curr_edge){ //v18.18: �Ը�, ����, �� global_time =0 ʱ�õ� tsdf23 ֱ�� w+1 @2018-4-10 17:27:08
                    prev_always_edge = true;
                }
                else if(!is_curr_edge && prev_always_edge){
                    prev_always_edge = false;

                    //weight_prev1 = min(weight_prev1, 30); //����1: w-p ֱ�ӽ�Ȩ�� 30��1s; //����, ��t-p=1, �� 1*30 �����Ժܴ�, ������
                    weight_prev1 = min(weight_prev1, 5);
                }
            }

            float tsdf_new1 = tsdf_prev1;
            int weight_new1 = weight_prev1;
            if(Wrk > 0)
                //&& !(!prev_always_edge && is_curr_edge && tsdf_curr > 0.99) ) //��: prevȷ�ϷǱ�Ե, curr�Ǳ�Ե, �� t-cȷʵ��, �򲻸��� t, w
                //&& (prev_always_edge || !is_curr_edge || tsdf_curr <= 0.99) ) //ͬ��, 
            {
                tsdf_new1 = (tsdf_prev1 * weight_prev1 + tsdf_curr * Wrk) / (weight_prev1 + Wrk);
                weight_new1 = min (weight_prev1 + Wrk, Tsdf::MAX_WEIGHT);
            }

            if(doDbgPrint){
                //printf("����tsdf_prev1: %f,, weight_prev1: %d; tsdf_prev1_real_m: %f, neg_near_zero: %s\n", tsdf_prev1, weight_prev1, tsdf_prev1_real_m, neg_near_zero ? "TTT":"FFF");
                printf("����tsdf_prev1: %f,, weight_prev1: %d;\n", tsdf_prev1, weight_prev1);
                printf("����tsdf_curr: %f,, Wrk: %d; \n", tsdf_curr, Wrk);
                printf("tsdf_new1: %f,, weight_new1: %d;;; prev_always_edge: %d\n", tsdf_new1, weight_new1, prev_always_edge);
            }

            if(weight_new1 == 0)
                tsdf_new1 = 0; //�Ͻ���, ������Ի��ơ�marching cubes����

            //pack ǰ, ��� w_new Ҫ���ϱ��λ:
            weight_new1 = (weight_new1 << 1) + prev_always_edge;

            pack_tsdf (tsdf_new1, weight_new1, *pos1);

          }//if-(Dp_scaled != 0 && sdf >= -tranc_dist) 
//           else{
//               if(doDbgPrint)
//                   printf("NOT (Dp_scaled != 0 && sdf >= -tranc_dist)\n");
//           }
          //else if(Dp_scaled != 0 && sdf < -tranc_dist) { //v18.12: �˴�+v18.8; ��ĳvox����������һ�ۣ���������ȫ���䣬����ʱ��̣�, 
                                                            //�����ʱ�䲻�ɼ�, ��������Ȩ(����); ��������ܺ�, ���� v18.11, ����ʱ�򿴼�һ��δ��������, Ҫ��
          else if(Dp_scaled != 0 
              && sdf < -tranc_dist &&  sdf > -4*tranc_dist   //v18.13: ��-2*tdist +v18.8, �ų� v18.12 ������ //v18.14 ��-4*tdist, ��ȥ�� v18.8, ����ԭ�� marching cubes
              && !prev_always_edge  //v18.17: ���ԷǱ�Եִ�� "-1 ����", �����Ǳ�Ե(��, ϸ����), �� -1 @2018-4-8 02:32:39
            )
          {
              //��-v18.17: Ų�� if ������
              //float tsdf_prev1;
              //int weight_prev1;
              //unpack_tsdf (*pos1, tsdf_prev1, weight_prev1);
              //bool prev_always_edge = weight_prev1 % 2;
              //weight_prev1 = weight_prev1 >> 1; //ȥ��ĩλ, 

              const int POS_VALID_WEIGHT_TH = 0; //30֡��һ��
              if(/*tsdf_prev1 >= 0.999 ||*/ //�� t_p ֮ǰ��"Զ��", �ǽ�����
                  tsdf_prev1 > 0 && weight_prev1 < POS_VALID_WEIGHT_TH) //��, �� t_p ��ֵ�����в��ȶ�
              {
                  weight_prev1 = max(0, weight_prev1-1);

                  if(doDbgPrint){
                      printf("����tsdf_prev1: %f,, weight_prev1-=1: %d;\n", tsdf_prev1, weight_prev1);
                  }
              }

              if(weight_prev1 == 0)
                  tsdf_prev1 = 0; //�Ͻ���, ������Ի��ơ�marching cubes����
              weight_prev1 = (weight_prev1 << 1) + prev_always_edge;

              pack_tsdf (tsdf_prev1, weight_prev1, *pos1);
          }
        }//if- 0 < (x,y) < (cols,rows)
      }// for(int z = 0; z < VOLUME_Z; ++z)
    }//tsdf23_v18

    __global__ void
    tsdf23normal_hack (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
                  const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
    {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
            return;

        const float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
        const float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
        float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

        float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

        float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
        float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
        float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

        float z_scaled = 0;

        float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
        float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

        float tranc_dist_inv = 1.0f / tranc_dist;

        short2* pos = volume.ptr (y) + x;
        int elem_step = volume.step * VOLUME_Y / sizeof(short2);

        //#pragma unroll
        for (int z = 0; z < VOLUME_Z;
            ++z,
            v_g_z += cell_size.z,
            z_scaled += cell_size.z,
            v_x += Rcurr_inv_0_z_scaled,
            v_y += Rcurr_inv_1_z_scaled,
            pos += elem_step)
        {
            float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
            if (inv_z < 0)
                continue;

            // project to current cam
            int2 coo =
            {
                __float2int_rn (v_x * inv_z + intr.cx),
                __float2int_rn (v_y * inv_z + intr.cy)
            };

            if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
            {
                float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

                float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

                if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
                {
                    float tsdf = fmin (1.0f, sdf * tranc_dist_inv);                                              

                    bool integrate = true;
                    if ((x > 0 &&  x < VOLUME_X-2) && (y > 0 && y < VOLUME_Y-2) && (z > 0 && z < VOLUME_Z-2))
                    {
                        const float qnan = numeric_limits<float>::quiet_NaN();
                        float3 normal = make_float3(qnan, qnan, qnan);

                        float Fn, Fp;
                        int Wn = 0, Wp = 0;
                        unpack_tsdf (*(pos + elem_step), Fn, Wn);
                        unpack_tsdf (*(pos - elem_step), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.z = (Fn - Fp)/cell_size.z;

                        unpack_tsdf (*(pos + volume.step/sizeof(short2) ), Fn, Wn);
                        unpack_tsdf (*(pos - volume.step/sizeof(short2) ), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.y = (Fn - Fp)/cell_size.y;

                        unpack_tsdf (*(pos + 1), Fn, Wn);
                        unpack_tsdf (*(pos - 1), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.x = (Fn - Fp)/cell_size.x;

                        if (normal.x != qnan && normal.y != qnan && normal.z != qnan)
                        {
                            float norm2 = dot(normal, normal);
                            if (norm2 >= 1e-10)
                            {
                                normal *= rsqrt(norm2);

                                float nt = v_g_x * normal.x + v_g_y * normal.y + v_g_z * normal.z;
                                float cosine = nt * rsqrt(v_g_x * v_g_x + v_g_y * v_g_y + v_g_z * v_g_z);

                                if (cosine < 0.5)
                                    integrate = false;
                            }
                        }
                    }

                    if (integrate)
                    {
                        //read and unpack
                        float tsdf_prev;
                        int weight_prev;
                        unpack_tsdf (*pos, tsdf_prev, weight_prev);

                        const int Wrk = 1;

                        float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                        int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

                        pack_tsdf (tsdf_new, weight_new, *pos);
                    }
                }
            }
        }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // tsdf23normal_hack
  }//namespace device

    __global__ void
    tsdf23test (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
            const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size, const pcl::gpu::tsdf_buffer buffer)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= buffer.voxels_size.x || y >= buffer.voxels_size.y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      short2* pos = volume.ptr (y) + x;
      
      // shift the pointer to relative indices
      shift_tsdf_pointer(&pos, buffer);
      
      int elem_step = volume.step * buffer.voxels_size.y / sizeof(short2);

//#pragma unroll
      for (int z = 0; z < buffer.voxels_size.z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos += elem_step)
      {
        
        // As the pointer is incremented in the for loop, we have to make sure that the pointer is never outside the memory
        if(pos > buffer.tsdf_memory_end)
          pos -= (buffer.tsdf_memory_end - buffer.tsdf_memory_start + 1);
        
        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
		// old code
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          {
            float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

            //read and unpack
            float tsdf_prev;
            int weight_prev;
            unpack_tsdf (*pos, tsdf_prev, weight_prev);

            const int Wrk = 1;

            float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
            int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

            pack_tsdf (tsdf_new, weight_new, *pos);
          }
        }
      }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth, const Intr& intr,
                                  const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, 
                                  float tranc_dist,
                                  //PtrStep<short2> volume, const pcl::gpu::tsdf_buffer* buffer, DeviceArray2D<float>& depthScaled)
                                  PtrStep<short2> volume, const pcl::gpu::tsdf_buffer* buffer, DeviceArray2D<float>& depthScaled, int3 vxlDbg) //zc: ����
{
  depthScaled.create (depth.rows, depth.cols);

  dim3 block_scale (32, 8);
  dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

  //scales depth along ray and converts mm -> meters. 
  scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
  cudaSafeCall ( hipGetLastError () );

  float3 cell_size;
  cell_size.x = volume_size.x / buffer->voxels_size.x;
  cell_size.y = volume_size.y / buffer->voxels_size.y;
  cell_size.z = volume_size.z / buffer->voxels_size.z;

  //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
  dim3 block (16, 16);
  dim3 grid (divUp (buffer->voxels_size.x, block.x), divUp (buffer->voxels_size.y, block.y));

  //tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, *buffer);    
  tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, *buffer, vxlDbg);    

//  for ( int i = 0; i < 100; i++ )
//    tsdf23test<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, *buffer);    

  //tsdf23normal_hack<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);

  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

void
pcl::device::integrateTsdfVolume_s2s (/*const PtrStepSz<ushort>& depth,*/ const Intr& intr,
    const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, float eta,
    PtrStep<short2> volume, DeviceArray2D<float>& depthScaled, int3 vxlDbg) //zc: ����
{
    //depthScaled.create (depth.rows, depth.cols);

    //dim3 block_scale (32, 8);
    //dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

    ////scales depth along ray and converts mm -> meters. 
    //scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
    //cudaSafeCall ( hipGetLastError () );

    float3 cell_size;
    cell_size.x = volume_size.x / VOLUME_X;
    cell_size.y = volume_size.y / VOLUME_Y;
    cell_size.z = volume_size.z / VOLUME_Z;

    //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
    dim3 block (16, 16);
    dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

    //tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, *buffer);    
    tsdf23_s2s<<<grid, block>>>(depthScaled, volume, tranc_dist, eta,
        Rcurr_inv, tcurr, intr, cell_size, vxlDbg);    

    //  for ( int i = 0; i < 100; i++ )
    //    tsdf23test<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, *buffer);    

    //tsdf23normal_hack<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);

    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
}//integrateTsdfVolume_s2s

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void 
pcl::device::integrateTsdfVolume_v11 (const PtrStepSz<ushort>& depth, const Intr& intr, const float3& volume_size, 
    const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, PtrStep<short2> volume, 
    PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, DeviceArray2D<unsigned char> incidAngleMask, 
    const MapArr& nmap_curr_g, const MapArr &nmap_model_g,
    const MapArr &weight_map, //v11.4
    DeviceArray2D<float>& depthScaled, int3 vxlDbg)
{
    depthScaled.create (depth.rows, depth.cols);

    dim3 block_scale (32, 8);
    dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

    //scales depth along ray and converts mm -> meters. 
    scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
    cudaSafeCall ( hipGetLastError () );

    float3 cell_size;
    cell_size.x = volume_size.x / VOLUME_X;
    cell_size.y = volume_size.y / VOLUME_Y;
    cell_size.z = volume_size.z / VOLUME_Z;

    //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
    dim3 block (16, 16);
    dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

    printf("vxlDbg@integrateTsdfVolume_v11: [%d, %d, %d]\n", vxlDbg.x, vxlDbg.y, vxlDbg.z);

    //tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);    
    //tsdf23_v11<<<grid, block>>>(depthScaled, volume, 
    tsdf23_v11_remake<<<grid, block>>>(depthScaled, volume, 
        volume2nd, flagVolume, surfNormVolume, vrayPrevVolume, incidAngleMask, 
        nmap_curr_g, nmap_model_g,
        weight_map,
        tranc_dist, Rcurr_inv, tcurr, intr, cell_size, vxlDbg);    
}//integrateTsdfVolume_v11

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void 
pcl::device::integrateTsdfVolume_v12 (const PtrStepSz<ushort>& depth, const Intr& intr, const float3& volume_size, 
    const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, PtrStep<short2> volume, 
    PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, DeviceArray2D<unsigned char> incidAngleMask, 
    const MapArr& nmap_curr_g, const MapArr &nmap_model_g,
    const MapArr &weight_map, //v11.4
    const PtrStepSz<ushort>& depth_model,
    DeviceArray2D<short>& diffDmap,
    DeviceArray2D<float>& depthScaled, int3 vxlDbg)
{
    depthScaled.create (depth.rows, depth.cols);

    dim3 block_scale (32, 8);
    dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

    //scales depth along ray and converts mm -> meters. 
    scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
    cudaSafeCall ( hipGetLastError () );

    //v12 ��һ��: �� diffDmap = depth(raw)-depth_model @2017-12-3 22:06:24
    //DeviceArray2D<short> diffDmap; //short, ���� ushort
    //��--�ֲ������ᵼ��: Error: unspecified launch failure       ..\..\..\gpu\containers\src\device_memory.cpp:276 //��: DeviceMemory2D::release() ����
    diffDmap.create(depth.rows, depth.cols);
    diffDmaps(depth, depth_model, diffDmap); //�� mm


    float3 cell_size;
    cell_size.x = volume_size.x / VOLUME_X;
    cell_size.y = volume_size.y / VOLUME_Y;
    cell_size.z = volume_size.z / VOLUME_Z;

    //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
    dim3 block (16, 16);
    dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

    printf("vxlDbg@integrateTsdfVolume_v12: [%d, %d, %d]\n", vxlDbg.x, vxlDbg.y, vxlDbg.z);

    //tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);    
    //tsdf23_v11<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v11_remake<<<grid, block>>>(depthScaled, volume, 
    tsdf23_v12<<<grid, block>>>(depthScaled, volume, 
         volume2nd, flagVolume, surfNormVolume, vrayPrevVolume, incidAngleMask, 
         nmap_curr_g, nmap_model_g,
         weight_map,
         diffDmap,
         tranc_dist, Rcurr_inv, tcurr, intr, cell_size, vxlDbg);    
}//integrateTsdfVolume_v12


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void 
pcl::device::integrateTsdfVolume_v13 (const PtrStepSz<ushort>& depth, const Intr& intr, const float3& volume_size, 
    const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, PtrStep<short2> volume, 
    PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, DeviceArray2D<unsigned char> incidAngleMask, 
    const MapArr& nmap_curr_g, const MapArr &nmap_model_g,
    const MapArr &weight_map, //v11.4
    const PtrStepSz<ushort>& depth_model,
    DeviceArray2D<short>& diffDmap,
    DeviceArray2D<float>& depthScaled, int3 vxlDbg)
{
    depthScaled.create (depth.rows, depth.cols);

    dim3 block_scale (32, 8);
    dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

    //scales depth along ray and converts mm -> meters. 
    scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
    cudaSafeCall ( hipGetLastError () );

    //v12 ��һ��: �� diffDmap = depth(raw)-depth_model @2017-12-3 22:06:24
    //DeviceArray2D<short> diffDmap; //short, ���� ushort
    //��--�ֲ������ᵼ��: Error: unspecified launch failure       ..\..\..\gpu\containers\src\device_memory.cpp:276 //��: DeviceMemory2D::release() ����
    diffDmap.create(depth.rows, depth.cols);
    diffDmaps(depth, depth_model, diffDmap); //�� mm

    float3 cell_size;
    cell_size.x = volume_size.x / VOLUME_X;
    cell_size.y = volume_size.y / VOLUME_Y;
    cell_size.z = volume_size.z / VOLUME_Z;

    //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
    dim3 block (16, 16);
    dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

    printf("vxlDbg@integrateTsdfVolume_v13: [%d, %d, %d]\n", vxlDbg.x, vxlDbg.y, vxlDbg.z);

    //tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);    
    //tsdf23_v11<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v13<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v14<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v15<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v16<<<grid, block>>>(depthScaled, volume,  //���� tranc_dist_real �õ�
    tsdf23_v17<<<grid, block>>>(depthScaled, volume,  //���� tdist, ��������� tdist
        volume2nd, flagVolume, surfNormVolume, vrayPrevVolume, incidAngleMask, 
        nmap_curr_g, nmap_model_g,
        weight_map,
        diffDmap,
        tranc_dist, Rcurr_inv, tcurr, intr, cell_size, vxlDbg);    
}//integrateTsdfVolume_v13

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void 
pcl::device::integrateTsdfVolume_v18 (const PtrStepSz<ushort>& depth, const Intr& intr, const float3& volume_size, 
    const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, PtrStep<short2> volume, 
    PtrStep<short2> volume2nd, PtrStep<bool> flagVolume, PtrStep<char4> surfNormVolume, PtrStep<char4> vrayPrevVolume, DeviceArray2D<unsigned char> incidAngleMask, 
    const MapArr& nmap_curr_g, const MapArr &nmap_model_g,
    const MapArr &weight_map, //v11.4
    const PtrStepSz<ushort>& depth_model,
    DeviceArray2D<short>& diffDmap,
    DeviceArray2D<float>& depthScaled, int3 vxlDbg)
{
    depthScaled.create (depth.rows, depth.cols);

    dim3 block_scale (32, 8);
    dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

    //scales depth along ray and converts mm -> meters. 
    scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
    cudaSafeCall ( hipGetLastError () );

    //v12 ��һ��: �� diffDmap = depth(raw)-depth_model @2017-12-3 22:06:24
    //DeviceArray2D<short> diffDmap; //short, ���� ushort
    //��--�ֲ������ᵼ��: Error: unspecified launch failure       ..\..\..\gpu\containers\src\device_memory.cpp:276 //��: DeviceMemory2D::release() ����
    diffDmap.create(depth.rows, depth.cols);
    diffDmaps(depth, depth_model, diffDmap); //�� mm

    float3 cell_size;
    cell_size.x = volume_size.x / VOLUME_X;
    cell_size.y = volume_size.y / VOLUME_Y;
    cell_size.z = volume_size.z / VOLUME_Z;

    //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
    dim3 block (16, 16);
    dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

    printf("vxlDbg@integrateTsdfVolume_v18: [%d, %d, %d]\n", vxlDbg.x, vxlDbg.y, vxlDbg.z);

    //tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);    
    //tsdf23_v11<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v13<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v14<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v15<<<grid, block>>>(depthScaled, volume, 
    //tsdf23_v16<<<grid, block>>>(depthScaled, volume,  //���� tranc_dist_real �õ�
    //tsdf23_v17<<<grid, block>>>(depthScaled, volume,  //���� tdist, ��������� tdist
    //test_kernel<<<grid, block>>>(vxlDbg); //v18.2
    tsdf23_v18<<<grid, block>>>(depthScaled, volume,  
        volume2nd, flagVolume, surfNormVolume, vrayPrevVolume, incidAngleMask, 
        nmap_curr_g, nmap_model_g,
        weight_map,
        depth_model, //v18.5, �����β�, ��Ҫ�ж� isnan
        diffDmap,
        tranc_dist, Rcurr_inv, tcurr, intr, cell_size, vxlDbg);    
}//integrateTsdfVolume_v18


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void 
pcl::device::clearTSDFSlice (PtrStep<short2> volume, pcl::gpu::tsdf_buffer* buffer, int shiftX, int shiftY, int shiftZ)
{
    int newX = buffer->origin_GRID.x + shiftX;
    int newY = buffer->origin_GRID.y + shiftY;

    int3 minBounds, maxBounds;
    
	/*
    //X
    if(newX >= 0)
    {
     minBounds.x = buffer->origin_GRID.x;
     maxBounds.x = newX;    
    }
    else
    {
     minBounds.x = newX + buffer->voxels_size.x; 
     maxBounds.x = buffer->origin_GRID.x + buffer->voxels_size.x;
    }
    
    if(minBounds.x > maxBounds.x)
     std::swap(minBounds.x, maxBounds.x);
      
   
    //Y
    if(newY >= 0)
    {
     minBounds.y = buffer->origin_GRID.y;
     maxBounds.y = newY;
    }
    else
    {
     minBounds.y = newY + buffer->voxels_size.y; 
     maxBounds.y = buffer->origin_GRID.y + buffer->voxels_size.y;
    }
    
    if(minBounds.y > maxBounds.y)
     std::swap(minBounds.y, maxBounds.y);
	 */
	if ( shiftX >= 0 ) {
		minBounds.x = buffer->origin_GRID.x;
		maxBounds.x = newX - 1;
	} else {
		minBounds.x = newX;
		maxBounds.x = buffer->origin_GRID.x - 1;
	}
	if ( minBounds.x < 0 ) {
		minBounds.x += buffer->voxels_size.x;
		maxBounds.x += buffer->voxels_size.x;
	}

	if ( shiftY >= 0 ) {
		minBounds.y = buffer->origin_GRID.y;
		maxBounds.y = newY - 1;
	} else {
		minBounds.y = newY;
		maxBounds.y = buffer->origin_GRID.y - 1;
	}
	if ( minBounds.y < 0 ) {
		minBounds.y += buffer->voxels_size.y;
		maxBounds.y += buffer->voxels_size.y;
	}
    //Z
     minBounds.z = buffer->origin_GRID.z;
     maxBounds.z = shiftZ;
  
    // call kernel
    dim3 block (32, 16);
    dim3 grid (1, 1, 1);
    grid.x = divUp (buffer->voxels_size.x, block.x);      
    grid.y = divUp (buffer->voxels_size.y, block.y);
    
    clearSliceKernel<<<grid, block>>>(volume, *buffer, minBounds, maxBounds);
    cudaSafeCall ( hipGetLastError () );
    cudaSafeCall (hipDeviceSynchronize ());
   
}//clearTSDFSlice

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//������ contour_cue_impl.cu, ��Ϊ�ֶ��޷���ӵ��˹���, �������� cmake, ���»���; ����ֱ��Դ�뿽��
namespace zc{

//@brief gpu kernel function to generate the Contour-Correspondence-Candidates
//@param[in] angleThreshCos, MAX cosine of the angle threshold
//@ע�� kernel ������������Ϊ GPU �ڴ�ָ�����󿽱���e.g., ����Ϊ float3 ���� float3&
__global__ void 
cccKernel(const float3 camPos, const PtrStep<float> vmap, const PtrStep<float> nmap, float angleThreshCos, PtrStepSz<_uchar> outMask){
    int x = threadIdx.x + blockIdx.x * blockDim.x,
        y = threadIdx.y + blockIdx.y * blockDim.y;
    //printf("### %d, %d\n", x, y);

    int cols = outMask.cols,
        rows = outMask.rows;

    if(!(x < cols && y < rows))
        return;

    outMask.ptr(y)[x] = 0;

    if(isnan(nmap.ptr(y)[x]) || isnan(vmap.ptr(y)[x])){
        //printf("\tisnan: %d, %d\n", x, y);
        return;
    }

    float3 n, vRay;
    n.x = nmap.ptr(y)[x];
    n.y = nmap.ptr(y + rows)[x];
    n.z = nmap.ptr(y + 2 * rows)[x];

    vRay.x = camPos.x - vmap.ptr(y)[x];
    vRay.y = camPos.y - vmap.ptr(y + rows)[x];
    vRay.z = camPos.z - vmap.ptr(y + 2 * rows)[x];

    double nMod = norm(n); //�����Ϻ����1��
    double vRayMod = norm(vRay);
    //printf("@@@ %f, %f\n", nMod, vRayMod);

    double cosine = dot(n, vRay) / (vRayMod * nMod);
    if(abs(cosine) < angleThreshCos)
        outMask.ptr(y)[x] = UCHAR_MAX;
}//cccKernel

void contourCorrespCandidate(const float3 &camPos, const MapArr &vmap, const MapArr &nmap, int angleThresh, pcl::device::MaskMap &outMask ){
    int cols = vmap.cols();
    int rows = vmap.rows() / 3;
    
    outMask.create(rows, cols);

    dim3 block(32, 8);
    dim3 grid(divUp(cols, block.x), divUp(rows, block.y));

    const float angleThreshCos = cos(angleThresh * 3.14159265359f / 180.f);
    //printf("vmap, nmap shape: [%d, %d], [%d, %d]\n", vmap.rows(), vmap.cols(), nmap.rows(), nmap.cols()); //test OK
    cccKernel<<<grid, block>>>(camPos, vmap, nmap, angleThreshCos, outMask);

    cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize()); //tmp, ��ʱ��ͼ�������� @2017-12-6 22:03:13
}//contourCorrespCandidate

__global__ void
calcWmapKernel(int rows, int cols, const PtrStep<float> vmapLocal, const PtrStep<float> nmapLocal, const PtrStepSz<_uchar> contMask, PtrStepSz<float> wmap_out){
    int x = threadIdx.x + blockIdx.x * blockDim.x,
        y = threadIdx.y + blockIdx.y * blockDim.y;

    const float qnan = pcl::device::numeric_limits<float>::quiet_NaN();
    if(!(x < cols && y < rows))
        return;

    //������:
    bool doDbgPrint = false;
//     if(x == 388 && y == 292)
//         doDbgPrint = true;

    wmap_out.ptr(y)[x] = 0; //Ĭ�ϳ�ʼȨ��=0

    float3 vray; //local
    vray.x = vmapLocal.ptr(y)[x];
    if(isnan(vray.x))
        return;

    vray.y = vmapLocal.ptr(y + rows)[x];
    vray.z = vmapLocal.ptr(y + 2 * rows)[x]; //meters

    float3 snorm;
    snorm.x = nmapLocal.ptr(y)[x];
    snorm.y = nmapLocal.ptr(y + rows)[x];
    snorm.z = nmapLocal.ptr(y + 2 * rows)[x];

    //���費ȷ�� normalize ��: Ҫ��һ��, ��ȷ�� snorm �����ӵ�: Ҫ abs
    float cosine = dot(vray, snorm) / (norm(snorm) * norm(vray));
    cosine = abs(cosine);

#if 0   //v0: KinectFusion �����ᵽ�� "������...", �ֲ�
    //wmap_out.ptr(y)[x] = cosine * zmin / max(zmin, vray.z); //���Ų��ܿ���, ��Χ���ڹ� ( cos->0, z->+inf ); ������� minXXfactor Լ�� ��
#elif 10 //v1: �п������ŷ�Χ
    const float minCosFactor = .5f; //cos min��������, ���� 90��, ����Ҳ�� 1/2, ����̫С
    const float cosMin = 0.5f; //60��, �� theta<60��, �������ӹ̶�Ϊ 1, ����ȫ������� 0~60��ʱ�����ֵ
    float cosFactor = 1;
    if(cosine < cosMin)
        cosFactor = 1 - (1 - 2 * cosine) * (1 - minCosFactor) / 1; //�����ĸ�� 1= (1-0)

    const float minZfactor = .5f; //���ֵ min��������
    const float zmin = 0.5f,
                zmax = 3.f; //meters, zmax �˴�����������޶������Ч���, ֻ��ȷ�� zmax ��, ����Ϊ minZfactor (ԭ������=1/6)

    float oldMinZfactor = zmin / zmax;
    //float zFactor = 1 - (1 - vray.z) * (1 - minZfactor)/ (1 - rawMinZfactor); //��
    float zFactor = zmin / min(zmax, max(zmin, vray.z)); //1/6 <= factor <= 1
    //��--[1/6, 1] -> [.5, 1]
    zFactor = 1 - (1 - zFactor) * (1 - minZfactor) / (1 - oldMinZfactor);

    float contFactor = 1;
    if(contMask.ptr(y)[x] != 0) //��Ե��Ȩ, ��ֹ����
        contFactor = 0.3f;

    wmap_out.ptr(y)[x] = cosFactor * zFactor * contFactor;
#endif 

}//calcWmapKernel

//@brief v2, ֮ǰ contMask ����������Ȩ�� mask, �ĳɸ�����, ƽ������Ȩ�� (������ edgeDistMap)
//@param[in] edgeDistMap, ����Ե���ؾ��� mat: ֵԽС,���ԵԽ��, tsdfȨ���Լ�tsdf�ض���ֵԽС; ��Ҫ��� vmap.z ת��������߶Ⱦ���,
//@param[in] fxy, ��һ������Լ��, �������� 
__global__ void
calcWmapKernel(int rows, int cols, const PtrStep<float> vmapLocal, const PtrStep<float> nmapLocal, const PtrStepSz<float> edgeDistMap, float fxy, PtrStepSz<float> wmap_out){
    int x = threadIdx.x + blockIdx.x * blockDim.x,
        y = threadIdx.y + blockIdx.y * blockDim.y;

    const float qnan = pcl::device::numeric_limits<float>::quiet_NaN();
    if(!(x < cols && y < rows))
        return;

    //������:
    bool doDbgPrint = false;
//     if(x == 388 && y == 292)
//         doDbgPrint = true;

    wmap_out.ptr(y)[x] = 0; //Ĭ�ϳ�ʼȨ��=0

    float3 vray; //local
    vray.x = vmapLocal.ptr(y)[x];
    if(isnan(vray.x))
        return;

    vray.y = vmapLocal.ptr(y + rows)[x];
    vray.z = vmapLocal.ptr(y + 2 * rows)[x]; //meters

    float3 snorm;
    snorm.x = nmapLocal.ptr(y)[x];
    snorm.y = nmapLocal.ptr(y + rows)[x];
    snorm.z = nmapLocal.ptr(y + 2 * rows)[x];

    //���費ȷ�� normalize ��: Ҫ��һ��, ��ȷ�� snorm �����ӵ�: Ҫ abs
    float cosine = dot(vray, snorm) / (norm(snorm) * norm(vray));
    cosine = abs(cosine); //ȡ���

#if 0   //v0: KinectFusion �����ᵽ�� "������...", �ֲ�
    //wmap_out.ptr(y)[x] = cosine * zmin / max(zmin, vray.z); //���Ų��ܿ���, ��Χ���ڹ� ( cos->0, z->+inf ); ������� minXXfactor Լ�� ��
#elif 10 //v1: �п������ŷ�Χ
    const float minCosFactor = .3f; //cos min��������, ���� 90��, ����Ҳ�� 1/2, ����̫С
    const float cosMin = 0.5f; //60��, �� theta<60��, �������ӹ̶�Ϊ 1, ����ȫ������� 0~60��ʱ�����ֵ
    float cosFactor = 1;
    if(cosine < cosMin) //ȷ����Ҫ cos >1
        cosFactor = 1 - (1 - 2 * cosine) * (1 - minCosFactor) / 1; //�����ĸ�� 1= (1-0)

    const float minZfactor = .5f; //���ֵ min��������
    const float zmin = 0.5f,
                zmax = 3.f; //meters, zmax �˴�����������޶������Ч���, ֻ��ȷ�� zmax ��, ����Ϊ minZfactor (ԭ������=1/6)

    float oldMinZfactor = zmin / zmax;
    //float zFactor = 1 - (1 - vray.z) * (1 - minZfactor)/ (1 - rawMinZfactor); //��
    float zFactor = zmin / min(zmax, max(zmin, vray.z)); //1/6 <= factor <= 1
    //��--[1/6, 1] -> [.5, 1]
    zFactor = 1 - (1 - zFactor) * (1 - minZfactor) / (1 - oldMinZfactor);

#if 0   //contMask ��������
    float contFactor = 1;
    if(contMask.ptr(y)[x] != 0) //��Ե��Ȩ, ��ֹ����
        contFactor = 0.3f;

    wmap_out.ptr(y)[x] = cosFactor * zFactor * contFactor;
#elif 1 //edgeDistMap ��������
    const float maxEdgeDist = 30; //in mm
    float edgeDistMm = edgeDistMap.ptr(y)[x] / fxy * vray.z * 1e3; //in mm

    float edgeDistFactor = 1.f;
    if(edgeDistMm < maxEdgeDist) //������ 1
        edgeDistFactor = edgeDistMm / maxEdgeDist;

    wmap_out.ptr(y)[x] = cosFactor * zFactor * edgeDistFactor;
#endif

#endif //�������޿���


}//calcWmapKernel-v2

//@param[in] vmapLocal, ��ʵֻҪ�� dmap ����, �ݲ���, ��֮ǰ calcWmapKernel ����һ��
__global__ void
edge2wmapKernel(int rows, int cols, const PtrStep<float> vmapLocal, const PtrStepSz<float> edgeDistMap, float fxy, PtrStepSz<float> wmap_out){
    int x = threadIdx.x + blockIdx.x * blockDim.x,
        y = threadIdx.y + blockIdx.y * blockDim.y;

    const float qnan = pcl::device::numeric_limits<float>::quiet_NaN();
    if(!(x < cols && y < rows))
        return;

    wmap_out.ptr(y)[x] = 0; //Ĭ�ϳ�ʼȨ��=0

    float3 vray; //local
    vray.x = vmapLocal.ptr(y)[x];
    if(isnan(vray.x))
        return;

    vray.y = vmapLocal.ptr(y + rows)[x];
    vray.z = vmapLocal.ptr(y + 2 * rows)[x]; //meters

    const float maxEdgeDist = 10; //in mm //30mm ̫��
    float edgeDistMm = edgeDistMap.ptr(y)[x] / fxy * vray.z * 1e3; //in mm

    float edgeDistFactor = min(1.f, edgeDistMm / maxEdgeDist);
    wmap_out.ptr(y)[x] = edgeDistFactor;
}//edge2wmapKernel

void calcWmap(const MapArr &vmapLocal, const MapArr &nmapLocal, const pcl::device::MaskMap &contMask, MapArr &wmap_out){
    int cols = vmapLocal.cols(),
        rows = vmapLocal.rows() / 3;

    wmap_out.create(rows, cols);

    dim3 block(32, 8);
    dim3 grid(divUp(cols, block.x), divUp(rows, block.y));

    calcWmapKernel<<<grid, block>>>(rows, cols, vmapLocal, nmapLocal, contMask, wmap_out);
    
    cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize()); //tmp, ��ʱ��ͼ�������� @2017-12-6 22:03:13
}//calcWmap

void calcWmap(const MapArr &vmapLocal, const MapArr &nmapLocal, const DeviceArray2D<float> &edgeDistMap, const float fxy, MapArr &wmap_out){
    int cols = vmapLocal.cols(),
        rows = vmapLocal.rows() / 3;

    wmap_out.create(rows, cols);

    dim3 block(32, 8);
    dim3 grid(divUp(cols, block.x), divUp(rows, block.y));

    calcWmapKernel<<<grid, block>>>(rows, cols, vmapLocal, nmapLocal, edgeDistMap, fxy, wmap_out);
    
    cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize()); //tmp, ��ʱ��ͼ�������� @2017-12-6 22:03:13
}//calcWmap

void edge2wmap(const MapArr &vmapLocal, const DeviceArray2D<float> &edgeDistMap, const float fxy, MapArr &wmap_out){
    int cols = vmapLocal.cols(),
        rows = vmapLocal.rows() / 3;

    wmap_out.create(rows, cols);

    dim3 block(32, 8);
    dim3 grid(divUp(cols, block.x), divUp(rows, block.y));

    edge2wmapKernel<<<grid, block>>>(rows, cols, vmapLocal, edgeDistMap, fxy, wmap_out);

    cudaSafeCall(hipGetLastError());

}//edge2wmap

__global__ void
transformVmapKernel(int rows, int cols, const PtrStep<float> vmap_src, const Mat33 Rmat, const float3 tvec, PtrStepSz<float> vmap_dst){
    int x = threadIdx.x + blockIdx.x * blockDim.x,
        y = threadIdx.y + blockIdx.y * blockDim.y;

    const float qnan = pcl::device::numeric_limits<float>::quiet_NaN();
    if(!(x < cols && y < rows))
        return;

    float3 vsrc, vdst = make_float3(qnan, qnan, qnan);
    vsrc.x = vmap_src.ptr(y)[x];

    if(!isnan(vsrc.x)){
        vsrc.y = vmap_src.ptr(y + rows)[x];
        vsrc.z = vmap_src.ptr(y + 2 * rows)[x];

        vdst = Rmat * vsrc + tvec;

        vmap_dst.ptr (y + rows)[x] = vdst.y;
        vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
    }

    //ȷʵӦ������������Ƿ� isnan(vdst.x)
    vmap_dst.ptr(y)[x] = vdst.x;
}//transformVmapKernel

void transformVmap( const MapArr &vmap_src, const Mat33 &Rmat, const float3 &tvec, MapArr &vmap_dst ){
    int cols = vmap_src.cols(),
        rows = vmap_src.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    
    dim3 block(32, 8);
    dim3 grid(divUp(cols, block.x), divUp(rows, block.y));

    transformVmapKernel<<<grid, block>>>(rows, cols, vmap_src, Rmat, tvec, vmap_dst);

    cudaSafeCall(hipGetLastError());
    //cudaSafeCall(hipDeviceSynchronize()); //tmp, ��ʱ��ͼ�������� @2017-12-6 22:03:13
}//transformVmap

}//namespace zc