#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 * 
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

//#include <pcl/gpu/utils/device/block.hpp>
//#include <pcl/gpu/utils/device/funcattrib.hpp>
#include "device.hpp"

namespace pcl
{
  namespace device
  {
    //typedef double float_type;
	typedef float float_type;

    struct Combined
    {
      enum
      {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
      };

      struct plus
      {
        __forceinline__ __device__ float
        operator () (const float_type &lhs, const volatile float_type& rhs) const 
        {
          return (lhs + rhs);
        }
      };

      Mat33 Rcurr;
      float3 tcurr;

      PtrStep<float> vmap_curr;
      PtrStep<float> nmap_curr;

      Mat33 Rprev_inv;
      float3 tprev;

      Intr intr;

      PtrStep<float> vmap_g_prev;
      PtrStep<float> nmap_g_prev;

      float distThres;
      float angleThres;

      int cols;
      int rows;

      mutable PtrStep<float_type> gbuf;

      __device__ __forceinline__ bool
      search (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

        float row[7];

        if (found_coresp)
        {
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        __shared__ float_type smem[CTA_SIZE];
        int tid = Block::flattenedThreadId ();

        int shift = 0;
        #pragma unroll
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
            __syncthreads ();
            smem[tid] = row[i] * row[j];
            __syncthreads ();

            Block::reduce<CTA_SIZE>(smem, plus ());

            if (tid == 0)
              gbuf.ptr (shift++)[blockIdx.x + gridDim.x * blockIdx.y] = smem[0];
          }
        }
      }
    };

    __global__ void
    combinedKernel (const Combined cs) 
    {
      cs ();
    }

    struct TranformReduction
    {
      enum
      {
        CTA_SIZE = 512,
        STRIDE = CTA_SIZE,

        B = 6, COLS = 6, ROWS = 6, DIAG = 6,
        UPPER_DIAG_MAT = (COLS * ROWS - DIAG) / 2 + DIAG,
        TOTAL = UPPER_DIAG_MAT + B,

        GRID_X = TOTAL
      };

      PtrStep<float_type> gbuf;
      int length;
      mutable float_type* output;

      __device__ __forceinline__ void
      operator () () const
      {
        const float_type *beg = gbuf.ptr (blockIdx.x);
        const float_type *end = beg + length;

        int tid = threadIdx.x;

        float_type sum = 0.f;
        for (const float_type *t = beg + tid; t < end; t += STRIDE)
          sum += *t;

        __shared__ float_type smem[CTA_SIZE];

        smem[tid] = sum;
        __syncthreads ();

		Block::reduce<CTA_SIZE>(smem, Combined::plus ());

        if (tid == 0)
          output[blockIdx.x] = smem[0];
      }
    };

    __global__ void
    TransformEstimatorKernel2 (const TranformReduction tr) 
    {
      tr ();
    }

    struct Combined2
    {
      enum
      {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
      };

      struct plus
      {
        __forceinline__ __device__ float
        operator () (const float_type &lhs, const volatile float_type& rhs) const 
        {
          return (lhs + rhs);
        }
      };

      Mat33 Rcurr;
      float3 tcurr;

      PtrStep<float> vmap_curr;
      PtrStep<float> nmap_curr;

      Mat33 Rprev_inv;
      float3 tprev;

      Intr intr;

      PtrStep<float> vmap_g_prev;
      PtrStep<float> nmap_g_prev;

      float distThres;
      float angleThres;

      int cols;
      int rows;

      mutable PtrStep<float_type> gbuf;

      __device__ __forceinline__ bool
      search (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];

        //zc: fix @2017-4-13 16:20:12
        if (isnan (vprev_g.x))
          return (false);

        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
      }

      __device__ __forceinline__ bool
      searchDbg (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

		//zc: dbg
		printf("\t@searchDbg: ukr.xy=(%d, %d); isnan(nprev_g.x): %d; isnan (vprev_g.x): %d\n", ukr.x, ukr.y, isnan(nprev_g.x), isnan(vprev_g.x));

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
      }//searchDbg

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

#if 0	//zc: dbg
		//if(x == 320 && y == 240){ //��
		if(x == cols/2 && y == rows/2){
			printf("@operator():: (x, y)=(%d, %d), found_coresp= %d; n=(%f, %f, %f), d=(%f, %f, %f), s=(%f, %f, %f)\n", x, y, 
				found_coresp, n.x, n.y, n.z, d.x, d.y, d.z, s.x, s.y, s.z);
		}
#endif

        float row[7];

        if (found_coresp)
        {
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
		  //zc: dbg
		  if(isnan(row[6])){ //��������ȫ��Ӧ�÷�������
			  printf("isnan(row[6]), (x,y)=(%d, %d); (rows, cols)=(%d, %d); n=(%f, %f, %f), d=(%f, %f, %f), s=(%f, %f, %f)\n", x, y, rows, cols,
				  n.x, n.y, n.z, d.x, d.y, d.z, s.x, s.y, s.z);
			  searchDbg(x, y, n, d, s);
		  }

#if 0	//����, ���ܼӵ�һ��, ��Ϊ��ͷ�������� ������С���� ��ʽ, �Ƿ������� @2017-6-1 11:06:13
		  //zc: ������ʦҪ��, ���� nmap ���ͷ���, //��ֻ�ܳͷ� R, ���� t @2017-5-31 11:16:49
		  //Ӱ�� row[0~2, 6], ��Ӱ�� row[3~5]
		  float3 ncurr;
		  ncurr.x = nmap_curr.ptr (y)[x];
		  ncurr.y = nmap_curr.ptr (y + rows)[x];
		  ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];
		  
		  float3 ncurr_g = Rcurr * ncurr;
		  if(dot(ncurr_g, n) < 0) //�жϷ���, ϣ���� nprev_g ����һ��
			  ncurr_g *= -1;

		  //ע��: n �� nprev_g 
		  float3 tmpv = ncurr_g - n;
		  *(float3*)&row[0] = *(float3*)&row[0] + cross(ncurr_g, tmpv); //3x1 ����
		  row[6] = row[6] - dot(tmpv, tmpv); //�ٱ��� ��ע������ ��-=��, ��ԭ��, �Ƶ���
#endif

#if 0
		  {
			  float3 cross_ng_v = cross(ncurr_g, tmpv);
			  float3 row03 = *(float3*)&row[0];
			  float3 row03_new = row03 + cross_ng_v;
			  //printf("ncurr_g=(%f, %f, %f), nprev_g=(%f, %f, %f)\n", ncurr_g.x, ncurr_g.y, ncurr_g.z, n.x, n.y, n.z);
			  printf("ncurr_g=(%f, %f, %f), nprev_g=(%f, %f, %f)\
					 \ntmpv=(%f, %f, %f), row03=(%f, %f, %f), cross_ng_v=(%f, %f, %f), row03_new=(%f, %f, %f), row6=%f, row6_new=%f\n", 
					 ncurr_g.x, ncurr_g.y, ncurr_g.z, n.x, n.y, n.z,
				  tmpv.x, tmpv.y, tmpv.z, 
				  row03.x, row03.y, row03.z,
				  cross_ng_v.x, cross_ng_v.y, cross_ng_v.z, 
				  row03_new.x, row03_new.y, row03_new.z, 
				  row[6], row[6] - dot(tmpv, tmpv));

		  }
#endif
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        int tid = Block::flattenedThreadId ();

        int shift = 0;
        #pragma unroll
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
              gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[i]*row[j];
          }
        }
      }

      __device__ __forceinline__ void
      operator () (int dummy) const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

#if 0	//zc: dbg
		//if(x == 320 && y == 240){ //��
		if(x == cols/2 && y == rows/2){
			printf("@operator():: (x, y)=(%d, %d), found_coresp= %d; n=(%f, %f, %f), d=(%f, %f, %f), s=(%f, %f, %f)\n", x, y, 
				found_coresp, n.x, n.y, n.z, d.x, d.y, d.z, s.x, s.y, s.z);
		}
#endif

        float row[7];

        if (found_coresp)
        {
#if 0	//��, ����Ҫ�� nmap �ͷ���, ���Ż� R, ���� t (ϵ������) @2017-6-1 14:47:31
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
#elif 1
          float3 ncurr;
          ncurr.x = nmap_curr.ptr (y)[x];
          ncurr.y = nmap_curr.ptr (y + rows)[x];
          ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];
          
          float3 ncurr_g = Rcurr * ncurr;
          if(dot(ncurr_g, n) < 0) //�жϷ���, ϣ���� nprev_g ����һ��
              ncurr_g *= -1;

          //ע��: n �� nprev_g 
#if 0	//�˴�˼·�� argmin(SUM(|(R*ng~-ng)*(ng~-ng)|))
          //������, ����, ���� @2017-6-2 17:48:13
          float3 tmpv = ncurr_g - n;
          *(float3*)&row[0] = cross(ncurr_g, tmpv); //3x1 ����
          row[3] = row[4] = row[5] = 0.f;
          row[6] = -dot(tmpv, tmpv); //�ٱ��� ��ע������ ��-=��, ��ԭ��, �Ƶ���

#elif 1	//������ʵ���� orthogonal-procrustes ����, ���ﳢ�Բ��л����� @2017-6-2 17:48:49
          //Ŀ��: argmin|RA-B| ==> R = svd(B*At), ���� A/B �� 3*N, �� BAt~3x3
          //row0~2 -> ncurr_g, 3~5-> nprev_g, [6]����, ������
          //֮�� gbuf[27] ֻ��ǰ 3x3=9 ��, 
          *(float3*)&row[0] = ncurr_g;
          *(float3*)&row[3] = n;
          row[6] = 0;
#endif

#endif
		  //zc: dbg
		  if(isnan(row[6])){ //��������ȫ��Ӧ�÷�������
			  printf("isnan(row[6]), (x,y)=(%d, %d); (rows, cols)=(%d, %d); n=(%f, %f, %f), d=(%f, %f, %f), s=(%f, %f, %f)\n", x, y, rows, cols,
				  n.x, n.y, n.z, d.x, d.y, d.z, s.x, s.y, s.z);
			  searchDbg(x, y, n, d, s);
		  }

#if 0	//����, ���ܼӵ�һ��, ��Ϊ��ͷ�������� ������С���� ��ʽ, �Ƿ������� @2017-6-1 11:06:13
		  //zc: ������ʦҪ��, ���� nmap ���ͷ���, //��ֻ�ܳͷ� R, ���� t @2017-5-31 11:16:49
		  //Ӱ�� row[0~2, 6], ��Ӱ�� row[3~5]
		  float3 ncurr;
		  ncurr.x = nmap_curr.ptr (y)[x];
		  ncurr.y = nmap_curr.ptr (y + rows)[x];
		  ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];
		  
		  float3 ncurr_g = Rcurr * ncurr;
		  if(dot(ncurr_g, n) < 0) //�жϷ���, ϣ���� nprev_g ����һ��
			  ncurr_g *= -1;

		  //ע��: n �� nprev_g 
		  float3 tmpv = ncurr_g - n;
		  *(float3*)&row[0] = *(float3*)&row[0] + cross(ncurr_g, tmpv); //3x1 ����
		  row[6] = row[6] - dot(tmpv, tmpv); //�ٱ��� ��ע������ ��-=��, ��ԭ��, �Ƶ���
#endif

#if 0
		  {
			  float3 cross_ng_v = cross(ncurr_g, tmpv);
			  float3 row03 = *(float3*)&row[0];
			  float3 row03_new = row03 + cross_ng_v;
			  //printf("ncurr_g=(%f, %f, %f), nprev_g=(%f, %f, %f)\n", ncurr_g.x, ncurr_g.y, ncurr_g.z, n.x, n.y, n.z);
			  printf("ncurr_g=(%f, %f, %f), nprev_g=(%f, %f, %f)\
					 \ntmpv=(%f, %f, %f), row03=(%f, %f, %f), cross_ng_v=(%f, %f, %f), row03_new=(%f, %f, %f), row6=%f, row6_new=%f\n", 
					 ncurr_g.x, ncurr_g.y, ncurr_g.z, n.x, n.y, n.z,
				  tmpv.x, tmpv.y, tmpv.z, 
				  row03.x, row03.y, row03.z,
				  cross_ng_v.x, cross_ng_v.y, cross_ng_v.z, 
				  row03_new.x, row03_new.y, row03_new.z, 
				  row[6], row[6] - dot(tmpv, tmpv));

		  }
#endif
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        int tid = Block::flattenedThreadId ();

        int shift = 0;
#if 0   //gbuf ���� 21������+6=27 ʱ
        #pragma unroll
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
              gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[i]*row[j];
          }
        }
#elif 1 //gbuf ����ǰ 3x3=9, �� orthogonal-procrustes ����ʱ @2017-6-2 17:55:44
        #pragma unroll
        for(int j=3; j<6; ++j){ //RA-B ������, ���� 3~5��Ӧ B
            #pragma unroll
            for(int i=0; i<3; ++i){ //0~2 ��Ӧ A
                gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[j] * row[i];
            }
        }
#endif
      }//operator () (int dummy) const


    };

    __global__ void
    combinedKernel2 (const Combined2 cs) 
    {
      cs ();
    }

    __global__ void
    combinedKernel2_nmap (const Combined2 cs) 
    {
      cs (1234567); //dummy ����
    }

    struct CombinedPrevSpace
    {
      enum
      {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
      };

      struct plus
      {
        __forceinline__ __device__ float
        operator () (const float_type &lhs, const volatile float_type& rhs) const 
        {
          return (lhs + rhs);
        }
      };

      Mat33 Rcurr;
      float3 tcurr;

      PtrStep<float> vmap_curr;
      PtrStep<float> nmap_curr;

      Mat33 Rprev_inv;
      float3 tprev;

      Intr intr;

      PtrStep<float> vmap_g_prev;
      PtrStep<float> nmap_g_prev;

      float distThres;
      float angleThres;

      int cols;
      int rows;

      mutable PtrStep<float_type> gbuf;

      __device__ __forceinline__ bool
      search (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = Rprev_inv * nprev_g;
        d = Rprev_inv * (vprev_g - tprev);
        s = vcurr_cp;
        return (true);
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

        float row[7];

        if (found_coresp)
        {
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        int tid = Block::flattenedThreadId ();

        int shift = 0;
        #pragma unroll
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
              gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[i]*row[j];
          }
        }
      }
    };

    __global__ void
    combinedKernelPrevSpace (const CombinedPrevSpace cs) 
    {
      cs ();
    }

  }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::estimateCombined (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf, 
                               float_type* matrixA_host, float_type* vectorB_host)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;
  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  /*
  Combined cs;

  cs.Rcurr = Rcurr;
  cs.tcurr = tcurr;

  cs.vmap_curr = vmap_curr;
  cs.nmap_curr = nmap_curr;

  cs.Rprev_inv = Rprev_inv;
  cs.tprev = tprev;

  cs.intr = intr;

  cs.vmap_g_prev = vmap_g_prev;
  cs.nmap_g_prev = nmap_g_prev;

  cs.distThres = distThres;
  cs.angleThres = angleThres;

  cs.cols = cols;
  cs.rows = rows;

//////////////////////////////

  mbuf.create (TranformReduction::TOTAL);
  if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
    gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);

  cs.gbuf = gbuf;

  combinedKernel<<<grid, block>>>(cs);
  cudaSafeCall ( hipGetLastError () );
  //cudaSafeCall(hipDeviceSynchronize());

  //printFuncAttrib(combinedKernel);

  TranformReduction tr;
  tr.gbuf = gbuf;
  tr.length = grid.x * grid.y;
  tr.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());
  */
  Combined2 cs2;

  cs2.Rcurr = Rcurr;
  cs2.tcurr = tcurr;

  cs2.vmap_curr = vmap_curr;
  cs2.nmap_curr = nmap_curr;

  cs2.Rprev_inv = Rprev_inv;
  cs2.tprev = tprev;

  cs2.intr = intr;

  cs2.vmap_g_prev = vmap_g_prev;
  cs2.nmap_g_prev = nmap_g_prev;

  cs2.distThres = distThres;
  cs2.angleThres = angleThres;

  cs2.cols = cols;
  cs2.rows = rows;

  cs2.gbuf = gbuf;

  combinedKernel2<<<grid, block>>>(cs2);
  cudaSafeCall ( hipGetLastError () );

  //zc: dbg *gbuf*
#if 0
  const int pxNUM = 640 * 480;
  //float_type gbuf_host[27];//*640*480]; //31MB ����ջ�ڴ����, ���� new
  float_type *gbuf_host = new float_type[27*pxNUM];
  gbuf.download(gbuf_host, pxNUM*sizeof(float_type));
  for(int i=0; i<27; i++){
	  float sum = 0;
	  for(int j=0; j<pxNUM; j++){
		  sum += gbuf_host[i*pxNUM + j];
	  }
	  printf("gbuf_host::sum(%d):=%f\n", i, sum);
  }
#endif

  TranformReduction tr2;
  tr2.gbuf = gbuf;
  tr2.length = cols * rows;
  tr2.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr2);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
}

//zc: nmap �ͷ���ר��, �� estimateCombined ������ combinedKernel2 �������� operator() @2017-6-1 13:11:25
void
pcl::device::estimateCombined_nmap (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf, 
                               float_type* matrixA_host, float_type* vectorB_host)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;
  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  /*
  Combined cs;

  cs.Rcurr = Rcurr;
  cs.tcurr = tcurr;

  cs.vmap_curr = vmap_curr;
  cs.nmap_curr = nmap_curr;

  cs.Rprev_inv = Rprev_inv;
  cs.tprev = tprev;

  cs.intr = intr;

  cs.vmap_g_prev = vmap_g_prev;
  cs.nmap_g_prev = nmap_g_prev;

  cs.distThres = distThres;
  cs.angleThres = angleThres;

  cs.cols = cols;
  cs.rows = rows;

//////////////////////////////

  mbuf.create (TranformReduction::TOTAL);
  if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
    gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);

  cs.gbuf = gbuf;

  combinedKernel<<<grid, block>>>(cs);
  cudaSafeCall ( hipGetLastError () );
  //cudaSafeCall(hipDeviceSynchronize());

  //printFuncAttrib(combinedKernel);

  TranformReduction tr;
  tr.gbuf = gbuf;
  tr.length = grid.x * grid.y;
  tr.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());
  */
  Combined2 cs2;

  cs2.Rcurr = Rcurr;
  cs2.tcurr = tcurr;

  cs2.vmap_curr = vmap_curr;
  cs2.nmap_curr = nmap_curr;

  cs2.Rprev_inv = Rprev_inv;
  cs2.tprev = tprev;

  cs2.intr = intr;

  cs2.vmap_g_prev = vmap_g_prev;
  cs2.nmap_g_prev = nmap_g_prev;

  cs2.distThres = distThres;
  cs2.angleThres = angleThres;

  cs2.cols = cols;
  cs2.rows = rows;

  cs2.gbuf = gbuf;

  //combinedKernel2<<<grid, block>>>(cs2);
  combinedKernel2_nmap<<<grid, block>>>(cs2); //zc
  
  cudaSafeCall ( hipGetLastError () );

  //zc: dbg *gbuf*
#if 0
  const int pxNUM = 640 * 480;
  //float_type gbuf_host[27];//*640*480]; //31MB ����ջ�ڴ����, ���� new
  float_type *gbuf_host = new float_type[27*pxNUM];
  gbuf.download(gbuf_host, pxNUM*sizeof(float_type));
  for(int i=0; i<27; i++){
	  float sum = 0;
	  for(int j=0; j<pxNUM; j++){
		  sum += gbuf_host[i*pxNUM + j];
	  }
	  printf("gbuf_host::sum(%d):=%f\n", i, sum);
  }
#endif

  TranformReduction tr2;
  tr2.gbuf = gbuf;
  tr2.length = cols * rows;
  tr2.output = mbuf;

  //TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr2);
  TransformEstimatorKernel2<<<9, TranformReduction::CTA_SIZE>>>(tr2); //9=3x3, ԭ TranformReduction::TOTAL=27
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

#if 0   //��ԭ TranformReduction::TOTAL=27
  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
#elif 1 //�� matrixA_host ����ǰ 3x3 (���� 6x6) 
  int shift = 0;
  for(int i=0; i<3; ++i)  //rows
    for(int j=0; j<3; ++j){
      float_type value = host_data[shift++];
      matrixA_host[i * 6 + j] = value;
    }

    //��-������, ��Ϊ matrixA_host ���� 66 ����
//   for(int i=0; i<9; ++i)
//       matrixA_host[i] = host_data[i];
#endif
}//estimateCombined_nmap

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::estimateCombinedPrevSpace (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf, 
                               float_type* matrixA_host, float_type* vectorB_host)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;
  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  CombinedPrevSpace cs3;

  cs3.Rcurr = Rcurr;
  cs3.tcurr = tcurr;

  cs3.vmap_curr = vmap_curr;
  cs3.nmap_curr = nmap_curr;

  cs3.Rprev_inv = Rprev_inv;
  cs3.tprev = tprev;

  cs3.intr = intr;

  cs3.vmap_g_prev = vmap_g_prev;
  cs3.nmap_g_prev = nmap_g_prev;

  cs3.distThres = distThres;
  cs3.angleThres = angleThres;

  cs3.cols = cols;
  cs3.rows = rows;

  cs3.gbuf = gbuf;

  combinedKernelPrevSpace<<<grid, block>>>(cs3);
  cudaSafeCall ( hipGetLastError () );

  TranformReduction tr2;
  tr2.gbuf = gbuf;
  tr2.length = cols * rows;
  tr2.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr2);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
}
