#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 * 
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

//#include <pcl/gpu/utils/device/block.hpp>
//#include <pcl/gpu/utils/device/funcattrib.hpp>
#include "device.hpp"
#include "zc_cuda_utils.hpp"
//#include <pcl/console/time.h> //zc: tictoc
#include <time.h>

namespace pcl
{
  namespace device
  {
    //typedef double float_type;
	typedef float float_type;

    struct Combined
    {
      enum
      {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
      };

      struct plus
      {
        __forceinline__ __device__ float
        operator () (const float_type &lhs, const volatile float_type& rhs) const 
        {
          return (lhs + rhs);
        }
      };

      Mat33 Rcurr;
      float3 tcurr;

      PtrStep<float> vmap_curr;
      PtrStep<float> nmap_curr;

      Mat33 Rprev_inv;
      float3 tprev;

      Intr intr;

      PtrStep<float> vmap_g_prev;
      PtrStep<float> nmap_g_prev;

      float distThres;
      float angleThres;

      int cols;
      int rows;

      mutable PtrStep<float_type> gbuf;

      __device__ __forceinline__ bool
      search (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

        float row[7];

        if (found_coresp)
        {
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        __shared__ float_type smem[CTA_SIZE];
        int tid = Block::flattenedThreadId ();

        int shift = 0;
        #pragma unroll
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
            __syncthreads ();
            smem[tid] = row[i] * row[j];
            __syncthreads ();

            Block::reduce<CTA_SIZE>(smem, plus ());

            if (tid == 0)
              gbuf.ptr (shift++)[blockIdx.x + gridDim.x * blockIdx.y] = smem[0];
          }
        }
      }
    };

    __global__ void
    combinedKernel (const Combined cs) 
    {
      cs ();
    }

    struct TranformReduction
    {
      enum
      {
        CTA_SIZE = 512,
        STRIDE = CTA_SIZE,

        B = 6, COLS = 6, ROWS = 6, DIAG = 6,
        UPPER_DIAG_MAT = (COLS * ROWS - DIAG) / 2 + DIAG,
        TOTAL = UPPER_DIAG_MAT + B,

        GRID_X = TOTAL
      };

      PtrStep<float_type> gbuf;
      int length;
      mutable float_type* output;

      __device__ __forceinline__ void
      operator () () const
      {
        const float_type *beg = gbuf.ptr (blockIdx.x);
        const float_type *end = beg + length;

        int tid = threadIdx.x;

        float_type sum = 0.f;
        for (const float_type *t = beg + tid; t < end; t += STRIDE)
          sum += *t;

        __shared__ float_type smem[CTA_SIZE];

        smem[tid] = sum;
        __syncthreads ();

		Block::reduce<CTA_SIZE>(smem, Combined::plus ());

        if (tid == 0)
          output[blockIdx.x] = smem[0];
      }
    };

    __global__ void
    TransformEstimatorKernel2 (const TranformReduction tr) 
    {
      tr ();
    }

    struct Combined2
    {
      enum
      {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
      };

      struct plus
      {
        __forceinline__ __device__ float
        operator () (const float_type &lhs, const volatile float_type& rhs) const 
        {
          return (lhs + rhs);
        }
      };

      Mat33 Rcurr;
      float3 tcurr;

      PtrStep<float> vmap_curr;
      PtrStep<float> nmap_curr;

      Mat33 Rprev_inv;
      float3 tprev;

      Intr intr;

      PtrStep<float> vmap_g_prev;
      PtrStep<float> nmap_g_prev;

      float distThres;
      float angleThres;

      int cols;
      int rows;

      mutable PtrStep<float_type> gbuf;

      __device__ __forceinline__ bool
      search (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];

        //zc: fix @2017-4-13 16:20:12
        if (isnan (vprev_g.x))
          return (false);

        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
      }

      __device__ __forceinline__ bool
      searchDbg (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

		//zc: dbg
		printf("\t@searchDbg: ukr.xy=(%d, %d); isnan(nprev_g.x): %d; isnan (vprev_g.x): %d\n", ukr.x, ukr.y, isnan(nprev_g.x), isnan(vprev_g.x));

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
      }//searchDbg

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

#if 0	//zc: dbg
		//if(x == 320 && y == 240){ //��
		if(x == cols/2 && y == rows/2){
			printf("@operator():: (x, y)=(%d, %d), found_coresp= %d; n=(%f, %f, %f), d=(%f, %f, %f), s=(%f, %f, %f)\n", x, y, 
				found_coresp, n.x, n.y, n.z, d.x, d.y, d.z, s.x, s.y, s.z);
		}
#endif

        float row[7];

        if (found_coresp)
        {
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
		  //zc: dbg
		  if(isnan(row[6])){ //��������ȫ��Ӧ�÷�������
			  printf("isnan(row[6]), (x,y)=(%d, %d); (rows, cols)=(%d, %d); n=(%f, %f, %f), d=(%f, %f, %f), s=(%f, %f, %f)\n", x, y, rows, cols,
				  n.x, n.y, n.z, d.x, d.y, d.z, s.x, s.y, s.z);
			  searchDbg(x, y, n, d, s);
		  }

#if 0	//����, ���ܼӵ�һ��, ��Ϊ��ͷ�������� ������С���� ��ʽ, �Ƿ������� @2017-6-1 11:06:13
		  //zc: ������ʦҪ��, ���� nmap ���ͷ���, //��ֻ�ܳͷ� R, ���� t @2017-5-31 11:16:49
		  //Ӱ�� row[0~2, 6], ��Ӱ�� row[3~5]
		  float3 ncurr;
		  ncurr.x = nmap_curr.ptr (y)[x];
		  ncurr.y = nmap_curr.ptr (y + rows)[x];
		  ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];
		  
		  float3 ncurr_g = Rcurr * ncurr;
		  if(dot(ncurr_g, n) < 0) //�жϷ���, ϣ���� nprev_g ����һ��
			  ncurr_g *= -1;

		  //ע��: n �� nprev_g 
		  float3 tmpv = ncurr_g - n;
		  *(float3*)&row[0] = *(float3*)&row[0] + cross(ncurr_g, tmpv); //3x1 ����
		  row[6] = row[6] - dot(tmpv, tmpv); //�ٱ��� ��ע������ ��-=��, ��ԭ��, �Ƶ���
#endif

#if 0
		  {
			  float3 cross_ng_v = cross(ncurr_g, tmpv);
			  float3 row03 = *(float3*)&row[0];
			  float3 row03_new = row03 + cross_ng_v;
			  //printf("ncurr_g=(%f, %f, %f), nprev_g=(%f, %f, %f)\n", ncurr_g.x, ncurr_g.y, ncurr_g.z, n.x, n.y, n.z);
			  printf("ncurr_g=(%f, %f, %f), nprev_g=(%f, %f, %f)\
					 \ntmpv=(%f, %f, %f), row03=(%f, %f, %f), cross_ng_v=(%f, %f, %f), row03_new=(%f, %f, %f), row6=%f, row6_new=%f\n", 
					 ncurr_g.x, ncurr_g.y, ncurr_g.z, n.x, n.y, n.z,
				  tmpv.x, tmpv.y, tmpv.z, 
				  row03.x, row03.y, row03.z,
				  cross_ng_v.x, cross_ng_v.y, cross_ng_v.z, 
				  row03_new.x, row03_new.y, row03_new.z, 
				  row[6], row[6] - dot(tmpv, tmpv));

		  }
#endif
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        int tid = Block::flattenedThreadId ();

        int shift = 0;
        #pragma unroll
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
              gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[i]*row[j];
          }
        }
      }

      __device__ __forceinline__ void
      operator () (int dummy) const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

#if 0	//zc: dbg
		//if(x == 320 && y == 240){ //��
		if(x == cols/2 && y == rows/2){
			printf("@operator():: (x, y)=(%d, %d), found_coresp= %d; n=(%f, %f, %f), d=(%f, %f, %f), s=(%f, %f, %f)\n", x, y, 
				found_coresp, n.x, n.y, n.z, d.x, d.y, d.z, s.x, s.y, s.z);
		}
#endif

        float row[7];

        if (found_coresp)
        {
#if 0	//��, ����Ҫ�� nmap �ͷ���, ���Ż� R, ���� t (ϵ������) @2017-6-1 14:47:31
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
#elif 1
          float3 ncurr;
          ncurr.x = nmap_curr.ptr (y)[x];
          ncurr.y = nmap_curr.ptr (y + rows)[x];
          ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];
          
          float3 ncurr_g = Rcurr * ncurr;
          if(dot(ncurr_g, n) < 0) //�жϷ���, ϣ���� nprev_g ����һ��
              ncurr_g *= -1;

          //ע��: n �� nprev_g 
#if 0	//�˴�˼·�� argmin(SUM(|(R*ng~-ng)*(ng~-ng)|))
          //������, ����, ���� @2017-6-2 17:48:13
          float3 tmpv = ncurr_g - n;
          *(float3*)&row[0] = cross(ncurr_g, tmpv); //3x1 ����
          row[3] = row[4] = row[5] = 0.f;
          row[6] = -dot(tmpv, tmpv); //�ٱ��� ��ע������ ��-=��, ��ԭ��, �Ƶ���

#elif 1	//������ʵ���� orthogonal-procrustes ����, ���ﳢ�Բ��л����� @2017-6-2 17:48:49
          //Ŀ��: argmin|RA-B| ==> R = svd(B*At), ���� A/B �� 3*N, �� BAt~3x3
          //row0~2 -> ncurr_g, 3~5-> nprev_g, [6]����, ������
          //֮�� gbuf[27] ֻ��ǰ 3x3=9 ��, 
          *(float3*)&row[0] = ncurr_g;
          *(float3*)&row[3] = n;
          row[6] = 0;
#endif

#endif
		  //zc: dbg
		  if(isnan(row[6])){ //��������ȫ��Ӧ�÷�������
			  printf("isnan(row[6]), (x,y)=(%d, %d); (rows, cols)=(%d, %d); n=(%f, %f, %f), d=(%f, %f, %f), s=(%f, %f, %f)\n", x, y, rows, cols,
				  n.x, n.y, n.z, d.x, d.y, d.z, s.x, s.y, s.z);
			  searchDbg(x, y, n, d, s);
		  }

#if 0	//����, ���ܼӵ�һ��, ��Ϊ��ͷ�������� ������С���� ��ʽ, �Ƿ������� @2017-6-1 11:06:13
		  //zc: ������ʦҪ��, ���� nmap ���ͷ���, //��ֻ�ܳͷ� R, ���� t @2017-5-31 11:16:49
		  //Ӱ�� row[0~2, 6], ��Ӱ�� row[3~5]
		  float3 ncurr;
		  ncurr.x = nmap_curr.ptr (y)[x];
		  ncurr.y = nmap_curr.ptr (y + rows)[x];
		  ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];
		  
		  float3 ncurr_g = Rcurr * ncurr;
		  if(dot(ncurr_g, n) < 0) //�жϷ���, ϣ���� nprev_g ����һ��
			  ncurr_g *= -1;

		  //ע��: n �� nprev_g 
		  float3 tmpv = ncurr_g - n;
		  *(float3*)&row[0] = *(float3*)&row[0] + cross(ncurr_g, tmpv); //3x1 ����
		  row[6] = row[6] - dot(tmpv, tmpv); //�ٱ��� ��ע������ ��-=��, ��ԭ��, �Ƶ���
#endif

#if 0
		  {
			  float3 cross_ng_v = cross(ncurr_g, tmpv);
			  float3 row03 = *(float3*)&row[0];
			  float3 row03_new = row03 + cross_ng_v;
			  //printf("ncurr_g=(%f, %f, %f), nprev_g=(%f, %f, %f)\n", ncurr_g.x, ncurr_g.y, ncurr_g.z, n.x, n.y, n.z);
			  printf("ncurr_g=(%f, %f, %f), nprev_g=(%f, %f, %f)\
					 \ntmpv=(%f, %f, %f), row03=(%f, %f, %f), cross_ng_v=(%f, %f, %f), row03_new=(%f, %f, %f), row6=%f, row6_new=%f\n", 
					 ncurr_g.x, ncurr_g.y, ncurr_g.z, n.x, n.y, n.z,
				  tmpv.x, tmpv.y, tmpv.z, 
				  row03.x, row03.y, row03.z,
				  cross_ng_v.x, cross_ng_v.y, cross_ng_v.z, 
				  row03_new.x, row03_new.y, row03_new.z, 
				  row[6], row[6] - dot(tmpv, tmpv));

		  }
#endif
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        int tid = Block::flattenedThreadId ();

        int shift = 0;
#if 0   //gbuf ���� 21������+6=27 ʱ
        #pragma unroll
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
              gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[i]*row[j];
          }
        }
#elif 1 //gbuf ����ǰ 3x3=9, �� orthogonal-procrustes ����ʱ @2017-6-2 17:55:44
        #pragma unroll
        for(int j=3; j<6; ++j){ //RA-B ������, ���� 3~5��Ӧ B
            #pragma unroll
            for(int i=0; i<3; ++i){ //0~2 ��Ӧ A
                gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[j] * row[i];
            }
        }
#endif
      }//operator () (int dummy) const


    };

    __global__ void
    combinedKernel2 (const Combined2 cs) 
    {
      cs ();
    }

    __global__ void
    combinedKernel2_nmap (const Combined2 cs) 
    {
      cs (1234567); //dummy ����
    }

    struct CombinedPrevSpace
    {
      enum
      {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
      };

      struct plus
      {
        __forceinline__ __device__ float
        operator () (const float_type &lhs, const volatile float_type& rhs) const 
        {
          return (lhs + rhs);
        }
      };

      Mat33 Rcurr;
      float3 tcurr;

      PtrStep<float> vmap_curr;
      PtrStep<float> nmap_curr;

      Mat33 Rprev_inv;
      float3 tprev;

      Intr intr;

      PtrStep<float> vmap_g_prev;
      PtrStep<float> nmap_g_prev;

      float distThres;
      float angleThres;

      int cols;
      int rows;

      mutable PtrStep<float_type> gbuf;

      __device__ __forceinline__ bool
      search (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = Rprev_inv * nprev_g;
        d = Rprev_inv * (vprev_g - tprev);
        s = vcurr_cp;
        return (true);
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s;
        bool found_coresp = false;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

        float row[7];

        if (found_coresp)
        {
          *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;
          row[6] = dot (n, d - s);
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        int tid = Block::flattenedThreadId ();

        int shift = 0;
        #pragma unroll
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
              gbuf.ptr (shift++)[ (blockIdx.x + gridDim.x * blockIdx.y) * CTA_SIZE + tid ] = row[i]*row[j];
          }
        }
      }
    };

    __global__ void
    combinedKernelPrevSpace (const CombinedPrevSpace cs) 
    {
      cs ();
    }

    __global__ void
    scaleDepth (const PtrStepSz<ushort> depth, PtrStep<float> scaled, const Intr intr);

    //__device__ __forceinline__ float3
    //getVoxelGCoo (int x, int y, int z) /*const*/
    //{
    //  float3 coo = make_float3 (x, y, z);
    //  coo += 0.5f;         //shift to cell center;

    //  coo.x *= cell_size.x;
    //  coo.y *= cell_size.y;
    //  coo.z *= cell_size.z;

    //  return coo;
    //}

    //��--count how many vxls are used in the cost function optimization
    __device__ int vxlValidCnt_device;
    __device__ float sumS2sErr_device;

    //�ο� tsdf23_v11_remake
    __global__ void
    estimateCombined_s2s_kernel(const PtrStepSz<float> depthScaled, PtrStep<short2> volume, PtrStep<short2> volume2, 
        const float tranc_dist, const float eta, //s2s (delta, eta)
        const Mat33 Rcurr_inv, const float3 tcurr, float6 xi_prev, 
        const Intr intr, const float3 cell_size, 
        PtrStep<float> gbuf, 
        int3 vxlDbg)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      //if (x >= VOLUME_X || y >= VOLUME_Y)
      if (x <= 1 || y <= 1 || x >= VOLUME_X-1 || y >= VOLUME_Y-1) //�� pos2 �õ�������tsdf�������ݶ�
          return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      //model /global
      short2* pos = volume.ptr (y) + x;
      int elem_step = volume.step * VOLUME_Y / sizeof(short2);

      //curr
      short2* pos2 = volume2.ptr (y) + x;
      int elem_step2 = volume2.step * VOLUME_Y / sizeof(short2);

      //float row[7]; //��ѭ����

      for (int z = 0; z < VOLUME_Z;
      //for (int z = 1; z < VOLUME_Z - 1; //������ idx ƫ��
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos += elem_step,
           pos2 += elem_step2)
      {
        bool doDbgPrint = false;
        if(x > 0 && y > 0 && z > 0 //����Ĭ�� 000, ����Чֵ, �������Ӵ˼��
            && vxlDbg.x == x && vxlDbg.y == y && vxlDbg.z == z)
            doDbgPrint = true;
        
        //�ŵ�for��ǰ��
        if(0 == z){ //for-loop-begin-set-0
            int tid = Block::flattenedThreadId ();
            int total_tid = (blockIdx.x + gridDim.x * blockIdx.y) * (blockDim.x * blockDim.y) + tid;

            int shift = 0;

            #pragma unroll
            for (int i = 0; i < 6; ++i)        //rows
                #pragma unroll
                for (int j = i; j < 7; ++j)          // cols + b
                    gbuf.ptr (shift++)[ total_tid ] = 0;
        }
        if(0 == z || VOLUME_Z -1 == z){
            if(doDbgPrint)
                printf("######################(0 == z || VOLUME_Z -1 == z)\n");
            continue;
        }

        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if(doDbgPrint)
            printf("esti-s2s_kernel:: coo.xy:(%d, %d)\n", coo.x, coo.y);

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          //float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          //float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          //if(doDbgPrint){
          //    printf("Dp_scaled, sdf, tranc_dist, %f, %f, %f\n", Dp_scaled, sdf, tranc_dist);
          //    printf("coo.xy:(%d, %d)\n", coo.x, coo.y);
          //}

          //float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          ////if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          //if (Dp_scaled != 0 && sdf >= -eta) //meters //�Ƚ� eta , ���� delta (tdist)
          //{
          //  //read and unpack
          //  float tsdf_prev;
          //  int weight_prev;
          //  unpack_tsdf (*pos, tsdf_prev, weight_prev);
          //  //v17, Ϊ��� v17 �� w ������ĩλ�����λ, ���������޸�: unpack ʱ /2, pack ʱ *2; @2018-1-22 02:01:27
          //  weight_prev = weight_prev >> 1;

          //  if(weight_prev == 0)
          //      continue;

          //  //��--prev, ��--curr
          //  float tsdf = fmin (1.0f, sdf * tranc_dist_inv);
          //  if(sdf < -tranc_dist)
          //      tsdf = -1.0f;

          //  const int Wrk = 1; //sdf>-eta ==> wrk=0 �Ѿ�����˵���, �������ж�
          //  if(tsdf == tsdf_prev)
          //      continue;

          float tsdf1;
          int weight1;
          unpack_tsdf (*pos, tsdf1, weight1);
          //v17, Ϊ��� v17 �� w ������ĩλ�����λ, ���������޸�: unpack ʱ /2, pack ʱ *2; @2018-1-22 02:01:27
          weight1 = weight1 >> 1;

          float tsdf2;
          int weight2;
          unpack_tsdf (*pos2, tsdf2, weight2);
          weight2 = weight2 >> 1;

          if(doDbgPrint)
              printf("F1/F2, W1/W2: %f, %f, %d, %d; pos1/2-addr: %p, %p, %d; %p, %p, %d\n", tsdf1, tsdf2, weight1, weight2, 
              (void*)pos, (void*)volume.ptr(), pos-volume.ptr(), (void*)pos2, (void*)volume2.ptr(), pos2-volume2.ptr());

          float row[7]; //���Ըķ�ѭ����, Ӧ�޲��

          if(0 != weight1 && 0 != weight2 && tsdf1 != tsdf2){
              //+++++++++++++++PhiFuncGradients
              //�ο� tsdf23normal_hack, ����: ����һ��, ���� cell-sz Ӧ��Ҳ��
              const float qnan = numeric_limits<float>::quiet_NaN();

              float3 dPhi_dX = make_float3(qnan, qnan, qnan);

              //const float m2mm = 1e3;

              float Fn, Fp;
              int Wn = 0, Wp = 0;
              unpack_tsdf (*(pos2 + elem_step2), Fn, Wn);
              unpack_tsdf (*(pos2 - elem_step2), Fp, Wp);
              Wn >>= 1; Wp >>= 1;
              if(doDbgPrint)
                  printf("\tz-Fn/Fp, Wn/Wp: %f, %f, %d, %d;\n", Fn, Fp, Wn, Wp);

              if(Wn != 0 && Wp != 0)
                  dPhi_dX.z = (Fn - Fp)/(2*cell_size.z); //csz in meters
                  //dPhi_dX.z = (Fn - Fp)/(2*cell_size.z*m2mm);
              else
                  continue;

              unpack_tsdf (*(pos2 + volume2.step/sizeof(short2) ), Fn, Wn);
              unpack_tsdf (*(pos2 - volume2.step/sizeof(short2) ), Fp, Wp);
              Wn >>= 1; Wp >>= 1;
              if(doDbgPrint)
                  printf("\ty-Fn/Fp, Wn/Wp: %f, %f, %d, %d;\n", Fn, Fp, Wn, Wp);

              if(Wn != 0 && Wp != 0)
                  dPhi_dX.y = (Fn - Fp)/(2*cell_size.y);
                  //dPhi_dX.y = (Fn - Fp)/(2*cell_size.y*m2mm);
              else
                  continue;

              unpack_tsdf (*(pos2 + 1), Fn, Wn);
              unpack_tsdf (*(pos2 - 1), Fp, Wp);
              Wn >>= 1; Wp >>= 1;
              if(doDbgPrint)
                  printf("\tx-Fn/Fp, Wn/Wp: %f, %f, %d, %d;\n", Fn, Fp, Wn, Wp);

              if(Wn != 0 && Wp != 0)
                  dPhi_dX.x = (Fn - Fp)/(2*cell_size.x);
                  //dPhi_dX.x = (Fn - Fp)/(2*cell_size.x*m2mm);
              else
                  continue;

              if(doDbgPrint)
                  printf("dPhi_dX.xyz: %f, %f, %f\n", dPhi_dX.x, dPhi_dX.y, dPhi_dX.z);

              //concatenate_matrix<<Eigen::MatrixXd::Identity(3,3),-selfCross(trans_point);
              //Eigen::Matrix<double, 1, 6> twist_partial = gradient * concatenate_matrix;
              //��д 1*3��3*6 = 1*6, ��ʽ8 chain rule
              *(float3*)&row[0] = dPhi_dX;
              //�Ƶ�����: g��u^= g^��u
              float3 pt_g; //in meters
              pt_g.x = v_g_x;
              pt_g.y = v_g_y;
              pt_g.z = v_g_z;
              //pt_g*=m2mm;

              //*(float3*)&row[3] = cross(dPhi_dX, pt_g);
              //*(float3*)&row[3] *= -1;
              *(float3*)&row[3] = cross(pt_g, dPhi_dX); //dPhi*(-^pt)

              //row[6] = dot(tsdf1 - tsdf2 + dot(*(float6*)&row[0], xi_prev), *(float3*)&row[0]);
          }
          else
              //row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;
              continue;

          atomicAdd(&vxlValidCnt_device, 1);
          atomicAdd(&sumS2sErr_device, (tsdf1-tsdf2)*(tsdf1-tsdf2) );

          int tid = Block::flattenedThreadId ();
          int total_tid = (blockIdx.x + gridDim.x * blockIdx.y) * (blockDim.x * blockDim.y) + tid;

          int shift = 0;

          //���������Ƿ� 0==z:
          #pragma unroll
          for (int i = 0; i < 6; ++i)        //rows
              #pragma unroll
              //for (int j = i; j < 7; ++j)          // cols + b
              for (int j = i; j < 6; ++j)          // cols, ����������, �ķ�Ϊ�����
                  gbuf.ptr (shift++)[ total_tid ] += row[i]*row[j]; //+=, NOT =

          //��֮ǰ����: gbuf:21+6 ֮ǰ6������, �����������
          //���� shift==21
          //float tmp = tsdf1 - tsdf2 + dot(*(float6*)&row[0], xi_prev);
          float tmp = tsdf2 - tsdf1 + dot(*(float6*)&row[0], xi_prev);
          float6 b = *(float6*)&row[0];
          b *= tmp;
          gbuf.ptr(shift++)[ total_tid ] += b.x;
          gbuf.ptr(shift++)[ total_tid ] += b.y;
          gbuf.ptr(shift++)[ total_tid ] += b.z;
          gbuf.ptr(shift++)[ total_tid ] += b.a;
          gbuf.ptr(shift++)[ total_tid ] += b.b;
          gbuf.ptr(shift++)[ total_tid ] += b.c;
        }//if-coo.xy >0 && <(rows,cols)
      }//for-z

    }//estimateCombined_s2s_kernel
  }//namespace device
}//namespace pcl


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::estimateCombined (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf, 
                               float_type* matrixA_host, float_type* vectorB_host)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;
  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  /*
  Combined cs;

  cs.Rcurr = Rcurr;
  cs.tcurr = tcurr;

  cs.vmap_curr = vmap_curr;
  cs.nmap_curr = nmap_curr;

  cs.Rprev_inv = Rprev_inv;
  cs.tprev = tprev;

  cs.intr = intr;

  cs.vmap_g_prev = vmap_g_prev;
  cs.nmap_g_prev = nmap_g_prev;

  cs.distThres = distThres;
  cs.angleThres = angleThres;

  cs.cols = cols;
  cs.rows = rows;

//////////////////////////////

  mbuf.create (TranformReduction::TOTAL);
  if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
    gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);

  cs.gbuf = gbuf;

  combinedKernel<<<grid, block>>>(cs);
  cudaSafeCall ( hipGetLastError () );
  //cudaSafeCall(hipDeviceSynchronize());

  //printFuncAttrib(combinedKernel);

  TranformReduction tr;
  tr.gbuf = gbuf;
  tr.length = grid.x * grid.y;
  tr.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());
  */
  Combined2 cs2;

  cs2.Rcurr = Rcurr;
  cs2.tcurr = tcurr;

  cs2.vmap_curr = vmap_curr;
  cs2.nmap_curr = nmap_curr;

  cs2.Rprev_inv = Rprev_inv;
  cs2.tprev = tprev;

  cs2.intr = intr;

  cs2.vmap_g_prev = vmap_g_prev;
  cs2.nmap_g_prev = nmap_g_prev;

  cs2.distThres = distThres;
  cs2.angleThres = angleThres;

  cs2.cols = cols;
  cs2.rows = rows;

  cs2.gbuf = gbuf;

  combinedKernel2<<<grid, block>>>(cs2);
  cudaSafeCall ( hipGetLastError () );

  //zc: dbg *gbuf*
#if 0
  const int pxNUM = 640 * 480;
  //float_type gbuf_host[27];//*640*480]; //31MB ����ջ�ڴ����, ���� new
  float_type *gbuf_host = new float_type[27*pxNUM];
  gbuf.download(gbuf_host, pxNUM*sizeof(float_type));
  for(int i=0; i<27; i++){
	  float sum = 0;
	  for(int j=0; j<pxNUM; j++){
		  sum += gbuf_host[i*pxNUM + j];
	  }
	  printf("gbuf_host::sum(%d):=%f\n", i, sum);
  }
#endif

  TranformReduction tr2;
  tr2.gbuf = gbuf;
  tr2.length = cols * rows;
  tr2.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr2);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
}

void
pcl::device::estimateCombined_s2s(const PtrStepSz<ushort>& depth_raw, const Intr& intr, const float3& volume_size, 
        const Mat33& Rcurr_inv, const float3& tcurr, const float6& xi_prev, 
        float tranc_dist, PtrStep<short2> volume, PtrStep<short2> volume2,
        //float delta, 
        float eta, //s2s TSDF param, delta is tranc_dist, 
        DeviceArray2D<float>& gbuf, DeviceArray<float>& mbuf, float* matrixA_host, float* vectorB_host,
        DeviceArray2D<float>& depthScaled, int &vxlValidCnt, float &sum_s2s_err, int3 vxlDbg /*= int3()*/)
{
    //pcl::console::TicToc tt;
    clock_t begt = clock();
  depthScaled.create (depth_raw.rows, depth_raw.cols);

  dim3 block_scale (32, 8);
  dim3 grid_scale (divUp (depth_raw.cols, block_scale.x), divUp (depth_raw.rows, block_scale.y));

  //scales depth along ray and converts mm -> meters. 
  scaleDepth<<<grid_scale, block_scale>>>(depth_raw, depthScaled, intr);
  cudaSafeCall ( hipGetLastError () );

  integrateTsdfVolume_s2s(/*depth_raw,*/ intr, volume_size, Rcurr_inv, tcurr,
      tranc_dist, eta, volume2, depthScaled, vxlDbg); //���� set vol-2
  printf("integrateTsdfVolume_s2s-volume2"); 
  //tt.toc_print();
  printf(" %d\n", clock()-begt);

  float3 cell_size;
  cell_size.x = volume_size.x / VOLUME_X;
  cell_size.y = volume_size.y / VOLUME_Y;
  cell_size.z = volume_size.z / VOLUME_Z;

  //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
  dim3 block (16, 16);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

  //vxlValidCnt_device
  //cudaSafeCall(hipMemset(&vxlValidCnt_device, 0, sizeof(int)) );
  int dummy0 = 0;
  cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(vxlValidCnt_device), &dummy0, sizeof(int)) );
  int dummy0f = 0;
  cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(sumS2sErr_device), &dummy0f, sizeof(float)) );

  estimateCombined_s2s_kernel<<<grid, block>>>(depthScaled, volume, volume2,
      tranc_dist, eta, Rcurr_inv, tcurr, xi_prev, intr, cell_size, 
      gbuf, 
      vxlDbg);    

  cudaSafeCall(hipMemcpyFromSymbol(&vxlValidCnt, HIP_SYMBOL(vxlValidCnt_device), sizeof(vxlValidCnt)) );
  cudaSafeCall(hipMemcpyFromSymbol(&sum_s2s_err, HIP_SYMBOL(sumS2sErr_device), sizeof(sum_s2s_err)) );

  TranformReduction tr2;
  tr2.gbuf = gbuf;
  //tr2.length = cols * rows;
  tr2.length = VOLUME_X * VOLUME_Y;
  tr2.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr2);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

//   int shift = 0;
//   for (int i = 0; i < 6; ++i)  //rows
//       for (int j = i; j < 7; ++j)    // cols + b
//       {
//           float_type value = host_data[shift++];
//           if (j == 6)       // vector b
//               vectorB_host[i] = value;
//           else
//               matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
//       }
  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
      //for (int j = i; j < 7; ++j)    // cols + b
      for (int j = i; j < 6; ++j)    // cols
          matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = host_data[shift++];
  
  for (int i = 0; i < 6; ++i)  //�����, �������ֵ
      vectorB_host[i] = host_data[shift++];
}//estimateCombined_s2s

//zc: nmap �ͷ���ר��, �� estimateCombined ������ combinedKernel2 �������� operator() @2017-6-1 13:11:25
void
pcl::device::estimateCombined_nmap (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf, 
                               float_type* matrixA_host, float_type* vectorB_host)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;
  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  /*
  Combined cs;

  cs.Rcurr = Rcurr;
  cs.tcurr = tcurr;

  cs.vmap_curr = vmap_curr;
  cs.nmap_curr = nmap_curr;

  cs.Rprev_inv = Rprev_inv;
  cs.tprev = tprev;

  cs.intr = intr;

  cs.vmap_g_prev = vmap_g_prev;
  cs.nmap_g_prev = nmap_g_prev;

  cs.distThres = distThres;
  cs.angleThres = angleThres;

  cs.cols = cols;
  cs.rows = rows;

//////////////////////////////

  mbuf.create (TranformReduction::TOTAL);
  if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
    gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);

  cs.gbuf = gbuf;

  combinedKernel<<<grid, block>>>(cs);
  cudaSafeCall ( hipGetLastError () );
  //cudaSafeCall(hipDeviceSynchronize());

  //printFuncAttrib(combinedKernel);

  TranformReduction tr;
  tr.gbuf = gbuf;
  tr.length = grid.x * grid.y;
  tr.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());
  */
  Combined2 cs2;

  cs2.Rcurr = Rcurr;
  cs2.tcurr = tcurr;

  cs2.vmap_curr = vmap_curr;
  cs2.nmap_curr = nmap_curr;

  cs2.Rprev_inv = Rprev_inv;
  cs2.tprev = tprev;

  cs2.intr = intr;

  cs2.vmap_g_prev = vmap_g_prev;
  cs2.nmap_g_prev = nmap_g_prev;

  cs2.distThres = distThres;
  cs2.angleThres = angleThres;

  cs2.cols = cols;
  cs2.rows = rows;

  cs2.gbuf = gbuf;

  //combinedKernel2<<<grid, block>>>(cs2);
  combinedKernel2_nmap<<<grid, block>>>(cs2); //zc
  
  cudaSafeCall ( hipGetLastError () );

  //zc: dbg *gbuf*
#if 0
  const int pxNUM = 640 * 480;
  //float_type gbuf_host[27];//*640*480]; //31MB ����ջ�ڴ����, ���� new
  float_type *gbuf_host = new float_type[27*pxNUM];
  gbuf.download(gbuf_host, pxNUM*sizeof(float_type));
  for(int i=0; i<27; i++){
	  float sum = 0;
	  for(int j=0; j<pxNUM; j++){
		  sum += gbuf_host[i*pxNUM + j];
	  }
	  printf("gbuf_host::sum(%d):=%f\n", i, sum);
  }
#endif

  TranformReduction tr2;
  tr2.gbuf = gbuf;
  tr2.length = cols * rows;
  tr2.output = mbuf;

  //TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr2);
  TransformEstimatorKernel2<<<9, TranformReduction::CTA_SIZE>>>(tr2); //9=3x3, ԭ TranformReduction::TOTAL=27
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

#if 0   //��ԭ TranformReduction::TOTAL=27
  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
#elif 1 //�� matrixA_host ����ǰ 3x3 (���� 6x6) 
  int shift = 0;
  for(int i=0; i<3; ++i)  //rows
    for(int j=0; j<3; ++j){
      float_type value = host_data[shift++];
      matrixA_host[i * 6 + j] = value;
    }

    //��-������, ��Ϊ matrixA_host ���� 66 ����
//   for(int i=0; i<9; ++i)
//       matrixA_host[i] = host_data[i];
#endif
}//estimateCombined_nmap

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::estimateCombinedPrevSpace (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf, 
                               float_type* matrixA_host, float_type* vectorB_host)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;
  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  CombinedPrevSpace cs3;

  cs3.Rcurr = Rcurr;
  cs3.tcurr = tcurr;

  cs3.vmap_curr = vmap_curr;
  cs3.nmap_curr = nmap_curr;

  cs3.Rprev_inv = Rprev_inv;
  cs3.tprev = tprev;

  cs3.intr = intr;

  cs3.vmap_g_prev = vmap_g_prev;
  cs3.nmap_g_prev = nmap_g_prev;

  cs3.distThres = distThres;
  cs3.angleThres = angleThres;

  cs3.cols = cols;
  cs3.rows = rows;

  cs3.gbuf = gbuf;

  combinedKernelPrevSpace<<<grid, block>>>(cs3);
  cudaSafeCall ( hipGetLastError () );

  TranformReduction tr2;
  tr2.gbuf = gbuf;
  tr2.length = cols * rows;
  tr2.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr2);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
}
